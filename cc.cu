//
// Created by gxl on 2021/1/5.
//
#include "cc.cuh"

void conventionParticipateCC(string ccPath) {
    cout << "===============conventionParticipateCC==============" << endl;
    uint testNumNodes = 0;
    ulong testNumEdge = 0;
    unsigned long transferSum = 0;
    uint *nodePointersI;
    uint *edgeList;
    auto startReadGraph = std::chrono::steady_clock::now();
    ifstream infile(ccPath, ios::in | ios::binary);
    infile.read((char *) &testNumNodes, sizeof(uint));
    uint numEdge = 0;
    infile.read((char *) &numEdge, sizeof(uint));
    testNumEdge = numEdge;
    cout << "vertex num: " << testNumNodes << " edge num: " << testNumEdge << endl;
    nodePointersI = new uint[testNumNodes];
    infile.read((char *) nodePointersI, sizeof(uint) * testNumNodes);
    edgeList = new uint[testNumEdge];
    infile.read((char *) edgeList, sizeof(uint) * testNumEdge);
    infile.close();
    unsigned long max_partition_size;
    unsigned long total_gpu_size;
    getMaxPartitionSize(max_partition_size, total_gpu_size, testNumNodes, 0.9, sizeof(uint), 5);
    uint partitionNum;
    if (testNumEdge > max_partition_size) {
        partitionNum = testNumEdge / max_partition_size + 1;
    } else {
        partitionNum = 1;
    }

    uint *degree = new uint[testNumNodes];
    uint *value = new uint[testNumNodes];
    bool *isActiveNodeList = new bool[testNumNodes];
    CommonPartitionInfo *partitionInfoList = new CommonPartitionInfo[partitionNum];
    bool *needTransferPartition = new bool[partitionNum];
    for (uint i = 0; i < testNumNodes; i++) {
        isActiveNodeList[i] = true;
        value[i] = i;
        if (i + 1 < testNumNodes) {
            degree[i] = nodePointersI[i + 1] - nodePointersI[i];
        } else {
            degree[i] = testNumEdge - nodePointersI[i];
        }
        if (degree[i] > max_partition_size) {
            cout << "node " << i << " degree > maxPartition " << endl;
            return;
        }
    }
    for (uint i = 0; i < partitionNum; i++) {
        partitionInfoList[i].startVertex = -1;
        partitionInfoList[i].endVertex = -1;
        partitionInfoList[i].nodePointerOffset = -1;
        partitionInfoList[i].partitionEdgeSize = -1;
    }
    int tempPartitionIndex = 0;
    uint tempNodeIndex = 0;
    while (tempNodeIndex < testNumNodes) {
        if (partitionInfoList[tempPartitionIndex].startVertex == -1) {
            partitionInfoList[tempPartitionIndex].startVertex = tempNodeIndex;
            partitionInfoList[tempPartitionIndex].endVertex = tempNodeIndex;
            partitionInfoList[tempPartitionIndex].nodePointerOffset = nodePointersI[tempNodeIndex];
            partitionInfoList[tempPartitionIndex].partitionEdgeSize = degree[tempNodeIndex];
            tempNodeIndex++;
        } else {
            if (partitionInfoList[tempPartitionIndex].partitionEdgeSize + degree[tempNodeIndex] > max_partition_size) {
                tempPartitionIndex++;
            } else {
                partitionInfoList[tempPartitionIndex].endVertex = tempNodeIndex;
                partitionInfoList[tempPartitionIndex].partitionEdgeSize += degree[tempNodeIndex];
                tempNodeIndex++;
            }
        }
    }

    uint *degreeD;
    bool *isActiveNodeListD;
    bool *nextActiveNodeListD;
    uint *nodePointerListD;
    uint *partitionEdgeListD;
    uint *valueD;

    hipMalloc(&degreeD, testNumNodes * sizeof(uint));
    hipMalloc(&valueD, testNumNodes * sizeof(uint));
    hipMalloc(&isActiveNodeListD, testNumNodes * sizeof(bool));
    hipMalloc(&nextActiveNodeListD, testNumNodes * sizeof(bool));
    hipMalloc(&nodePointerListD, testNumNodes * sizeof(uint));
    hipMalloc(&partitionEdgeListD, max_partition_size * sizeof(uint));

    hipMemcpy(degreeD, degree, testNumNodes * sizeof(uint), hipMemcpyHostToDevice);
    hipMemcpy(nodePointerListD, nodePointersI, testNumNodes * sizeof(uint), hipMemcpyHostToDevice);
    hipMemset(nextActiveNodeListD, 0, testNumNodes * sizeof(bool));
    //cacaulate the active node And make active node array
    dim3 grid = dim3(56, 1, 1);
    dim3 block = dim3(1024, 1, 1);

    int testTimes = 1;
    long timeSum = 0;
    for (int i = 0; i < testTimes; i++) {
        for (int j = 0; j < testNumNodes; j++) {
            isActiveNodeList[j] = true;
            value[j] = j;
        }
        hipMemcpy(valueD, value, testNumNodes * sizeof(uint), hipMemcpyHostToDevice);
        uint activeSum = 0;
        int iteration = 0;

        auto startProcessing = std::chrono::steady_clock::now();
        while (true) {
            uint activeNodeNum = 0;
            checkNeedTransferPartitionOpt(needTransferPartition, partitionInfoList, isActiveNodeList, partitionNum,
                                          testNumNodes, activeNodeNum);
            if (activeNodeNum <= 0) {
                break;
            } else {
                //cout << "iteration " << iteration << " activeNodes " << activeNodeNum << endl;
                activeSum += activeNodeNum;
            }
            hipMemcpy(isActiveNodeListD, isActiveNodeList, testNumNodes * sizeof(bool), hipMemcpyHostToDevice);
            for (int j = 0; j < partitionNum; j++) {
                if (needTransferPartition[j]) {
                    hipMemcpy(partitionEdgeListD, edgeList + partitionInfoList[j].nodePointerOffset,
                               partitionInfoList[j].partitionEdgeSize * sizeof(uint), hipMemcpyHostToDevice);
                    transferSum += partitionInfoList[j].partitionEdgeSize;
                    ccKernel_CommonPartition<<<grid, block>>>(partitionInfoList[j].startVertex,
                                                              partitionInfoList[j].endVertex,
                                                              partitionInfoList[j].nodePointerOffset,
                                                              isActiveNodeListD, nodePointerListD,
                                                              partitionEdgeListD, degreeD, valueD,
                                                              nextActiveNodeListD);
                    hipDeviceSynchronize();
                    gpuErrorcheck(hipPeekAtLastError())
                }
            }
            hipMemcpy(isActiveNodeList, nextActiveNodeListD, testNumNodes * sizeof(bool), hipMemcpyDeviceToHost);
            hipMemset(nextActiveNodeListD, 0, testNumNodes * sizeof(bool));
            iteration++;
        }

        cout << "cpu transfer to gpu " << transferSum * sizeof(uint) << "byte" << endl;
        cout << " activeSum " << activeSum << endl;
        auto endRead = std::chrono::steady_clock::now();
        long durationRead = std::chrono::duration_cast<std::chrono::milliseconds>(endRead - startProcessing).count();
        cout << " finish time : " << durationRead << " ms" << endl;
    }

    free(nodePointersI);
    free(edgeList);
    free(degree);
    free(isActiveNodeList);
    hipFree(isActiveNodeListD);
    hipFree(nextActiveNodeListD);
    hipFree(nodePointerListD);
    hipFree(partitionEdgeListD);
    //todo free partitionInfoList needTransferPartition
}

int needCpu = 0;
int notNeedCpu = 0;

long processingTimeSum = 0;
long cpuTimeSum = 0;
long allTimeSum = 0;
long validSwapSum = 0;
int trestSum = 0;

void ccShare(string ccPath) {
    uint testNumNodes = 0;
    ulong testNumEdge = 0;
    uint *nodePointersI;
    uint *edgeList;

    auto startReadGraph = std::chrono::steady_clock::now();
    ifstream infile(ccPath, ios::in | ios::binary);
    infile.read((char *) &testNumNodes, sizeof(uint));
    uint numEdge = 0;
    infile.read((char *) &numEdge, sizeof(uint));
    testNumEdge = numEdge;
    cout << "vertex num: " << testNumNodes << " edge num: " << testNumEdge << endl;
    gpuErrorcheck(hipMallocManaged(&nodePointersI, (testNumNodes + 1) * sizeof(uint)));
    infile.read((char *) nodePointersI, sizeof(uint) * testNumNodes);
    gpuErrorcheck(hipMallocManaged(&edgeList, (numEdge) * sizeof(uint)));
    hipMemAdvise(nodePointersI, (testNumNodes + 1) * sizeof(uint), hipMemAdviseSetReadMostly, 0);
    hipMemAdvise(edgeList, (numEdge) * sizeof(uint), hipMemAdviseSetReadMostly, 0);
    infile.read((char *) edgeList, sizeof(uint) * testNumEdge);
    infile.close();
    //preprocessData(nodePointersI, edgeList, testNumNodes, testNumEdge);
    auto endReadGraph = std::chrono::steady_clock::now();
    long durationReadGraph = std::chrono::duration_cast<std::chrono::milliseconds>(
            endReadGraph - startReadGraph).count();
    cout << "read graph time : " << durationReadGraph << "ms" << endl;
    int testTimes = 1;
    long timeSum = 0;
    for (int i = 0; i < testTimes; i++) {
        timeSum += ccCaculateInShare(testNumNodes, testNumEdge, nodePointersI, edgeList);
        //timeSum += bfsCaculateInShare(testNumNodes, testNumEdge, nodePointersI, edgeList, 53037907);
        break;
    }
    cout << "need cpu " << needCpu << " not need cpu " << notNeedCpu << endl;
    cout << "processingTime " << processingTimeSum / testTimes << " cpu time " << cpuTimeSum / testTimes << " all Time "
         << allTimeSum / testTimes << endl;
    cout << "mean time is " << timeSum / testTimes << endl;
    cout << "mean validSwapSum is " << validSwapSum / testTimes << endl;
    cout << trestSum << endl;
}

long ccCaculateInShare(uint testNumNodes, uint testNumEdge, uint *nodePointersI, uint *edgeList) {
    auto start = std::chrono::steady_clock::now();
    uint *degree;
    uint *value;
    //uint *recordActiveNodes = new uint[testNumNodes];
    gpuErrorcheck(hipMallocManaged(&degree, testNumNodes * sizeof(uint)));
    gpuErrorcheck(hipMallocManaged(&value, testNumNodes * sizeof(uint)));

    auto startPreCaculate = std::chrono::steady_clock::now();
    for (uint i = 0; i < testNumNodes - 1; i++) {
        degree[i] = nodePointersI[i + 1] - nodePointersI[i];
    }

    degree[testNumNodes - 1] = testNumEdge - nodePointersI[testNumNodes - 1];
    bool *label;
    gpuErrorcheck(hipMallocManaged(&label, testNumNodes * sizeof(bool)));
    for (uint i = 0; i < testNumNodes; i++) {
        label[i] = true;
        value[i] = i;
    }
    uint *activeNodeList;
    hipMallocManaged(&activeNodeList, testNumNodes * sizeof(uint));
    //cacaulate the active node And make active node array
    uint *activeNodeLabelingD;
    gpuErrorcheck(hipMallocManaged(&activeNodeLabelingD, testNumNodes * sizeof(unsigned int)));
    uint *activeNodeLabelingPrefixD;
    gpuErrorcheck(hipMallocManaged(&activeNodeLabelingPrefixD, testNumNodes * sizeof(unsigned int)));
    dim3 grid = dim3(56, 1, 1);
    dim3 block = dim3(1024, 1, 1);

    setLabeling<<<grid, block>>>(testNumNodes, label, activeNodeLabelingD);
    thrust::device_ptr<unsigned int> ptr_labeling(activeNodeLabelingD);
    thrust::device_ptr<unsigned int> ptr_labeling_prefixsum(activeNodeLabelingPrefixD);
    uint activeNodesNum = thrust::reduce(ptr_labeling, ptr_labeling + testNumNodes);
    int iter = 0;
    uint nodeSum = activeNodesNum;

    auto endPreCaculate = std::chrono::steady_clock::now();
    long durationPreCaculate = std::chrono::duration_cast<std::chrono::milliseconds>(
            endPreCaculate - startPreCaculate).count();
    cout << "durationPreCaculate time : " << durationPreCaculate << " ms" << endl;
    auto startProcessing = std::chrono::steady_clock::now();
    //vector<vector<uint>> visitRecordByIteration;
    while (activeNodesNum > 0) {
        iter++;
        thrust::exclusive_scan(ptr_labeling, ptr_labeling + testNumNodes, ptr_labeling_prefixsum);
        setActiveNodeArray<<<grid, block>>>(testNumNodes, activeNodeList, label, activeNodeLabelingPrefixD);
        setLabelDefault<<<grid, block>>>(activeNodesNum, activeNodeList, label);
        cc_kernel<<<grid, block>>>(activeNodesNum, activeNodeList, nodePointersI, degree, edgeList, value, label);
        hipDeviceSynchronize();
        //visitRecordByIteration.push_back(countDataByIteration(testNumEdge, testNumNodes, nodePointersI, degree, activeNodeLabelingD));
        gpuErrorcheck(hipPeekAtLastError());
        setLabeling<<<grid, block>>>(testNumNodes, label, activeNodeLabelingD);
        activeNodesNum = thrust::reduce(ptr_labeling, ptr_labeling + testNumNodes);
        nodeSum += activeNodesNum;
        cout << "iter: " << iter << " activeNodes: " << activeNodesNum << endl;
    }
    hipDeviceSynchronize();
    //writeTrunkVistInIteration(visitRecordByIteration, "./CountByIterationCC.txt");

    cout << "nodeSum: " << nodeSum << endl;

    auto endRead = std::chrono::steady_clock::now();
    long durationRead = std::chrono::duration_cast<std::chrono::milliseconds>(endRead - startProcessing).count();
    cout << "iter sum is " << iter << " finish time : " << durationRead << " ms" << endl;

    hipFree(degree);
    hipFree(label);
    hipFree(value);
    hipFree(activeNodeList);
    hipFree(activeNodeLabelingD);
    hipFree(activeNodeLabelingPrefixD);
    return durationRead;
}

void ccKernelThread(uint staticNodeNum, uint *activeNodeListD,
                    uint *staticNodePointerD, uint *degreeD,
                    uint *staticEdgeListD, uint *valueD,
                    uint *isActiveD1,
                    uint *isActiveD2,
                    bool *isFinishedManaged, dim3 grid, dim3 block, hipStream_t steamStatic) {
    uint itr = 0;
    bool isFinishedHost = true;
    do {
        itr++;
        isFinishedHost = true;
        hipMemcpy(isFinishedManaged, &isFinishedHost, sizeof(bool), hipMemcpyHostToDevice);
        cc_kernelStaticSwapOpt2Label<<<grid, block, 0, steamStatic>>>(staticNodeNum, activeNodeListD,
                                                                      staticNodePointerD, degreeD,
                                                                      staticEdgeListD, valueD,
                                                                      itr % 2 == 1 ? isActiveD1 : isActiveD2,
                                                                      itr % 2 == 1 ? isActiveD2 : isActiveD1,
                                                                      isFinishedManaged);
        hipDeviceSynchronize();
        hipMemcpy(&isFinishedHost, isFinishedManaged, sizeof(bool), hipMemcpyDeviceToHost);
        isFinishedHost = true;
    } while (!isFinishedHost);
}

void ccOpt(string ccPath, float adviseK) {
    uint testNumNodes = 0;
    ulong testNumEdge = 0;
    uint *nodePointersI;
    uint *edgeList;
    bool isUseShare = true;

    auto startReadGraph = std::chrono::steady_clock::now();
    ifstream infile(ccPath, ios::in | ios::binary);
    infile.read((char *) &testNumNodes, sizeof(uint));
    uint numEdge = 0;
    infile.read((char *) &numEdge, sizeof(uint));
    testNumEdge = numEdge;
    cout << "vertex num: " << testNumNodes << " edge num: " << testNumEdge << endl;

    nodePointersI = new uint[testNumNodes + 1];
    infile.read((char *) nodePointersI, sizeof(uint) * testNumNodes);
    edgeList = new uint[testNumEdge + 1];
    infile.read((char *) edgeList, sizeof(uint) * testNumEdge);
    infile.close();
    auto endReadGraph = std::chrono::steady_clock::now();
    long durationReadGraph = std::chrono::duration_cast<std::chrono::milliseconds>(
            endReadGraph - startReadGraph).count();
    cout << "read graph time : " << durationReadGraph << "ms" << endl;
    int testTimes = 1;
    long timeSum = 0;
    for (int i = 0; i < testTimes; i++) {
        //timeSum += ccCaculateCommonMemoryInnerAsync(testNumNodes, testNumEdge, nodePointersI, edgeList, adviseK);
        //break;
        timeSum += ccCaculateCommonMemoryInnerAsyncRandom(testNumNodes, testNumEdge, nodePointersI, edgeList, adviseK);
        cout << i << "========================================" << endl;
    }
}

struct TempConnectedComponent {
    uint index;
    uint nodeSum;
    uint edgeSum;
};

long ccCaculateCommonMemoryInnerAsync(uint testNumNodes, uint testNumEdge, uint *nodePointersI, uint *edgeList,
                                      float adviseK) {
    cout << "=========ccCaculateCommonMemoryInnerAsync1========" << endl;
    ulong edgeIterationMax = 0;
    auto start = std::chrono::steady_clock::now();
    auto startPreCaculate = std::chrono::steady_clock::now();
    //CPU
    long durationRead;
    ulong transferSum = 0;
    unsigned long max_partition_size;
    unsigned long total_gpu_size;
    uint maxStaticNode = 0;
    uint *degree;
    uint *value;
    uint *label;
    bool *isInStatic;
    uint *overloadNodeList;
    uint *staticNodePointer;
    uint *activeNodeList;
    uint *activeOverloadNodePointers;
    vector<PartEdgeListInfo> partEdgeListInfoArr;
    /*
     * overloadEdgeList overload edge list in every iteration
     * */
    uint *overloadEdgeList;
    FragmentData *fragmentData;
    bool isFromTail = true;
    //GPU
    uint *staticEdgeListD;
    uint *overloadEdgeListD;
    bool *isInStaticD;
    uint *overloadNodeListD;
    uint *staticNodePointerD;
    uint *nodePointerD;
    uint *degreeD;
    // async need two labels
    uint *isActiveD1;
    uint *isActiveD2;
    uint *isStaticActive;
    uint *isOverloadActive;
    uint *valueD;
    uint *activeNodeListD;
    uint *activeNodeLabelingPrefixD;
    uint *activeOverloadNodePointersD;
    uint *activeOverloadDegreeD;
    bool *isFinishedDevice;

    degree = new uint[testNumNodes];
    value = new uint[testNumNodes];
    label = new uint[testNumNodes];
    isInStatic = new bool[testNumNodes];
    overloadNodeList = new uint[testNumNodes];
    staticNodePointer = new uint[testNumNodes];
    activeNodeList = new uint[testNumNodes];
    activeOverloadNodePointers = new uint[testNumNodes];

    getMaxPartitionSize(max_partition_size, total_gpu_size, testNumNodes, adviseK, sizeof(uint), testNumEdge, 15);
    //caculate degree
    uint meanDegree = testNumEdge / testNumNodes;
    cout << " meanDegree " << meanDegree << endl;
    uint degree0Sum = 0;
    for (uint i = 0; i < testNumNodes - 1; i++) {
        if (nodePointersI[i] > testNumEdge) {
            cout << i << "   " << nodePointersI[i] << endl;
            break;
        }
        degree[i] = nodePointersI[i + 1] - nodePointersI[i];
    }
    degree[testNumNodes - 1] = testNumEdge - nodePointersI[testNumNodes - 1];
    memcpy(staticNodePointer, nodePointersI, testNumNodes * sizeof(uint));

    //caculate static staticEdgeListD
    gpuErrorcheck(hipMalloc(&isFinishedDevice, 1 * sizeof(bool)));
    gpuErrorcheck(hipMalloc(&staticEdgeListD, max_partition_size * sizeof(uint)));
    auto startmove = std::chrono::steady_clock::now();
    gpuErrorcheck(hipMemcpy(staticEdgeListD, edgeList, max_partition_size * sizeof(uint), hipMemcpyHostToDevice));
    auto endMove = std::chrono::steady_clock::now();
    long testDuration = std::chrono::duration_cast<std::chrono::milliseconds>(
            endMove - startmove).count();

    gpuErrorcheck(hipMalloc(&isInStaticD, testNumNodes * sizeof(bool)))
    gpuErrorcheck(hipMalloc(&overloadNodeListD, testNumNodes * sizeof(uint)));
    gpuErrorcheck(hipMalloc(&staticNodePointerD, testNumNodes * sizeof(uint)))
    gpuErrorcheck(hipMemcpy(staticNodePointerD, nodePointersI, testNumNodes * sizeof(uint), hipMemcpyHostToDevice));
    gpuErrorcheck(hipMalloc(&nodePointerD, testNumNodes * sizeof(uint)));
    gpuErrorcheck(hipMemcpy(nodePointerD, nodePointersI, testNumNodes * sizeof(uint), hipMemcpyHostToDevice));

    for (uint i = 0; i < testNumNodes; i++) {
        label[i] = 1;
        value[i] = i;

        if (nodePointersI[i] < max_partition_size && (nodePointersI[i] + degree[i] - 1) < max_partition_size) {
            isInStatic[i] = true;
            if (i > maxStaticNode) maxStaticNode = i;
        } else {
            isInStatic[i] = false;
        }
    }
    hipMemcpy(isInStaticD, isInStatic, testNumNodes * sizeof(bool), hipMemcpyHostToDevice);
    cout << "max_partition_size: " << max_partition_size << "  maxStaticNode: " << maxStaticNode << endl;
    uint partOverloadSize = total_gpu_size - max_partition_size;
    uint overloadSize = testNumEdge - nodePointersI[maxStaticNode + 1];
    cout << " partOverloadSize " << partOverloadSize << " overloadSize " << overloadSize << endl;
    overloadEdgeList = (uint *) malloc(overloadSize * sizeof(uint));
    gpuErrorcheck(hipMalloc(&overloadEdgeListD, partOverloadSize * sizeof(uint)));
    gpuErrorcheck(hipMalloc(&degreeD, testNumNodes * sizeof(uint)));
    gpuErrorcheck(hipMalloc(&isActiveD1, testNumNodes * sizeof(uint)));
    gpuErrorcheck(hipMalloc(&isActiveD2, testNumNodes * sizeof(uint)));
    gpuErrorcheck(hipMalloc(&isStaticActive, testNumNodes * sizeof(uint)));
    gpuErrorcheck(hipMalloc(&isOverloadActive, testNumNodes * sizeof(uint)));
    gpuErrorcheck(hipMalloc(&valueD, testNumNodes * sizeof(uint)));
    gpuErrorcheck(hipMalloc(&activeNodeLabelingPrefixD, testNumNodes * sizeof(unsigned int)));
    gpuErrorcheck(hipMalloc(&activeNodeListD, testNumNodes * sizeof(uint)));
    gpuErrorcheck(hipMalloc(&activeOverloadNodePointersD, testNumNodes * sizeof(uint)));
    gpuErrorcheck(hipMalloc(&activeOverloadDegreeD, testNumNodes * sizeof(uint)));
    gpuErrorcheck(hipMemcpy(degreeD, degree, testNumNodes * sizeof(uint), hipMemcpyHostToDevice));
    gpuErrorcheck(hipMemcpy(valueD, value, testNumNodes * sizeof(uint), hipMemcpyHostToDevice));
    gpuErrorcheck(hipMemcpy(isActiveD1, label, testNumNodes * sizeof(uint), hipMemcpyHostToDevice));
    gpuErrorcheck(hipMemset(isActiveD2, 0, testNumNodes * sizeof(uint)));
    gpuErrorcheck(hipMemset(isStaticActive, 0, testNumNodes * sizeof(uint)));
    gpuErrorcheck(hipMemset(isOverloadActive, 0, testNumNodes * sizeof(uint)));

    //cacaulate the active node And make active node array
    dim3 grid = dim3(56, 1, 1);
    dim3 block = dim3(1024, 1, 1);

    //setLabeling<<<grid, block>>>(testNumNodes, labelD, activeNodeLabelingD);
    thrust::device_ptr<unsigned int> ptr_labeling(isActiveD1);
    thrust::device_ptr<unsigned int> ptr_labelingTest(isActiveD2);
    thrust::device_ptr<unsigned int> ptr_labeling_static(isStaticActive);
    thrust::device_ptr<unsigned int> ptr_labeling_overload(isOverloadActive);
    thrust::device_ptr<unsigned int> ptr_labeling_prefixsum(activeNodeLabelingPrefixD);
    thrust::device_ptr<unsigned int> ptrOverloadDegree(activeOverloadDegreeD);
    thrust::device_ptr<unsigned int> ptrOverloadPrefixsum(activeOverloadNodePointersD);

    uint activeNodesNum = thrust::reduce(ptr_labeling, ptr_labeling + testNumNodes);
    int iter = 0;
    uint nodeSum = activeNodesNum;
    ulong overloadEdgeSum = 0;
    auto startCpu = std::chrono::steady_clock::now();
    auto endReadCpu = std::chrono::steady_clock::now();
    long durationReadCpu = 0;

    auto startSwap = std::chrono::steady_clock::now();
    auto endSwap = std::chrono::steady_clock::now();
    long durationSwap = 0;

    auto startGpuProcessing = std::chrono::steady_clock::now();
    auto endGpuProcessing = std::chrono::steady_clock::now();
    long durationGpuProcessing = 0;

    auto startOverloadGpuProcessing = std::chrono::steady_clock::now();
    auto endOverloadGpuProcessing = std::chrono::steady_clock::now();
    long durationOverloadGpuProcessing = 0;

    auto startPreGpuProcessing = std::chrono::steady_clock::now();
    auto endPreGpuProcessing = std::chrono::steady_clock::now();
    long durationPreGpuProcessing = 0;
    auto endPreCaculate = std::chrono::steady_clock::now();
    long durationPreCaculate = std::chrono::duration_cast<std::chrono::milliseconds>(
            endPreCaculate - startPreCaculate).count();
    cout << "durationPreCaculate time : " << durationPreCaculate << " ms" << endl;
    hipStream_t steamStatic, streamDynamic;
    hipStreamCreate(&steamStatic);
    hipStreamCreate(&streamDynamic);
    auto startMemoryTraverse = std::chrono::steady_clock::now();
    auto endMemoryTraverse = std::chrono::steady_clock::now();
    long durationMemoryTraverse = 0;
    auto startProcessing = std::chrono::steady_clock::now();
    //uint cursorStartSwap = staticFragmentNum + 1;
    uint swapValidNodeSum = 0;
    uint swapValidEdgeSum = 0;
    uint swapNotValidNodeSum = 0;
    uint swapNotValidEdgeSum = 0;
    uint visitEdgeSum = 0;
    uint swapInEdgeSum = 0;
    uint headSum;
    uint tailSum;

    while (activeNodesNum > 0) {
        iter++;
        //cout << "iter " << iter << " activeNodesNum " << activeNodesNum << endl;
        startPreGpuProcessing = std::chrono::steady_clock::now();
        //cleanStaticAndOverloadLabel<<<grid, block>>>(testNumNodes, isStaticActive, isOverloadActive);
        setStaticAndOverloadLabel<<<grid, block>>>(testNumNodes, isActiveD1, isStaticActive, isOverloadActive,
                                                   isInStaticD);
        uint staticNodeNum = thrust::reduce(ptr_labeling_static, ptr_labeling_static + testNumNodes);
        if (staticNodeNum > 0) {
            //cout << "iter " << iter << " staticNodeNum " << staticNodeNum << endl;
            thrust::exclusive_scan(ptr_labeling_static, ptr_labeling_static + testNumNodes, ptr_labeling_prefixsum);
            setStaticActiveNodeArray<<<grid, block>>>(testNumNodes, activeNodeListD, isStaticActive,
                                                      activeNodeLabelingPrefixD);
        }
        uint overloadNodeNum = thrust::reduce(ptr_labeling_overload, ptr_labeling_overload + testNumNodes);
        uint overloadEdgeNum = 0;
        if (overloadNodeNum > 0) {
            //cout << "iter " << iter << " overloadNodeNum " << overloadNodeNum << endl;

            thrust::exclusive_scan(ptr_labeling_overload, ptr_labeling_overload + testNumNodes, ptr_labeling_prefixsum);
            setOverloadNodePointerSwap<<<grid, block>>>(testNumNodes, overloadNodeListD, activeOverloadDegreeD,
                                                        isOverloadActive,
                                                        activeNodeLabelingPrefixD, degreeD);

            thrust::exclusive_scan(ptrOverloadDegree, ptrOverloadDegree + overloadNodeNum, activeOverloadNodePointersD);
            overloadEdgeNum = thrust::reduce(thrust::device, ptrOverloadDegree,
                                             ptrOverloadDegree + overloadNodeNum, 0);
            //cout << "iter " << iter << " overloadEdgeNum " << overloadEdgeNum << endl;
            overloadEdgeSum += overloadEdgeNum;
            if (overloadEdgeNum > edgeIterationMax) {
                edgeIterationMax = overloadEdgeNum;
            }
        }
        endPreGpuProcessing = std::chrono::steady_clock::now();
        durationPreGpuProcessing += std::chrono::duration_cast<std::chrono::milliseconds>(
                endPreGpuProcessing - startPreGpuProcessing).count();
        startGpuProcessing = std::chrono::steady_clock::now();
        mixDynamicPartLabel<<<grid, block, 0, steamStatic>>>(staticNodeNum, 0, activeNodeListD, isActiveD1,
                                                             isActiveD2);
        thread staticCCKernel = thread(ccKernelThread, staticNodeNum, activeNodeListD, staticNodePointerD, degreeD,
                                       staticEdgeListD, valueD, isActiveD1, isActiveD2, isFinishedDevice, grid, block,
                                       steamStatic);
        /*if (staticCCKernel.joinable()) {
            staticCCKernel.join();
        }*/

        if (overloadNodeNum > 0) {
            startCpu = std::chrono::steady_clock::now();
            /*hipMemcpyAsync(staticActiveNodeList, activeNodeListD, activeNodesNum * sizeof(uint), hipMemcpyDeviceToHost,
                            streamDynamic);*/
            hipMemcpyAsync(overloadNodeList, overloadNodeListD, overloadNodeNum * sizeof(uint), hipMemcpyDeviceToHost,
                            streamDynamic);
            hipMemcpyAsync(activeOverloadNodePointers, activeOverloadNodePointersD, overloadNodeNum * sizeof(uint),
                            hipMemcpyDeviceToHost, streamDynamic);

            int threadNum = 20;
            if (overloadNodeNum < 50) {
                threadNum = 1;
            }
            thread runThreads[threadNum];
            for (int i = 0; i < threadNum; i++) {
                runThreads[i] = thread(fillDynamic,
                                       i,
                                       threadNum,
                                       0,
                                       overloadNodeNum,
                                       degree,
                                       activeOverloadNodePointers,
                                       nodePointersI,
                                       overloadNodeList,
                                       overloadEdgeList,
                                       edgeList);
            }

            for (unsigned int t = 0; t < threadNum; t++) {
                runThreads[t].join();
            }
            caculatePartInfoForEdgeList(activeOverloadNodePointers, overloadNodeList, degree, partEdgeListInfoArr,
                                        overloadNodeNum, partOverloadSize, overloadEdgeNum);

            endReadCpu = std::chrono::steady_clock::now();
            durationReadCpu += std::chrono::duration_cast<std::chrono::milliseconds>(endReadCpu - startCpu).count();
            if (staticCCKernel.joinable()) {
                staticCCKernel.join();
            }
            endGpuProcessing = std::chrono::steady_clock::now();
            durationGpuProcessing += std::chrono::duration_cast<std::chrono::milliseconds>(
                    endGpuProcessing - startGpuProcessing).count();

            for (auto &i : partEdgeListInfoArr) {
                startMemoryTraverse = std::chrono::steady_clock::now();
                gpuErrorcheck(hipMemcpy(overloadEdgeListD, overloadEdgeList +
                                                            activeOverloadNodePointers[i.partStartIndex],
                                         i.partEdgeNums * sizeof(uint), hipMemcpyHostToDevice))
                transferSum += i.partEdgeNums;
                endMemoryTraverse = std::chrono::steady_clock::now();
                durationMemoryTraverse += std::chrono::duration_cast<std::chrono::milliseconds>(
                        endMemoryTraverse - startMemoryTraverse).count();
                /*cout << "iter " << iter << " part " << i << " durationMemoryTraverse "
                     << durationMemoryTraverse << endl;*/
                startOverloadGpuProcessing = std::chrono::steady_clock::now();
                mixDynamicPartLabel<<<grid, block, 0, streamDynamic>>>(i.partActiveNodeNums,
                                                                       i.partStartIndex,
                                                                       overloadNodeListD, isActiveD1,
                                                                       isActiveD2);
                uint itr = 0;
                bool isFinishedHost = true;
                do {
                    itr++;
                    isFinishedHost = true;
                    hipMemcpy(isFinishedDevice, &isFinishedHost, sizeof(bool), hipMemcpyHostToDevice);

                    cc_kernelDynamicSwap2Label<<<grid, block, 0, streamDynamic>>>(i.partStartIndex,
                                                                                  i.partActiveNodeNums,
                                                                                  overloadNodeListD, degreeD,
                                                                                  valueD, itr % 2 == 1 ? isActiveD1
                                                                                                       : isActiveD2,
                                                                                  itr % 2 == 1 ? isActiveD2
                                                                                               : isActiveD1,
                                                                                  overloadEdgeListD,
                                                                                  activeOverloadNodePointersD,
                                                                                  isFinishedDevice);
                    hipDeviceSynchronize();
                    hipMemcpy(&isFinishedHost, isFinishedDevice, sizeof(bool), hipMemcpyDeviceToHost);
                    isFinishedHost = true;
                } while (!isFinishedHost);
                endOverloadGpuProcessing = std::chrono::steady_clock::now();
                durationOverloadGpuProcessing += std::chrono::duration_cast<std::chrono::milliseconds>(
                        endOverloadGpuProcessing - startOverloadGpuProcessing).count();
                /*cout << "iter " << iter << " part " << i << " durationOverloadGpuProcessing "
                     << durationOverloadGpuProcessing << endl;*/
            }
            gpuErrorcheck(hipPeekAtLastError())

        } else {
            if (staticCCKernel.joinable()) {
                staticCCKernel.join();
            }
            endGpuProcessing = std::chrono::steady_clock::now();
            durationGpuProcessing += std::chrono::duration_cast<std::chrono::milliseconds>(
                    endGpuProcessing - startGpuProcessing).count();
        }
        mixCommonLabel<<<grid, block, 0, streamDynamic>>>(testNumNodes, isActiveD1, isActiveD2);
        //hipDeviceSynchronize();
        //cout << "mixDynamicPartLabel" << " =========hipDeviceSynchronize()==========" << endl;
        //hipMemcpy(label, isActiveD, testNumNodes * sizeof(uint), hipMemcpyDeviceToHost);
        startPreGpuProcessing = std::chrono::steady_clock::now();
        activeNodesNum = thrust::reduce(ptr_labeling, ptr_labeling + testNumNodes);
        nodeSum += activeNodesNum;
        endPreGpuProcessing = std::chrono::steady_clock::now();
        durationPreGpuProcessing += std::chrono::duration_cast<std::chrono::milliseconds>(
                endPreGpuProcessing - startPreGpuProcessing).count();
    }
    hipDeviceSynchronize();
    hipMemcpy(value, valueD, testNumNodes * sizeof(uint), hipMemcpyDeviceToHost);
    transferSum += max_partition_size;
    cout << "transferSum: " << transferSum * 4 << "byte" << endl;
    cout << "iterationSum " << iter << endl;
    double edgeIterationAvg = (double) overloadEdgeSum / (double) testNumEdge / iter;
    double edgeIterationMaxAvg = (double) edgeIterationMax / (double) testNumEdge;
    cout << "edgeIterationAvg " << edgeIterationAvg << " edgeIterationMaxAvg " << edgeIterationMaxAvg << endl;
    cout << "nodeSum: " << nodeSum << endl;
    auto endRead = std::chrono::steady_clock::now();
    durationRead = std::chrono::duration_cast<std::chrono::milliseconds>(endRead - startProcessing).count();
    cout << "finish time : " << durationRead << " ms" << endl;
    cout << "total time : " << durationRead + testDuration << " ms" << endl;
    cout << "cpu time : " << durationReadCpu << " ms" << endl;
    cout << "pre fact processing time : " << durationGpuProcessing << " ms" << endl;
    cout << "overload fact processing time : " << durationOverloadGpuProcessing << " ms" << endl;
    cout << "durationMemoryTraverse : " << durationMemoryTraverse << " ms" << endl;
    cout << "durationOverloadGpuProcessing : " << durationOverloadGpuProcessing << " ms" << endl;

    cout << "gpu pre processing time : " << durationPreGpuProcessing << " ms" << endl;
    cout << "swap processing time : " << durationSwap << " ms" << endl;
    cout << "overloadEdgeSum : " << overloadEdgeSum << " " << endl;

    cout << "swapValidNodeSum " << swapValidNodeSum << " swapValidEdgeSum " << swapValidEdgeSum << endl;
    cout << "swapNotValidNodeSum " << swapNotValidNodeSum << " swapNotValidEdgeSum " << swapNotValidEdgeSum
         << " visitSum " << visitEdgeSum << " swapInEdgeSum " << swapInEdgeSum << endl;

    cout << "headSum " << headSum << " tailSum " << tailSum << endl;
    /*hipFree(nodePointerD);
    hipFree(staticEdgeListD);
    hipFree(degreeD);
    hipFree(isActiveD1);
    hipFree(isActiveD2);
    hipFree(valueD);
    hipFree(activeNodeListD);
    hipFree(activeNodeLabelingPrefixD);
    hipFree(activeOverloadNodePointersD);
    hipFree(activeOverloadDegreeD);
    hipFree(isInStaticD);
    hipFree(staticNodePointerD);
    hipFree(overloadNodeListD);

    delete[]            label;
    delete[]            degree;
    delete[]            value;
    delete[]            staticActiveNodeList;
    delete[]            activeOverloadNodePointers;
    delete[] isInStatic;
    delete[] overloadNodeList;
    delete[] staticNodePointer;
    delete[] fragmentData;
    return durationRead;*/
}

void conventionParticipateCCInLong() {
    cout << "===============conventionParticipateCCInLong==============" << endl;
    uint testNumNodes = 0;
    ulong testNumEdge = 0;
    unsigned long transferSum = 0;
    uint *nodePointersI;
    uint *edgeList;
    auto startReadGraph = std::chrono::steady_clock::now();
    ifstream infile(testGraphPath, ios::in | ios::binary);
    infile.read((char *) &testNumNodes, sizeof(uint));
    uint numEdge = 0;
    infile.read((char *) &numEdge, sizeof(uint));
    testNumEdge = numEdge;
    cout << "vertex num: " << testNumNodes << " edge num: " << testNumEdge << endl;
    nodePointersI = new uint[testNumNodes];
    infile.read((char *) nodePointersI, sizeof(uint) * testNumNodes);
    edgeList = new uint[testNumEdge];
    infile.read((char *) edgeList, sizeof(uint) * testNumEdge);
    infile.close();
    unsigned long max_partition_size;
    unsigned long total_gpu_size;
    getMaxPartitionSize(max_partition_size, total_gpu_size, testNumNodes, 0.9, sizeof(uint), 5);
    uint partitionNum;
    if (testNumEdge > max_partition_size) {
        partitionNum = testNumEdge / max_partition_size + 1;
    } else {
        partitionNum = 1;
    }

    uint *degree = new uint[testNumNodes];
    uint *value = new uint[testNumNodes];
    bool *isActiveNodeList = new bool[testNumNodes];
    CommonPartitionInfo *partitionInfoList = new CommonPartitionInfo[partitionNum];
    bool *needTransferPartition = new bool[partitionNum];
    for (uint i = 0; i < testNumNodes; i++) {
        isActiveNodeList[i] = true;
        value[i] = i;
        if (i + 1 < testNumNodes) {
            degree[i] = nodePointersI[i + 1] - nodePointersI[i];
        } else {
            degree[i] = testNumEdge - nodePointersI[i];
        }
        if (degree[i] > max_partition_size) {
            cout << "node " << i << " degree > maxPartition " << endl;
            return;
        }
    }
    for (uint i = 0; i < partitionNum; i++) {
        partitionInfoList[i].startVertex = -1;
        partitionInfoList[i].endVertex = -1;
        partitionInfoList[i].nodePointerOffset = -1;
        partitionInfoList[i].partitionEdgeSize = -1;
    }
    int tempPartitionIndex = 0;
    uint tempNodeIndex = 0;
    while (tempNodeIndex < testNumNodes) {
        if (partitionInfoList[tempPartitionIndex].startVertex == -1) {
            partitionInfoList[tempPartitionIndex].startVertex = tempNodeIndex;
            partitionInfoList[tempPartitionIndex].endVertex = tempNodeIndex;
            partitionInfoList[tempPartitionIndex].nodePointerOffset = nodePointersI[tempNodeIndex];
            partitionInfoList[tempPartitionIndex].partitionEdgeSize = degree[tempNodeIndex];
            tempNodeIndex++;
        } else {
            if (partitionInfoList[tempPartitionIndex].partitionEdgeSize + degree[tempNodeIndex] > max_partition_size) {
                tempPartitionIndex++;
            } else {
                partitionInfoList[tempPartitionIndex].endVertex = tempNodeIndex;
                partitionInfoList[tempPartitionIndex].partitionEdgeSize += degree[tempNodeIndex];
                tempNodeIndex++;
            }
        }
    }

    uint *degreeD;
    bool *isActiveNodeListD;
    bool *nextActiveNodeListD;
    uint *nodePointerListD;
    uint *partitionEdgeListD;
    uint *valueD;

    hipMalloc(&degreeD, testNumNodes * sizeof(uint));
    hipMalloc(&valueD, testNumNodes * sizeof(uint));
    hipMalloc(&isActiveNodeListD, testNumNodes * sizeof(bool));
    hipMalloc(&nextActiveNodeListD, testNumNodes * sizeof(bool));
    hipMalloc(&nodePointerListD, testNumNodes * sizeof(uint));
    hipMalloc(&partitionEdgeListD, max_partition_size * sizeof(uint));

    hipMemcpy(degreeD, degree, testNumNodes * sizeof(uint), hipMemcpyHostToDevice);
    hipMemcpy(nodePointerListD, nodePointersI, testNumNodes * sizeof(uint), hipMemcpyHostToDevice);
    hipMemset(nextActiveNodeListD, 0, testNumNodes * sizeof(bool));
    //cacaulate the active node And make active node array
    dim3 grid = dim3(56, 1, 1);
    dim3 block = dim3(1024, 1, 1);

    int testTimes = 1;
    long timeSum = 0;
    for (int i = 0; i < testTimes; i++) {
        for (int j = 0; j < testNumNodes; j++) {
            isActiveNodeList[j] = true;
            value[j] = j;
        }
        hipMemcpy(valueD, value, testNumNodes * sizeof(uint), hipMemcpyHostToDevice);
        uint activeSum = 0;
        int iteration = 0;

        auto startProcessing = std::chrono::steady_clock::now();
        while (true) {
            uint activeNodeNum = 0;
            checkNeedTransferPartition(needTransferPartition, partitionInfoList, isActiveNodeList, partitionNum,
                                       testNumNodes, activeNodeNum);
            if (activeNodeNum <= 0) {
                break;
            } else {
                cout << "iteration " << iteration << " activeNodes " << activeNodeNum << endl;
                activeSum += activeNodeNum;
            }
            hipMemcpy(isActiveNodeListD, isActiveNodeList, testNumNodes * sizeof(bool), hipMemcpyHostToDevice);
            for (int j = 0; j < partitionNum; j++) {
                if (needTransferPartition[j]) {
                    hipMemcpy(partitionEdgeListD, edgeList + partitionInfoList[j].nodePointerOffset,
                               partitionInfoList[j].partitionEdgeSize * sizeof(uint), hipMemcpyHostToDevice);
                    transferSum += partitionInfoList[j].partitionEdgeSize;
                    ccKernel_CommonPartition<<<grid, block>>>(partitionInfoList[j].startVertex,
                                                              partitionInfoList[j].endVertex,
                                                              partitionInfoList[j].nodePointerOffset,
                                                              isActiveNodeListD, nodePointerListD,
                                                              partitionEdgeListD, degreeD, valueD,
                                                              nextActiveNodeListD);
                    hipDeviceSynchronize();
                    gpuErrorcheck(hipPeekAtLastError())
                }
            }
            hipMemcpy(isActiveNodeList, nextActiveNodeListD, testNumNodes * sizeof(bool), hipMemcpyDeviceToHost);
            hipMemset(nextActiveNodeListD, 0, testNumNodes * sizeof(bool));
            iteration++;
        }

        cout << "cpu transfer to gpu " << transferSum * sizeof(uint) << "byte" << endl;
        cout << " activeSum " << activeSum << endl;
        auto endRead = std::chrono::steady_clock::now();
        long durationRead = std::chrono::duration_cast<std::chrono::milliseconds>(endRead - startProcessing).count();
        cout << " finish time : " << durationRead << " ms" << endl;
    }

    free(nodePointersI);
    free(edgeList);
    free(degree);
    free(isActiveNodeList);
    hipFree(isActiveNodeListD);
    hipFree(nextActiveNodeListD);
    hipFree(nodePointerListD);
    hipFree(partitionEdgeListD);
}

long
ccCaculateCommonMemoryInnerAsyncRecordVisit(uint testNumNodes, uint testNumEdge, uint *nodePointersI, uint *edgeList,
                                            float adviseK) {
    cout << "=========ccCaculateCommonMemoryInnerAsync1========" << endl;
    ulong edgeIterationMax = 0;
    auto start = std::chrono::steady_clock::now();
    auto startPreCaculate = std::chrono::steady_clock::now();
    //CPU
    long durationRead;
    ulong transferSum = 0;
    unsigned long max_partition_size;
    unsigned long total_gpu_size;
    uint maxStaticNode = 0;
    uint *degree;
    uint *value;
    uint *label;
    bool *isInStatic;
    uint *overloadNodeList;
    uint *staticNodePointer;
    uint *activeNodeList;
    uint *activeOverloadNodePointers;
    vector<PartEdgeListInfo> partEdgeListInfoArr;
    /*
     * overloadEdgeList overload edge list in every iteration
     * */
    uint *overloadEdgeList;
    FragmentData *fragmentData;
    bool isFromTail = true;
    //GPU
    uint *staticEdgeListD;
    uint *overloadEdgeListD;
    bool *isInStaticD;
    uint *overloadNodeListD;
    uint *staticNodePointerD;
    uint *nodePointerD;
    uint *degreeD;
    // async need two labels
    uint *isActiveD1;
    uint *isActiveD2;
    uint *isStaticActive;
    uint *isOverloadActive;
    uint *valueD;
    uint *activeNodeListD;
    uint *activeNodeLabelingPrefixD;
    uint *activeOverloadNodePointersD;
    uint *activeOverloadDegreeD;
    bool *isFinishedDevice;
    uint *vertexVisitRecord;
    uint *vertexVisitRecordD;
    vertexVisitRecord = new uint[testNumNodes];
    hipMalloc(&vertexVisitRecordD, testNumNodes * sizeof(uint));
    hipMemset(vertexVisitRecordD, 0, testNumNodes * sizeof(uint));
    degree = new uint[testNumNodes];
    value = new uint[testNumNodes];
    label = new uint[testNumNodes];
    isInStatic = new bool[testNumNodes];
    overloadNodeList = new uint[testNumNodes];
    staticNodePointer = new uint[testNumNodes];
    activeNodeList = new uint[testNumNodes];
    activeOverloadNodePointers = new uint[testNumNodes];

    getMaxPartitionSize(max_partition_size, total_gpu_size, testNumNodes, adviseK, sizeof(uint), testNumEdge, 15);
    //caculate degree
    uint meanDegree = testNumEdge / testNumNodes;
    cout << " meanDegree " << meanDegree << endl;
    uint degree0Sum = 0;
    for (uint i = 0; i < testNumNodes - 1; i++) {
        if (nodePointersI[i] > testNumEdge) {
            cout << i << "   " << nodePointersI[i] << endl;
            break;
        }
        degree[i] = nodePointersI[i + 1] - nodePointersI[i];
    }
    degree[testNumNodes - 1] = testNumEdge - nodePointersI[testNumNodes - 1];
    memcpy(staticNodePointer, nodePointersI, testNumNodes * sizeof(uint));

    //caculate static staticEdgeListD
    gpuErrorcheck(hipMalloc(&isFinishedDevice, 1 * sizeof(bool)));
    gpuErrorcheck(hipMalloc(&staticEdgeListD, max_partition_size * sizeof(uint)));
    auto startmove = std::chrono::steady_clock::now();
    gpuErrorcheck(hipMemcpy(staticEdgeListD, edgeList, max_partition_size * sizeof(uint), hipMemcpyHostToDevice));
    auto endMove = std::chrono::steady_clock::now();
    long testDuration = std::chrono::duration_cast<std::chrono::milliseconds>(
            endMove - startmove).count();

    gpuErrorcheck(hipMalloc(&isInStaticD, testNumNodes * sizeof(bool)))
    gpuErrorcheck(hipMalloc(&overloadNodeListD, testNumNodes * sizeof(uint)));
    gpuErrorcheck(hipMalloc(&staticNodePointerD, testNumNodes * sizeof(uint)))
    gpuErrorcheck(hipMemcpy(staticNodePointerD, nodePointersI, testNumNodes * sizeof(uint), hipMemcpyHostToDevice));
    gpuErrorcheck(hipMalloc(&nodePointerD, testNumNodes * sizeof(uint)));
    gpuErrorcheck(hipMemcpy(nodePointerD, nodePointersI, testNumNodes * sizeof(uint), hipMemcpyHostToDevice));

    for (uint i = 0; i < testNumNodes; i++) {
        label[i] = 1;
        value[i] = i;

        if (nodePointersI[i] < max_partition_size && (nodePointersI[i] + degree[i] - 1) < max_partition_size) {
            isInStatic[i] = true;
            if (i > maxStaticNode) maxStaticNode = i;
        } else {
            isInStatic[i] = false;
        }
    }
    hipMemcpy(isInStaticD, isInStatic, testNumNodes * sizeof(bool), hipMemcpyHostToDevice);
    cout << "max_partition_size: " << max_partition_size << "  maxStaticNode: " << maxStaticNode << endl;
    uint partOverloadSize = total_gpu_size - max_partition_size;
    uint overloadSize = testNumEdge - nodePointersI[maxStaticNode + 1];
    cout << " partOverloadSize " << partOverloadSize << " overloadSize " << overloadSize << endl;
    overloadEdgeList = (uint *) malloc(overloadSize * sizeof(uint));
    gpuErrorcheck(hipMalloc(&overloadEdgeListD, partOverloadSize * sizeof(uint)));
    gpuErrorcheck(hipMalloc(&degreeD, testNumNodes * sizeof(uint)));
    gpuErrorcheck(hipMalloc(&isActiveD1, testNumNodes * sizeof(uint)));
    gpuErrorcheck(hipMalloc(&isActiveD2, testNumNodes * sizeof(uint)));
    gpuErrorcheck(hipMalloc(&isStaticActive, testNumNodes * sizeof(uint)));
    gpuErrorcheck(hipMalloc(&isOverloadActive, testNumNodes * sizeof(uint)));
    gpuErrorcheck(hipMalloc(&valueD, testNumNodes * sizeof(uint)));
    gpuErrorcheck(hipMalloc(&activeNodeLabelingPrefixD, testNumNodes * sizeof(unsigned int)));
    gpuErrorcheck(hipMalloc(&activeNodeListD, testNumNodes * sizeof(uint)));
    gpuErrorcheck(hipMalloc(&activeOverloadNodePointersD, testNumNodes * sizeof(uint)));
    gpuErrorcheck(hipMalloc(&activeOverloadDegreeD, testNumNodes * sizeof(uint)));
    gpuErrorcheck(hipMemcpy(degreeD, degree, testNumNodes * sizeof(uint), hipMemcpyHostToDevice));
    gpuErrorcheck(hipMemcpy(valueD, value, testNumNodes * sizeof(uint), hipMemcpyHostToDevice));
    gpuErrorcheck(hipMemcpy(isActiveD1, label, testNumNodes * sizeof(uint), hipMemcpyHostToDevice));
    gpuErrorcheck(hipMemset(isActiveD2, 0, testNumNodes * sizeof(uint)));
    gpuErrorcheck(hipMemset(isStaticActive, 0, testNumNodes * sizeof(uint)));
    gpuErrorcheck(hipMemset(isOverloadActive, 0, testNumNodes * sizeof(uint)));

    //cacaulate the active node And make active node array
    dim3 grid = dim3(56, 1, 1);
    dim3 block = dim3(1024, 1, 1);

    //setLabeling<<<grid, block>>>(testNumNodes, labelD, activeNodeLabelingD);
    thrust::device_ptr<unsigned int> ptr_labeling(isActiveD1);
    thrust::device_ptr<unsigned int> ptr_labelingTest(isActiveD2);
    thrust::device_ptr<unsigned int> ptr_labeling_static(isStaticActive);
    thrust::device_ptr<unsigned int> ptr_labeling_overload(isOverloadActive);
    thrust::device_ptr<unsigned int> ptr_labeling_prefixsum(activeNodeLabelingPrefixD);
    thrust::device_ptr<unsigned int> ptrOverloadDegree(activeOverloadDegreeD);
    thrust::device_ptr<unsigned int> ptrOverloadPrefixsum(activeOverloadNodePointersD);

    uint activeNodesNum = thrust::reduce(ptr_labeling, ptr_labeling + testNumNodes);
    int iter = 0;
    uint nodeSum = activeNodesNum;
    ulong overloadEdgeSum = 0;
    auto startCpu = std::chrono::steady_clock::now();
    auto endReadCpu = std::chrono::steady_clock::now();
    long durationReadCpu = 0;

    auto startSwap = std::chrono::steady_clock::now();
    auto endSwap = std::chrono::steady_clock::now();
    long durationSwap = 0;

    auto startGpuProcessing = std::chrono::steady_clock::now();
    auto endGpuProcessing = std::chrono::steady_clock::now();
    long durationGpuProcessing = 0;

    auto startOverloadGpuProcessing = std::chrono::steady_clock::now();
    auto endOverloadGpuProcessing = std::chrono::steady_clock::now();
    long durationOverloadGpuProcessing = 0;

    auto startPreGpuProcessing = std::chrono::steady_clock::now();
    auto endPreGpuProcessing = std::chrono::steady_clock::now();
    long durationPreGpuProcessing = 0;
    auto endPreCaculate = std::chrono::steady_clock::now();
    long durationPreCaculate = std::chrono::duration_cast<std::chrono::milliseconds>(
            endPreCaculate - startPreCaculate).count();
    cout << "durationPreCaculate time : " << durationPreCaculate << " ms" << endl;
    hipStream_t steamStatic, streamDynamic;
    hipStreamCreate(&steamStatic);
    hipStreamCreate(&streamDynamic);
    auto startMemoryTraverse = std::chrono::steady_clock::now();
    auto endMemoryTraverse = std::chrono::steady_clock::now();
    long durationMemoryTraverse = 0;
    auto startProcessing = std::chrono::steady_clock::now();
    //uint cursorStartSwap = staticFragmentNum + 1;
    uint swapValidNodeSum = 0;
    uint swapValidEdgeSum = 0;
    uint swapNotValidNodeSum = 0;
    uint swapNotValidEdgeSum = 0;
    uint visitEdgeSum = 0;
    uint swapInEdgeSum = 0;
    uint headSum;
    uint tailSum;

    while (activeNodesNum > 0) {
        iter++;
        //cout << "iter " << iter << " activeNodesNum " << activeNodesNum << endl;
        startPreGpuProcessing = std::chrono::steady_clock::now();
        //cleanStaticAndOverloadLabel<<<grid, block>>>(testNumNodes, isStaticActive, isOverloadActive);
        setStaticAndOverloadLabelAndRecord<<<grid, block>>>(testNumNodes, isActiveD1, isStaticActive, isOverloadActive,
                                                            isInStaticD, vertexVisitRecordD);
        uint staticNodeNum = thrust::reduce(ptr_labeling_static, ptr_labeling_static + testNumNodes);
        if (staticNodeNum > 0) {
            //cout << "iter " << iter << " staticNodeNum " << staticNodeNum << endl;
            thrust::exclusive_scan(ptr_labeling_static, ptr_labeling_static + testNumNodes, ptr_labeling_prefixsum);
            setStaticActiveNodeArray<<<grid, block>>>(testNumNodes, activeNodeListD, isStaticActive,
                                                      activeNodeLabelingPrefixD);
        }
        uint overloadNodeNum = thrust::reduce(ptr_labeling_overload, ptr_labeling_overload + testNumNodes);
        uint overloadEdgeNum = 0;
        if (overloadNodeNum > 0) {
            //cout << "iter " << iter << " overloadNodeNum " << overloadNodeNum << endl;

            thrust::exclusive_scan(ptr_labeling_overload, ptr_labeling_overload + testNumNodes, ptr_labeling_prefixsum);
            setOverloadNodePointerSwap<<<grid, block>>>(testNumNodes, overloadNodeListD, activeOverloadDegreeD,
                                                        isOverloadActive,
                                                        activeNodeLabelingPrefixD, degreeD);

            thrust::exclusive_scan(ptrOverloadDegree, ptrOverloadDegree + overloadNodeNum, activeOverloadNodePointersD);
            overloadEdgeNum = thrust::reduce(thrust::device, ptrOverloadDegree,
                                             ptrOverloadDegree + overloadNodeNum, 0);
            //cout << "iter " << iter << " overloadEdgeNum " << overloadEdgeNum << endl;
            overloadEdgeSum += overloadEdgeNum;
            if (overloadEdgeNum > edgeIterationMax) {
                edgeIterationMax = overloadEdgeNum;
            }
        }
        endPreGpuProcessing = std::chrono::steady_clock::now();
        durationPreGpuProcessing += std::chrono::duration_cast<std::chrono::milliseconds>(
                endPreGpuProcessing - startPreGpuProcessing).count();
        startGpuProcessing = std::chrono::steady_clock::now();
        mixDynamicPartLabel<<<grid, block, 0, steamStatic>>>(staticNodeNum, 0, activeNodeListD, isActiveD1,
                                                             isActiveD2);
        thread staticCCKernel = thread(ccKernelThread, staticNodeNum, activeNodeListD, staticNodePointerD, degreeD,
                                       staticEdgeListD, valueD, isActiveD1, isActiveD2, isFinishedDevice, grid, block,
                                       steamStatic);
        if (staticCCKernel.joinable()) {
            staticCCKernel.join();
        }

        if (overloadNodeNum > 0) {
            startCpu = std::chrono::steady_clock::now();
            /*hipMemcpyAsync(staticActiveNodeList, activeNodeListD, activeNodesNum * sizeof(uint), hipMemcpyDeviceToHost,
                            streamDynamic);*/
            hipMemcpyAsync(overloadNodeList, overloadNodeListD, overloadNodeNum * sizeof(uint), hipMemcpyDeviceToHost,
                            streamDynamic);
            hipMemcpyAsync(activeOverloadNodePointers, activeOverloadNodePointersD, overloadNodeNum * sizeof(uint),
                            hipMemcpyDeviceToHost, streamDynamic);

            int threadNum = 20;
            if (overloadNodeNum < 50) {
                threadNum = 1;
            }
            thread runThreads[threadNum];
            for (int i = 0; i < threadNum; i++) {
                runThreads[i] = thread(fillDynamic,
                                       i,
                                       threadNum,
                                       0,
                                       overloadNodeNum,
                                       degree,
                                       activeOverloadNodePointers,
                                       nodePointersI,
                                       overloadNodeList,
                                       overloadEdgeList,
                                       edgeList);
            }

            for (unsigned int t = 0; t < threadNum; t++) {
                runThreads[t].join();
            }
            caculatePartInfoForEdgeList(activeOverloadNodePointers, overloadNodeList, degree, partEdgeListInfoArr,
                                        overloadNodeNum, partOverloadSize, overloadEdgeNum);

            endReadCpu = std::chrono::steady_clock::now();
            durationReadCpu += std::chrono::duration_cast<std::chrono::milliseconds>(endReadCpu - startCpu).count();
            if (staticCCKernel.joinable()) {
                staticCCKernel.join();
            }
            endGpuProcessing = std::chrono::steady_clock::now();
            durationGpuProcessing += std::chrono::duration_cast<std::chrono::milliseconds>(
                    endGpuProcessing - startGpuProcessing).count();

            for (int i = 0; i < partEdgeListInfoArr.size(); i++) {
                startMemoryTraverse = std::chrono::steady_clock::now();
                gpuErrorcheck(hipMemcpy(overloadEdgeListD, overloadEdgeList +
                                                            activeOverloadNodePointers[partEdgeListInfoArr[i].partStartIndex],
                                         partEdgeListInfoArr[i].partEdgeNums * sizeof(uint), hipMemcpyHostToDevice))
                transferSum += partEdgeListInfoArr[i].partEdgeNums;
                endMemoryTraverse = std::chrono::steady_clock::now();
                durationMemoryTraverse += std::chrono::duration_cast<std::chrono::milliseconds>(
                        endMemoryTraverse - startMemoryTraverse).count();
                /*cout << "iter " << iter << " part " << i << " durationMemoryTraverse "
                     << durationMemoryTraverse << endl;*/
                startOverloadGpuProcessing = std::chrono::steady_clock::now();
                mixDynamicPartLabel<<<grid, block, 0, streamDynamic>>>(partEdgeListInfoArr[i].partActiveNodeNums,
                                                                       partEdgeListInfoArr[i].partStartIndex,
                                                                       overloadNodeListD, isActiveD1,
                                                                       isActiveD2);
                uint itr = 0;
                bool isFinishedHost = true;
                do {
                    itr++;
                    isFinishedHost = true;
                    hipMemcpy(isFinishedDevice, &isFinishedHost, sizeof(bool), hipMemcpyHostToDevice);

                    cc_kernelDynamicSwap2Label<<<grid, block, 0, streamDynamic>>>(partEdgeListInfoArr[i].partStartIndex,
                                                                                  partEdgeListInfoArr[i].partActiveNodeNums,
                                                                                  overloadNodeListD, degreeD,
                                                                                  valueD, itr % 2 == 1 ? isActiveD1
                                                                                                       : isActiveD2,
                                                                                  itr % 2 == 1 ? isActiveD2
                                                                                               : isActiveD1,
                                                                                  overloadEdgeListD,
                                                                                  activeOverloadNodePointersD,
                                                                                  isFinishedDevice);
                    hipDeviceSynchronize();
                    hipMemcpy(&isFinishedHost, isFinishedDevice, sizeof(bool), hipMemcpyDeviceToHost);
                    isFinishedHost = true;
                } while (!isFinishedHost);
                endOverloadGpuProcessing = std::chrono::steady_clock::now();
                durationOverloadGpuProcessing += std::chrono::duration_cast<std::chrono::milliseconds>(
                        endOverloadGpuProcessing - startOverloadGpuProcessing).count();
                /*cout << "iter " << iter << " part " << i << " durationOverloadGpuProcessing "
                     << durationOverloadGpuProcessing << endl;*/
            }
            gpuErrorcheck(hipPeekAtLastError())

        } else {
            if (staticCCKernel.joinable()) {
                staticCCKernel.join();
            }
            endGpuProcessing = std::chrono::steady_clock::now();
            durationGpuProcessing += std::chrono::duration_cast<std::chrono::milliseconds>(
                    endGpuProcessing - startGpuProcessing).count();
        }
        mixCommonLabel<<<grid, block, 0, streamDynamic>>>(testNumNodes, isActiveD1, isActiveD2);
        //hipDeviceSynchronize();
        //cout << "mixDynamicPartLabel" << " =========hipDeviceSynchronize()==========" << endl;
        //hipMemcpy(label, isActiveD, testNumNodes * sizeof(uint), hipMemcpyDeviceToHost);
        startPreGpuProcessing = std::chrono::steady_clock::now();
        activeNodesNum = thrust::reduce(ptr_labeling, ptr_labeling + testNumNodes);
        nodeSum += activeNodesNum;
        endPreGpuProcessing = std::chrono::steady_clock::now();
        durationPreGpuProcessing += std::chrono::duration_cast<std::chrono::milliseconds>(
                endPreGpuProcessing - startPreGpuProcessing).count();
    }
    hipDeviceSynchronize();
    hipMemcpy(value, valueD, testNumNodes * sizeof(uint), hipMemcpyDeviceToHost);
    hipMemcpy(vertexVisitRecord, vertexVisitRecordD, testNumNodes * sizeof(uint), hipMemcpyDeviceToHost);
    uint partNum = 50;
    uint partSize = testNumEdge / partNum;
    vector<uint> partVistRecordList(partNum + 1);
    uint partSizeCursor = 0;
    for (uint i = 0; i < testNumNodes; i++) {
        uint edgeStartIndex = nodePointersI[i];
        uint edgeEndIndex = nodePointersI[i] + degree[i];
        uint maxPartIndex = partSizeCursor * partSize + partSize;

        if (edgeStartIndex < maxPartIndex && edgeEndIndex < maxPartIndex) {
            partVistRecordList[partSizeCursor] += vertexVisitRecord[i] * degree[i];
        } else if (edgeStartIndex < maxPartIndex && edgeEndIndex >= maxPartIndex) {
            partVistRecordList[partSizeCursor] += vertexVisitRecord[i] * (maxPartIndex - edgeStartIndex);
            partSizeCursor += 1;
            partVistRecordList[partSizeCursor] += vertexVisitRecord[i] * (edgeEndIndex - maxPartIndex);
        } else {
            partSizeCursor += 1;
            partVistRecordList[partSizeCursor] += vertexVisitRecord[i] * degree[i];
        }
    }
    for (uint i = 0; i < partNum + 1; i++) {
        cout << "part " << i << " is " << partVistRecordList[i] << endl;
    }
    for (uint i = 0; i < partNum + 1; i++) {
        cout << partVistRecordList[i] << "\t";
    }
    transferSum += max_partition_size;
    cout << "transferSum: " << transferSum * 4 << "byte" << endl;
    cout << "iterationSum " << iter << endl;
    double edgeIterationAvg = (double) overloadEdgeSum / (double) testNumEdge / iter;
    double edgeIterationMaxAvg = (double) edgeIterationMax / (double) testNumEdge;
    cout << "edgeIterationAvg " << edgeIterationAvg << " edgeIterationMaxAvg " << edgeIterationMaxAvg << endl;
    cout << "nodeSum: " << nodeSum << endl;
    auto endRead = std::chrono::steady_clock::now();
    durationRead = std::chrono::duration_cast<std::chrono::milliseconds>(endRead - startProcessing).count();
    cout << "finish time : " << durationRead << " ms" << endl;
    cout << "total time : " << durationRead + testDuration << " ms" << endl;
    cout << "cpu time : " << durationReadCpu << " ms" << endl;
    cout << "pre fact processing time : " << durationGpuProcessing << " ms" << endl;
    cout << "overload fact processing time : " << durationOverloadGpuProcessing << " ms" << endl;
    cout << "durationMemoryTraverse : " << durationMemoryTraverse << " ms" << endl;
    cout << "durationOverloadGpuProcessing : " << durationOverloadGpuProcessing << " ms" << endl;

    cout << "gpu pre processing time : " << durationPreGpuProcessing << " ms" << endl;
    cout << "swap processing time : " << durationSwap << " ms" << endl;
    cout << "overloadEdgeSum : " << overloadEdgeSum << " " << endl;

    cout << "swapValidNodeSum " << swapValidNodeSum << " swapValidEdgeSum " << swapValidEdgeSum << endl;
    cout << "swapNotValidNodeSum " << swapNotValidNodeSum << " swapNotValidEdgeSum " << swapNotValidEdgeSum
         << " visitSum " << visitEdgeSum << " swapInEdgeSum " << swapInEdgeSum << endl;

    cout << "headSum " << headSum << " tailSum " << tailSum << endl;
    /*hipFree(nodePointerD);
    hipFree(staticEdgeListD);
    hipFree(degreeD);
    hipFree(isActiveD1);
    hipFree(isActiveD2);
    hipFree(valueD);
    hipFree(activeNodeListD);
    hipFree(activeNodeLabelingPrefixD);
    hipFree(activeOverloadNodePointersD);
    hipFree(activeOverloadDegreeD);
    hipFree(isInStaticD);
    hipFree(staticNodePointerD);
    hipFree(overloadNodeListD);

    delete[]            label;
    delete[]            degree;
    delete[]            value;
    delete[]            staticActiveNodeList;
    delete[]            activeOverloadNodePointers;
    delete[] isInStatic;
    delete[] overloadNodeList;
    delete[] staticNodePointer;
    delete[] fragmentData;
    return durationRead;*/
}


void ccShareTrace(string ccPath) {
    uint testNumNodes = 0;
    ulong testNumEdge = 0;
    uint *nodePointersI;
    uint *edgeList;

    auto startReadGraph = std::chrono::steady_clock::now();
    ifstream infile(ccPath, ios::in | ios::binary);
    infile.read((char *) &testNumNodes, sizeof(uint));
    uint numEdge = 0;
    infile.read((char *) &numEdge, sizeof(uint));
    testNumEdge = numEdge;
    cout << "vertex num: " << testNumNodes << " edge num: " << testNumEdge << endl;
    nodePointersI = new uint[testNumNodes];
    infile.read((char *) nodePointersI, sizeof(uint) * testNumNodes);
    gpuErrorcheck(hipMallocManaged(&edgeList, (numEdge) * sizeof(uint)));
    hipMemAdvise(nodePointersI, (testNumNodes + 1) * sizeof(uint), hipMemAdviseSetReadMostly, 0);
    hipMemAdvise(edgeList, (numEdge) * sizeof(uint), hipMemAdviseSetReadMostly, 0);
    infile.read((char *) edgeList, sizeof(uint) * testNumEdge);
    infile.close();
    auto endReadGraph = std::chrono::steady_clock::now();
    long durationReadGraph = std::chrono::duration_cast<std::chrono::milliseconds>(
            endReadGraph - startReadGraph).count();
    cout << "read graph time : " << durationReadGraph << "ms" << endl;
    int testTimes = 1;
    long timeSum = 0;
    for (int i = 0; i < testTimes; i++) {
        timeSum += ccCaculateInShareTrace(testNumNodes, testNumEdge, nodePointersI, edgeList);
        break;
    }
    cout << "need cpu " << needCpu << " not need cpu " << notNeedCpu << endl;
    cout << "processingTime " << processingTimeSum / testTimes << " cpu time " << cpuTimeSum / testTimes << " all Time "
         << allTimeSum / testTimes << endl;
    cout << "mean time is " << timeSum / testTimes << endl;
    cout << "mean validSwapSum is " << validSwapSum / testTimes << endl;
    cout << trestSum << endl;
}


long ccCaculateInShareTrace(uint testNumNodes, uint testNumEdge, uint *nodePointersI, uint *edgeList) {
    auto start = std::chrono::steady_clock::now();
    uint *degree = new uint[testNumNodes];
    uint *value = new uint[testNumNodes];
    uint sourceCode = 0;

    auto startPreCaculate = std::chrono::steady_clock::now();
    for (uint i = 0; i < testNumNodes - 1; i++) {
        degree[i] = nodePointersI[i + 1] - nodePointersI[i];
    }

    degree[testNumNodes - 1] = testNumEdge - nodePointersI[testNumNodes - 1];
    bool *label = new bool[testNumNodes];
    for (uint i = 0; i < testNumNodes; i++) {
        label[i] = true;
        value[i] = i;
    }

    label[sourceCode] = true;
    value[sourceCode] = 1;
    uint *activeNodeListD;
    uint *degreeD;
    uint *valueD;
    bool *labelD;
    uint *nodePointersD;
    hipMalloc(&activeNodeListD, testNumNodes * sizeof(uint));
    hipMalloc(&nodePointersD, testNumNodes * sizeof(uint));
    hipMalloc(&degreeD, testNumNodes * sizeof(uint));
    hipMalloc(&valueD, testNumNodes * sizeof(uint));
    hipMalloc(&labelD, testNumNodes * sizeof(bool));
    hipMemcpy(degreeD, degree, testNumNodes * sizeof(uint), hipMemcpyHostToDevice);
    hipMemcpy(valueD, value, testNumNodes * sizeof(uint), hipMemcpyHostToDevice);
    hipMemcpy(labelD, label, testNumNodes * sizeof(bool), hipMemcpyHostToDevice);
    hipMemcpy(nodePointersD, nodePointersI, testNumNodes * sizeof(uint), hipMemcpyHostToDevice);
    //cacaulate the active node And make active node array
    uint *activeNodeLabelingD;
    gpuErrorcheck(hipMalloc(&activeNodeLabelingD, testNumNodes * sizeof(unsigned int)));
    uint *activeNodeLabelingPrefixD;
    gpuErrorcheck(hipMalloc(&activeNodeLabelingPrefixD, testNumNodes * sizeof(unsigned int)));
    dim3 grid = dim3(56, 1, 1);
    dim3 block = dim3(1024, 1, 1);

    auto endPreCaculate = std::chrono::steady_clock::now();
    long durationPreCaculate = std::chrono::duration_cast<std::chrono::milliseconds>(
            endPreCaculate - startPreCaculate).count();
    cout << "durationPreCaculate time : " << durationPreCaculate << " ms" << endl;

    setLabeling<<<grid, block>>>(testNumNodes, labelD, activeNodeLabelingD);
    thrust::device_ptr<unsigned int> ptr_labeling(activeNodeLabelingD);
    thrust::device_ptr<unsigned int> ptr_labeling_prefixsum(activeNodeLabelingPrefixD);
    cout << "before reduce" << endl;
    uint activeNodesNum = thrust::reduce(ptr_labeling, ptr_labeling + testNumNodes);
    cout << "after reduce" << endl;
    int iter = 0;
    uint nodeSum = activeNodesNum;

    cout << "durationPreCaculate time : " << durationPreCaculate << " ms" << endl;
    auto startProcessing = std::chrono::steady_clock::now();
    while (activeNodesNum > 0) {
        iter++;
        thrust::exclusive_scan(ptr_labeling, ptr_labeling + testNumNodes, ptr_labeling_prefixsum);
        setActiveNodeArray<<<grid, block>>>(testNumNodes, activeNodeListD, labelD, activeNodeLabelingPrefixD);
        setLabelDefault<<<grid, block>>>(activeNodesNum, activeNodeListD, labelD);
        cc_kernel<<<grid, block>>>(activeNodesNum, activeNodeListD, nodePointersD, degreeD, edgeList, valueD, labelD);
        hipDeviceSynchronize();
        gpuErrorcheck(hipPeekAtLastError());
        for (uint j = 0; j < testNumEdge; j++) {
            uint temp = edgeList[j];
            if (temp >= 0) {
                uint a = temp + 1;
            }

        }
        setLabeling<<<grid, block>>>(testNumNodes, labelD, activeNodeLabelingD);
        activeNodesNum = thrust::reduce(ptr_labeling, ptr_labeling + testNumNodes);
        nodeSum += activeNodesNum;
        //cout << "iter: " << iter << " activeNodes: " << activeNodesNum << endl;
    }
    hipDeviceSynchronize();

    cout << "nodeSum: " << nodeSum << endl;

    auto endRead = std::chrono::steady_clock::now();
    long durationRead = std::chrono::duration_cast<std::chrono::milliseconds>(endRead - startProcessing).count();
    cout << "iter sum is " << iter << " finish time : " << durationRead << " ms" << endl;

    return durationRead;
}

long ccCaculateCommonMemoryInnerAsyncRandom(uint testNumNodes, uint testNumEdge, uint *nodePointersI, uint *edgeList,
                                            float adviseK) {
    cout << "=========ccCaculateCommonMemoryInnerAsync1========" << endl;
    ulong edgeIterationMax = 0;
    auto start = std::chrono::steady_clock::now();
    auto startPreCaculate = std::chrono::steady_clock::now();
    //CPU
    long durationRead;
    ulong transferSum = 0;
    unsigned long max_partition_size;
    unsigned long total_gpu_size;
    uint maxStaticNode = 0;
    uint *degree;
    uint *value;
    uint *label;
    bool *isInStatic;
    uint *overloadNodeList;
    uint *staticNodePointer;
    uint *activeNodeList;
    uint *activeOverloadNodePointers;
    vector<PartEdgeListInfo> partEdgeListInfoArr;
    /*
     * overloadEdgeList overload edge list in every iteration
     * */
    uint *overloadEdgeList;
    FragmentData *fragmentData;
    bool isFromTail = true;
    //GPU
    uint *staticEdgeListD;
    uint *overloadEdgeListD;
    bool *isInStaticD;
    uint *overloadNodeListD;
    uint *staticNodePointerD;
    uint *nodePointerD;
    uint *degreeD;
    // async need two labels
    uint *isActiveD1;
    uint *isActiveD2;
    uint *isStaticActive;
    uint *isOverloadActive;
    uint *valueD;
    uint *activeNodeListD;
    uint *activeNodeLabelingPrefixD;
    uint *activeOverloadNodePointersD;
    uint *activeOverloadDegreeD;
    bool *isFinishedDevice;

    degree = new uint[testNumNodes];
    value = new uint[testNumNodes];
    label = new uint[testNumNodes];
    isInStatic = new bool[testNumNodes];
    overloadNodeList = new uint[testNumNodes];
    staticNodePointer = new uint[testNumNodes];
    activeNodeList = new uint[testNumNodes];
    activeOverloadNodePointers = new uint[testNumNodes];

    getMaxPartitionSize(max_partition_size, total_gpu_size, testNumNodes, adviseK, sizeof(uint), testNumEdge, 15);
    gpuErrorcheck(hipMalloc(&isFinishedDevice, 1 * sizeof(bool)));
    //caculate degree
    calculateDegree(testNumNodes, nodePointersI, testNumEdge, degree);
    //memcpy(staticNodePointer, nodePointersI, testNumNodes * sizeof(uint));
    uint edgesInStatic = 0;
    float startRate = (1 - (float) max_partition_size / (float) testNumEdge) / 2;
    uint startIndex = (float) testNumNodes * startRate;
    uint tempStaticSum = 0;
    /*for (uint i = testNumNodes - 1; i >= 0; i--) {
        tempStaticSum += degree[i];
        if (tempStaticSum > max_partition_size) {
            startIndex = i;
            break;
        }
    }*/
    //startIndex = 0;
    if (nodePointersI[startIndex] + max_partition_size > testNumEdge) {
        startIndex = (float) testNumNodes * 0.1f;
    }
    for (uint i = 0; i < testNumNodes; i++) {
        label[i] = 1;
        value[i] = i;
        if (i >= startIndex && nodePointersI[i] < nodePointersI[startIndex] + max_partition_size - degree[i]) {
            isInStatic[i] = true;
            staticNodePointer[i] = nodePointersI[i] - nodePointersI[startIndex];
            if (i > maxStaticNode) {
                maxStaticNode = i;
            }
            edgesInStatic += degree[i];
        } else {
            isInStatic[i] = false;
        }
    }

    gpuErrorcheck(hipMalloc(&staticEdgeListD, max_partition_size * sizeof(uint)));
    auto startmove = std::chrono::steady_clock::now();
    gpuErrorcheck(
            hipMemcpy(staticEdgeListD, edgeList + nodePointersI[startIndex], max_partition_size * sizeof(uint),
                       hipMemcpyHostToDevice));
    auto endMove = std::chrono::steady_clock::now();
    long testDuration = std::chrono::duration_cast<std::chrono::milliseconds>(
            endMove - startmove).count();
    cout << "move duration " << testDuration << endl;

    gpuErrorcheck(hipMalloc(&isInStaticD, testNumNodes * sizeof(bool)))
    gpuErrorcheck(hipMalloc(&overloadNodeListD, testNumNodes * sizeof(uint)));
    gpuErrorcheck(hipMalloc(&staticNodePointerD, testNumNodes * sizeof(uint)))
    gpuErrorcheck(
            hipMemcpy(staticNodePointerD, staticNodePointer, testNumNodes * sizeof(uint), hipMemcpyHostToDevice));
    hipMemcpy(isInStaticD, isInStatic, testNumNodes * sizeof(bool), hipMemcpyHostToDevice);

    uint partOverloadSize = total_gpu_size - max_partition_size;
    uint overloadSize = testNumEdge - edgesInStatic;
    cout << " partOverloadSize " << partOverloadSize << " overloadSize " << overloadSize << endl;
    overloadEdgeList = (uint *) malloc(overloadSize * sizeof(uint));
    gpuErrorcheck(hipMalloc(&overloadEdgeListD, partOverloadSize * sizeof(uint)));
    gpuErrorcheck(hipMalloc(&degreeD, testNumNodes * sizeof(uint)));
    gpuErrorcheck(hipMalloc(&isActiveD1, testNumNodes * sizeof(uint)));
    gpuErrorcheck(hipMalloc(&isActiveD2, testNumNodes * sizeof(uint)));
    gpuErrorcheck(hipMalloc(&isStaticActive, testNumNodes * sizeof(uint)));
    gpuErrorcheck(hipMalloc(&isOverloadActive, testNumNodes * sizeof(uint)));
    gpuErrorcheck(hipMalloc(&valueD, testNumNodes * sizeof(uint)));
    gpuErrorcheck(hipMalloc(&activeNodeLabelingPrefixD, testNumNodes * sizeof(unsigned int)));
    gpuErrorcheck(hipMalloc(&activeNodeListD, testNumNodes * sizeof(uint)));
    gpuErrorcheck(hipMalloc(&activeOverloadNodePointersD, testNumNodes * sizeof(uint)));
    gpuErrorcheck(hipMalloc(&activeOverloadDegreeD, testNumNodes * sizeof(uint)));
    gpuErrorcheck(hipMemcpy(degreeD, degree, testNumNodes * sizeof(uint), hipMemcpyHostToDevice));
    gpuErrorcheck(hipMemcpy(valueD, value, testNumNodes * sizeof(uint), hipMemcpyHostToDevice));
    gpuErrorcheck(hipMemcpy(isActiveD1, label, testNumNodes * sizeof(uint), hipMemcpyHostToDevice));
    gpuErrorcheck(hipMemset(isActiveD2, 0, testNumNodes * sizeof(uint)));
    gpuErrorcheck(hipMemset(isStaticActive, 0, testNumNodes * sizeof(uint)));
    gpuErrorcheck(hipMemset(isOverloadActive, 0, testNumNodes * sizeof(uint)));

    //cacaulate the active node And make active node array
    dim3 grid = dim3(56, 1, 1);
    dim3 block = dim3(1024, 1, 1);

    //setLabeling<<<grid, block>>>(testNumNodes, labelD, activeNodeLabelingD);
    thrust::device_ptr<unsigned int> ptr_labeling(isActiveD1);
    thrust::device_ptr<unsigned int> ptr_labelingTest(isActiveD2);
    thrust::device_ptr<unsigned int> ptr_labeling_static(isStaticActive);
    thrust::device_ptr<unsigned int> ptr_labeling_overload(isOverloadActive);
    thrust::device_ptr<unsigned int> ptr_labeling_prefixsum(activeNodeLabelingPrefixD);
    thrust::device_ptr<unsigned int> ptrOverloadDegree(activeOverloadDegreeD);
    thrust::device_ptr<unsigned int> ptrOverloadPrefixsum(activeOverloadNodePointersD);

    uint activeNodesNum = thrust::reduce(ptr_labeling, ptr_labeling + testNumNodes);
    int iter = 0;
    uint nodeSum = activeNodesNum;
    ulong overloadEdgeSum = 0;
    auto startCpu = std::chrono::steady_clock::now();
    auto endReadCpu = std::chrono::steady_clock::now();
    long durationReadCpu = 0;

    auto startSwap = std::chrono::steady_clock::now();
    auto endSwap = std::chrono::steady_clock::now();
    long durationSwap = 0;

    auto startGpuProcessing = std::chrono::steady_clock::now();
    auto endGpuProcessing = std::chrono::steady_clock::now();
    long durationGpuProcessing = 0;

    auto startOverloadGpuProcessing = std::chrono::steady_clock::now();
    auto endOverloadGpuProcessing = std::chrono::steady_clock::now();
    long durationOverloadGpuProcessing = 0;

    auto startPreGpuProcessing = std::chrono::steady_clock::now();
    auto endPreGpuProcessing = std::chrono::steady_clock::now();
    long durationPreGpuProcessing = 0;
    auto endPreCaculate = std::chrono::steady_clock::now();
    long durationPreCaculate = std::chrono::duration_cast<std::chrono::milliseconds>(
            endPreCaculate - startPreCaculate).count();
    cout << "durationPreCaculate time : " << durationPreCaculate << " ms" << endl;
    hipStream_t steamStatic, streamDynamic;
    hipStreamCreate(&steamStatic);
    hipStreamCreate(&streamDynamic);
    auto startMemoryTraverse = std::chrono::steady_clock::now();
    auto endMemoryTraverse = std::chrono::steady_clock::now();
    long durationMemoryTraverse = 0;
    //uint cursorStartSwap = staticFragmentNum + 1;
    uint swapValidNodeSum = 0;
    uint swapValidEdgeSum = 0;
    uint swapNotValidNodeSum = 0;
    uint swapNotValidEdgeSum = 0;
    uint visitEdgeSum = 0;
    uint swapInEdgeSum = 0;
    uint headSum;
    uint tailSum;

    long TIME = 0;
    int testTimes = 10;
    for (int testIndex = 0; testIndex < testTimes; testIndex++) {

        for (uint i = 0; i < testNumNodes; i++) {
            label[i] = 1;
            value[i] = i;
        }
        hipMemcpy(isInStaticD, isInStatic, testNumNodes * sizeof(bool), hipMemcpyHostToDevice);
        gpuErrorcheck(hipMemcpy(valueD, value, testNumNodes * sizeof(uint), hipMemcpyHostToDevice));
        gpuErrorcheck(hipMemcpy(isActiveD1, label, testNumNodes * sizeof(uint), hipMemcpyHostToDevice));
        gpuErrorcheck(hipMemset(isActiveD2, 0, testNumNodes * sizeof(uint)));
        gpuErrorcheck(hipMemset(isStaticActive, 0, testNumNodes * sizeof(uint)));
        gpuErrorcheck(hipMemset(isOverloadActive, 0, testNumNodes * sizeof(uint)));
        activeNodesNum = thrust::reduce(ptr_labeling, ptr_labeling + testNumNodes);
        iter = 0;

        auto startProcessing = std::chrono::steady_clock::now();
        auto startTest = std::chrono::steady_clock::now();
        auto endTest = std::chrono::steady_clock::now();
        long durationTest = 0;
        while (activeNodesNum > 0) {
            iter++;
            //cout << "iter " << iter << " activeNodesNum " << activeNodesNum << endl;
            startPreGpuProcessing = std::chrono::steady_clock::now();
            //cleanStaticAndOverloadLabel<<<grid, block>>>(testNumNodes, isStaticActive, isOverloadActive);
            setStaticAndOverloadLabel<<<grid, block>>>(testNumNodes, isActiveD1, isStaticActive, isOverloadActive,
                                                       isInStaticD);
            uint staticNodeNum = thrust::reduce(ptr_labeling_static, ptr_labeling_static + testNumNodes);
            if (staticNodeNum > 0) {
                //cout << "iter " << iter << " staticNodeNum " << staticNodeNum << endl;
                thrust::exclusive_scan(ptr_labeling_static, ptr_labeling_static + testNumNodes, ptr_labeling_prefixsum);
                setStaticActiveNodeArray<<<grid, block>>>(testNumNodes, activeNodeListD, isStaticActive,
                                                          activeNodeLabelingPrefixD);
            }
            uint overloadNodeNum = thrust::reduce(ptr_labeling_overload, ptr_labeling_overload + testNumNodes);
            uint overloadEdgeNum = 0;
            if (overloadNodeNum > 0) {
                //cout << "iter " << iter << " overloadNodeNum " << overloadNodeNum << endl;

                thrust::exclusive_scan(ptr_labeling_overload, ptr_labeling_overload + testNumNodes,
                                       ptr_labeling_prefixsum);
                setOverloadNodePointerSwap<<<grid, block>>>(testNumNodes, overloadNodeListD, activeOverloadDegreeD,
                                                            isOverloadActive,
                                                            activeNodeLabelingPrefixD, degreeD);

                thrust::exclusive_scan(ptrOverloadDegree, ptrOverloadDegree + overloadNodeNum,
                                       activeOverloadNodePointersD);
                overloadEdgeNum = thrust::reduce(thrust::device, ptrOverloadDegree,
                                                 ptrOverloadDegree + overloadNodeNum, 0);
                //cout << "iter " << iter << " overloadEdgeNum " << overloadEdgeNum << endl;
                overloadEdgeSum += overloadEdgeNum;
                if (overloadEdgeNum > edgeIterationMax) {
                    edgeIterationMax = overloadEdgeNum;
                }
            }
            endPreGpuProcessing = std::chrono::steady_clock::now();
            durationPreGpuProcessing += std::chrono::duration_cast<std::chrono::milliseconds>(
                    endPreGpuProcessing - startPreGpuProcessing).count();
            startGpuProcessing = std::chrono::steady_clock::now();
            mixDynamicPartLabel<<<grid, block, 0, steamStatic>>>(staticNodeNum, 0, activeNodeListD, isActiveD1,
                                                                 isActiveD2);
            thread staticCCKernel = thread(ccKernelThread, staticNodeNum, activeNodeListD, staticNodePointerD, degreeD,
                                           staticEdgeListD, valueD, isActiveD1, isActiveD2, isFinishedDevice, grid,
                                           block,
                                           steamStatic);
            /*if (staticCCKernel.joinable()) {
                staticCCKernel.join();
            }*/

            if (overloadNodeNum > 0) {
                startCpu = std::chrono::steady_clock::now();
                /*hipMemcpyAsync(staticActiveNodeList, activeNodeListD, activeNodesNum * sizeof(uint), hipMemcpyDeviceToHost,
                                streamDynamic);*/
                hipMemcpyAsync(overloadNodeList, overloadNodeListD, overloadNodeNum * sizeof(uint),
                                hipMemcpyDeviceToHost,
                                streamDynamic);
                hipMemcpyAsync(activeOverloadNodePointers, activeOverloadNodePointersD, overloadNodeNum * sizeof(uint),
                                hipMemcpyDeviceToHost, streamDynamic);

                int threadNum = 20;
                if (overloadNodeNum < 50) {
                    threadNum = 1;
                }
                thread runThreads[threadNum];
                for (int i = 0; i < threadNum; i++) {
                    runThreads[i] = thread(fillDynamic,
                                           i,
                                           threadNum,
                                           0,
                                           overloadNodeNum,
                                           degree,
                                           activeOverloadNodePointers,
                                           nodePointersI,
                                           overloadNodeList,
                                           overloadEdgeList,
                                           edgeList);
                }

                for (unsigned int t = 0; t < threadNum; t++) {
                    runThreads[t].join();
                }
                caculatePartInfoForEdgeList(activeOverloadNodePointers, overloadNodeList, degree, partEdgeListInfoArr,
                                            overloadNodeNum, partOverloadSize, overloadEdgeNum);

                endReadCpu = std::chrono::steady_clock::now();
                durationReadCpu += std::chrono::duration_cast<std::chrono::milliseconds>(endReadCpu - startCpu).count();
                if (staticCCKernel.joinable()) {
                    staticCCKernel.join();
                }
                endGpuProcessing = std::chrono::steady_clock::now();
                durationGpuProcessing += std::chrono::duration_cast<std::chrono::milliseconds>(
                        endGpuProcessing - startGpuProcessing).count();

                for (auto &i : partEdgeListInfoArr) {
                    startMemoryTraverse = std::chrono::steady_clock::now();
                    gpuErrorcheck(hipMemcpy(overloadEdgeListD, overloadEdgeList +
                                                                activeOverloadNodePointers[i.partStartIndex],
                                             i.partEdgeNums * sizeof(uint), hipMemcpyHostToDevice))
                    transferSum += i.partEdgeNums;
                    endMemoryTraverse = std::chrono::steady_clock::now();
                    durationMemoryTraverse += std::chrono::duration_cast<std::chrono::milliseconds>(
                            endMemoryTraverse - startMemoryTraverse).count();
                    /*cout << "iter " << iter << " part " << i << " durationMemoryTraverse "
                         << durationMemoryTraverse << endl;*/
                    startOverloadGpuProcessing = std::chrono::steady_clock::now();
                    mixDynamicPartLabel<<<grid, block, 0, streamDynamic>>>(i.partActiveNodeNums,
                                                                           i.partStartIndex,
                                                                           overloadNodeListD, isActiveD1,
                                                                           isActiveD2);
                    uint itr = 0;
                    bool isFinishedHost = true;
                    do {
                        itr++;
                        isFinishedHost = true;
                        hipMemcpy(isFinishedDevice, &isFinishedHost, sizeof(bool), hipMemcpyHostToDevice);

                        cc_kernelDynamicSwap2Label<<<grid, block, 0, streamDynamic>>>(i.partStartIndex,
                                                                                      i.partActiveNodeNums,
                                                                                      overloadNodeListD, degreeD,
                                                                                      valueD, itr % 2 == 1 ? isActiveD1
                                                                                                           : isActiveD2,
                                                                                      itr % 2 == 1 ? isActiveD2
                                                                                                   : isActiveD1,
                                                                                      overloadEdgeListD,
                                                                                      activeOverloadNodePointersD,
                                                                                      isFinishedDevice);
                        hipDeviceSynchronize();
                        hipMemcpy(&isFinishedHost, isFinishedDevice, sizeof(bool), hipMemcpyDeviceToHost);
                        isFinishedHost = true;
                    } while (!isFinishedHost);
                    endOverloadGpuProcessing = std::chrono::steady_clock::now();
                    durationOverloadGpuProcessing += std::chrono::duration_cast<std::chrono::milliseconds>(
                            endOverloadGpuProcessing - startOverloadGpuProcessing).count();
                    /*cout << "iter " << iter << " part " << i << " durationOverloadGpuProcessing "
                         << durationOverloadGpuProcessing << endl;*/
                }
                gpuErrorcheck(hipPeekAtLastError())

            } else {
                if (staticCCKernel.joinable()) {
                    staticCCKernel.join();
                }
                endGpuProcessing = std::chrono::steady_clock::now();
                durationGpuProcessing += std::chrono::duration_cast<std::chrono::milliseconds>(
                        endGpuProcessing - startGpuProcessing).count();
            }
            mixCommonLabel<<<grid, block, 0, streamDynamic>>>(testNumNodes, isActiveD1, isActiveD2);
            //hipDeviceSynchronize();
            //cout << "mixDynamicPartLabel" << " =========hipDeviceSynchronize()==========" << endl;
            //hipMemcpy(label, isActiveD, testNumNodes * sizeof(uint), hipMemcpyDeviceToHost);
            startPreGpuProcessing = std::chrono::steady_clock::now();
            activeNodesNum = thrust::reduce(ptr_labeling, ptr_labeling + testNumNodes);
            nodeSum += activeNodesNum;
            endPreGpuProcessing = std::chrono::steady_clock::now();
            durationPreGpuProcessing += std::chrono::duration_cast<std::chrono::milliseconds>(
                    endPreGpuProcessing - startPreGpuProcessing).count();
        }
        hipDeviceSynchronize();
        hipMemcpy(value, valueD, testNumNodes * sizeof(uint), hipMemcpyDeviceToHost);
        transferSum += max_partition_size;
        cout << "transferSum: " << transferSum * 4 << "byte" << endl;
        cout << "iterationSum " << iter << endl;
        double edgeIterationAvg = (double) overloadEdgeSum / (double) testNumEdge / iter;
        double edgeIterationMaxAvg = (double) edgeIterationMax / (double) testNumEdge;
        cout << "edgeIterationAvg " << edgeIterationAvg << " edgeIterationMaxAvg " << edgeIterationMaxAvg << endl;
        cout << "nodeSum: " << nodeSum << endl;
        auto endRead = std::chrono::steady_clock::now();
        durationRead = std::chrono::duration_cast<std::chrono::milliseconds>(endRead - startProcessing).count();
        cout << "finish time : " << durationRead << " ms" << endl;
        cout << "total time : " << durationRead + testDuration << " ms" << endl;
        cout << "cpu time : " << durationReadCpu << " ms" << endl;
        cout << "pre fact processing time : " << durationGpuProcessing << " ms" << endl;
        cout << "overload fact processing time : " << durationOverloadGpuProcessing << " ms" << endl;
        cout << "durationMemoryTraverse : " << durationMemoryTraverse << " ms" << endl;
        cout << "durationOverloadGpuProcessing : " << durationOverloadGpuProcessing << " ms" << endl;

        cout << "gpu pre processing time : " << durationPreGpuProcessing << " ms" << endl;
        cout << "swap processing time : " << durationSwap << " ms" << endl;
        cout << "overloadEdgeSum : " << overloadEdgeSum << " " << endl;

        cout << "swapValidNodeSum " << swapValidNodeSum << " swapValidEdgeSum " << swapValidEdgeSum << endl;
        cout << "swapNotValidNodeSum " << swapNotValidNodeSum << " swapNotValidEdgeSum " << swapNotValidEdgeSum
             << " visitSum " << visitEdgeSum << " swapInEdgeSum " << swapInEdgeSum << endl;

        cout << "headSum " << headSum << " tailSum " << tailSum << endl;
        TIME += durationRead;
    }
    cout << "TIME " << (float) TIME / (float) testTimes << endl;
    /*hipFree(nodePointerD);
    hipFree(staticEdgeListD);
    hipFree(degreeD);
    hipFree(isActiveD1);
    hipFree(isActiveD2);
    hipFree(valueD);
    hipFree(activeNodeListD);
    hipFree(activeNodeLabelingPrefixD);
    hipFree(activeOverloadNodePointersD);
    hipFree(activeOverloadDegreeD);
    hipFree(isInStaticD);
    hipFree(staticNodePointerD);
    hipFree(overloadNodeListD);

    delete[]            label;
    delete[]            degree;
    delete[]            value;
    delete[]            staticActiveNodeList;
    delete[]            activeOverloadNodePointers;
    delete[] isInStatic;
    delete[] overloadNodeList;
    delete[] staticNodePointer;
    delete[] fragmentData;
    return durationRead;*/
}