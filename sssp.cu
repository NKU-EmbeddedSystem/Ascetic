#include "hip/hip_runtime.h"
//
// Created by gxl on 2021/1/6.
//
#include "sssp.cuh"

void conventionParticipateSSSP(uint sourceNodeSample, string ssspPath) {
    cout << "===============conventionParticipateSSSP==============" << endl;
    uint testNumNodes = 0;
    ulong testNumEdge = 0;
    ulong traverseSum = 0;
    uint *nodePointersI;
    EdgeWithWeight *edgeList;
    auto startReadGraph = std::chrono::steady_clock::now();
    ifstream infile(ssspPath, ios::in | ios::binary);
    infile.read((char *) &testNumNodes, sizeof(uint));
    uint numEdge = 0;
    infile.read((char *) &numEdge, sizeof(uint));
    testNumEdge = numEdge;
    cout << "vertex num: " << testNumNodes << " edge num: " << testNumEdge << endl;
    nodePointersI = new uint[testNumNodes];

    infile.read((char *) nodePointersI, sizeof(uint) * testNumNodes);
    edgeList = new EdgeWithWeight[testNumEdge];
    infile.read((char *) edgeList, sizeof(EdgeWithWeight) * testNumEdge);
    infile.close();
    unsigned long max_partition_size;
    unsigned long total_gpu_size;
    getMaxPartitionSize(max_partition_size, total_gpu_size, testNumNodes, 0.9, sizeof(EdgeWithWeight), 5);
    uint partitionNum;
    if (testNumEdge > max_partition_size) {
        partitionNum = testNumEdge / max_partition_size + 1;
    } else {
        partitionNum = 1;
    }

    uint *degree = new uint[testNumNodes];
    uint *value = new uint[testNumNodes];
    bool *isActiveNodeList = new bool[testNumNodes];
    CommonPartitionInfo *partitionInfoList = new CommonPartitionInfo[partitionNum];
    bool *needTransferPartition = new bool[partitionNum];
    for (uint i = 0; i < testNumNodes; i++) {
        isActiveNodeList[i] = false;
        value[i] = UINT_MAX;
        if (i + 1 < testNumNodes) {
            degree[i] = nodePointersI[i + 1] - nodePointersI[i];
        } else {
            degree[i] = testNumEdge - nodePointersI[i];
        }
        if (degree[i] > max_partition_size) {
            cout << "node " << i << " degree > maxPartition " << endl;
            return;
        }
    }
    for (uint i = 0; i < partitionNum; i++) {
        partitionInfoList[i].startVertex = -1;
        partitionInfoList[i].endVertex = -1;
        partitionInfoList[i].nodePointerOffset = -1;
        partitionInfoList[i].partitionEdgeSize = -1;
    }
    int tempPartitionIndex = 0;
    uint tempNodeIndex = 0;
    while (tempNodeIndex < testNumNodes) {
        if (partitionInfoList[tempPartitionIndex].startVertex == -1) {
            partitionInfoList[tempPartitionIndex].startVertex = tempNodeIndex;
            partitionInfoList[tempPartitionIndex].endVertex = tempNodeIndex;
            partitionInfoList[tempPartitionIndex].nodePointerOffset = nodePointersI[tempNodeIndex];
            partitionInfoList[tempPartitionIndex].partitionEdgeSize = degree[tempNodeIndex];
            tempNodeIndex++;
        } else {
            if (partitionInfoList[tempPartitionIndex].partitionEdgeSize + degree[tempNodeIndex] > max_partition_size) {
                tempPartitionIndex++;
            } else {
                partitionInfoList[tempPartitionIndex].endVertex = tempNodeIndex;
                partitionInfoList[tempPartitionIndex].partitionEdgeSize += degree[tempNodeIndex];
                tempNodeIndex++;
            }
        }
    }

    uint *degreeD;
    bool *isActiveNodeListD;
    bool *nextActiveNodeListD;
    uint *nodePointerListD;
    EdgeWithWeight *partitionEdgeListD;
    uint *valueD;

    hipMalloc(&degreeD, testNumNodes * sizeof(uint));
    hipMalloc(&valueD, testNumNodes * sizeof(uint));
    hipMalloc(&isActiveNodeListD, testNumNodes * sizeof(bool));
    hipMalloc(&nextActiveNodeListD, testNumNodes * sizeof(bool));
    hipMalloc(&nodePointerListD, testNumNodes * sizeof(uint));
    hipMalloc(&partitionEdgeListD, max_partition_size * sizeof(EdgeWithWeight));

    hipMemcpy(degreeD, degree, testNumNodes * sizeof(uint), hipMemcpyHostToDevice);
    hipMemcpy(nodePointerListD, nodePointersI, testNumNodes * sizeof(uint), hipMemcpyHostToDevice);
    hipMemset(nextActiveNodeListD, 0, testNumNodes * sizeof(bool));
    //cacaulate the active node And make active node array
    dim3 grid = dim3(56, 1, 1);
    dim3 block = dim3(1024, 1, 1);

    int testTimes = 1;
    long timeSum = 0;
    for (int i = 0; i < testTimes; i++) {
        uint sourceNode = rand() % testNumNodes;
        sourceNode = sourceNodeSample;
        //sourceNode = 25838548;
        //sourceNode = 26890152;
        //sourceNode = 47235513;
        cout << "sourceNode " << sourceNode << endl;
        for (int j = 0; j < testNumNodes; j++) {
            isActiveNodeList[j] = false;
            value[j] = UINT_MAX;
        }
        isActiveNodeList[sourceNode] = true;
        value[sourceNode] = 1;
        hipMemcpy(valueD, value, testNumNodes * sizeof(uint), hipMemcpyHostToDevice);
        uint activeSum = 0;
        int iteration = 0;

        auto startProcessing = std::chrono::steady_clock::now();
        while (true) {
            uint activeNodeNum = 0;
            checkNeedTransferPartitionOpt(needTransferPartition, partitionInfoList, isActiveNodeList, partitionNum,
                                          testNumNodes, activeNodeNum);
            if (activeNodeNum <= 0) {
                break;
            } else {
                //cout << "iteration " << iteration << " activeNodes " << activeNodeNum << endl;
                activeSum += activeNodeNum;
            }
            hipMemcpy(isActiveNodeListD, isActiveNodeList, testNumNodes * sizeof(bool), hipMemcpyHostToDevice);
            for (int j = 0; j < partitionNum; j++) {
                if (needTransferPartition[j]) {
                    hipMemcpy(partitionEdgeListD, edgeList + partitionInfoList[j].nodePointerOffset,
                               partitionInfoList[j].partitionEdgeSize * sizeof(EdgeWithWeight), hipMemcpyHostToDevice);
                    traverseSum += partitionInfoList[j].partitionEdgeSize * sizeof(EdgeWithWeight);
                    ssspKernel_CommonPartition<<<grid, block>>>(partitionInfoList[j].startVertex,
                                                                partitionInfoList[j].endVertex,
                                                                partitionInfoList[j].nodePointerOffset,
                                                                isActiveNodeListD, nodePointerListD,
                                                                partitionEdgeListD, degreeD, valueD,
                                                                nextActiveNodeListD);
                    hipDeviceSynchronize();
                    gpuErrorcheck(hipPeekAtLastError())
                }
            }
            hipMemcpy(isActiveNodeList, nextActiveNodeListD, testNumNodes * sizeof(bool), hipMemcpyDeviceToHost);
            hipMemset(nextActiveNodeListD, 0, testNumNodes * sizeof(bool));
            iteration++;
        }
        cout << "traverseSum " << traverseSum << endl;
        cout << " activeSum " << activeSum << endl;
        auto endRead = std::chrono::steady_clock::now();
        long durationRead = std::chrono::duration_cast<std::chrono::milliseconds>(endRead - startProcessing).count();
        cout << " finish time : " << durationRead << " ms" << endl;
    }

    free(nodePointersI);
    free(edgeList);
    free(degree);
    free(isActiveNodeList);
    hipFree(isActiveNodeListD);
    hipFree(nextActiveNodeListD);
    hipFree(nodePointerListD);
    hipFree(partitionEdgeListD);
}

void ssspOpt(uint sourceNodeSample, string ssspPath, float adviseK) {
    uint testNumNodes = 0;
    ulong testNumEdge = 0;
    uint *nodePointersI;
    EdgeWithWeight *edgeList;
    bool isUseShare = true;

    auto startReadGraph = std::chrono::steady_clock::now();
    ifstream infile(ssspPath, ios::in | ios::binary);
    infile.read((char *) &testNumNodes, sizeof(uint));
    uint numEdge = 0;
    infile.read((char *) &numEdge, sizeof(uint));
    testNumEdge = numEdge;
    cout << "vertex num: " << testNumNodes << " edge num: " << testNumEdge << endl;

    nodePointersI = new uint[testNumNodes + 1];
    infile.read((char *) nodePointersI, sizeof(uint) * testNumNodes);
    edgeList = new EdgeWithWeight[testNumEdge + 1];
    infile.read((char *) edgeList, sizeof(EdgeWithWeight) * testNumEdge);
    infile.close();
    auto endReadGraph = std::chrono::steady_clock::now();
    long durationReadGraph = std::chrono::duration_cast<std::chrono::milliseconds>(
            endReadGraph - startReadGraph).count();
    cout << "read graph time : " << durationReadGraph << "ms" << endl;
    int testTimes = 1;
    long timeSum = 0;
    for (int i = 0; i < testTimes; i++) {
        //uint sourceNode = 25838548;
        //uint sourceNode = 26890152;
        uint sourceNode = 47235513;
        sourceNode = sourceNodeSample;
        cout << i << " sourceNode " << sourceNode << endl;
        timeSum += ssspCaculateCommonMemoryInnerAsyncRandom(testNumNodes, testNumEdge, nodePointersI, edgeList,
                                                            sourceNode, adviseK);
        //timeSum += ssspCaculateCommonMemoryInnerAsyncVisitRecord(testNumNodes, testNumEdge, nodePointersI, edgeList, sourceNode, adviseK);
        //break;
        cout << i << "========================================" << endl;
    }
}

void ssspShare(uint sourceNodeSample, string ssspPath) {
    uint testNumNodes = 0;
    ulong testNumEdge = 0;
    uint *nodePointersI;
    EdgeWithWeight *edgeList;

    auto startReadGraph = std::chrono::steady_clock::now();
    ifstream infile(ssspPath, ios::in | ios::binary);
    infile.read((char *) &testNumNodes, sizeof(uint));
    uint numEdge = 0;
    infile.read((char *) &numEdge, sizeof(uint));
    testNumEdge = numEdge;
    cout << "vertex num: " << testNumNodes << " edge num: " << testNumEdge << endl;
    gpuErrorcheck(hipMallocManaged(&nodePointersI, (testNumNodes + 1) * sizeof(uint)));
    infile.read((char *) nodePointersI, sizeof(uint) * testNumNodes);
    gpuErrorcheck(hipMallocManaged(&edgeList, (numEdge) * sizeof(EdgeWithWeight)));
    hipMemAdvise(nodePointersI, (testNumNodes + 1) * sizeof(uint), hipMemAdviseSetReadMostly, 0);
    hipMemAdvise(edgeList, (numEdge) * sizeof(EdgeWithWeight), hipMemAdviseSetReadMostly, 0);
    infile.read((char *) edgeList, sizeof(EdgeWithWeight) * testNumEdge);
    infile.close();
    //preprocessData(nodePointersI, edgeList, testNumNodes, testNumEdge);
    auto endReadGraph = std::chrono::steady_clock::now();
    long durationReadGraph = std::chrono::duration_cast<std::chrono::milliseconds>(
            endReadGraph - startReadGraph).count();
    cout << "read graph time : " << durationReadGraph << "ms" << endl;
    int testTimes = 1;
    long timeSum = 0;
    for (int i = 0; i < testTimes; i++) {
        uint sourceNode = sourceNodeSample;
        cout << i << " sourceNode " << sourceNode << endl;
        ssspCaculateInShare(testNumNodes, testNumEdge, nodePointersI, edgeList, sourceNode);
        break;
    }
}

long ssspCaculateInShare(uint testNumNodes, uint testNumEdge, uint *nodePointersI, EdgeWithWeight *edgeList,
                         uint sourceNode) {
    cout << "==================ssspshare==============" << endl;
    auto start = std::chrono::steady_clock::now();
    uint *degree;
    uint *value;
    uint sourceCode = 0;
    gpuErrorcheck(hipMallocManaged(&degree, testNumNodes * sizeof(uint)));
    gpuErrorcheck(hipMallocManaged(&value, testNumNodes * sizeof(uint)));

    auto startPreCaculate = std::chrono::steady_clock::now();
    for (uint i = 0; i < testNumNodes - 1; i++) {
        degree[i] = nodePointersI[i + 1] - nodePointersI[i];
    }

    degree[testNumNodes - 1] = testNumEdge - nodePointersI[testNumNodes - 1];
    sourceCode = sourceNode;
    bool *label;
    gpuErrorcheck(hipMallocManaged(&label, testNumNodes * sizeof(bool)));
    for (uint i = 0; i < testNumNodes; i++) {
        label[i] = false;
        value[i] = UINT_MAX - 1;
    }

    label[sourceCode] = true;
    value[sourceCode] = 1;
    uint *activeNodeList;
    hipMallocManaged(&activeNodeList, testNumNodes * sizeof(uint));
    //cacaulate the active node And make active node array
    uint *activeNodeLabelingD;
    gpuErrorcheck(hipMallocManaged(&activeNodeLabelingD, testNumNodes * sizeof(unsigned int)));
    uint *activeNodeLabelingPrefixD;
    gpuErrorcheck(hipMallocManaged(&activeNodeLabelingPrefixD, testNumNodes * sizeof(unsigned int)));
    dim3 grid = dim3(56, 1, 1);
    dim3 block = dim3(1024, 1, 1);

    auto endPreCaculate = std::chrono::steady_clock::now();
    long durationPreCaculate = std::chrono::duration_cast<std::chrono::milliseconds>(
            endPreCaculate - startPreCaculate).count();
    cout << "durationPreCaculate time : " << durationPreCaculate << " ms" << endl;

    setLabeling<<<grid, block>>>(testNumNodes, label, activeNodeLabelingD);
    thrust::device_ptr<unsigned int> ptr_labeling(activeNodeLabelingD);
    thrust::device_ptr<unsigned int> ptr_labeling_prefixsum(activeNodeLabelingPrefixD);
    uint activeNodesNum = thrust::reduce(ptr_labeling, ptr_labeling + testNumNodes);
    int iter = 0;
    uint nodeSum = activeNodesNum;
    //vector<vector<uint>> visitRecordByIteration;
    while (activeNodesNum > 0) {
        iter++;
        thrust::exclusive_scan(ptr_labeling, ptr_labeling + testNumNodes, ptr_labeling_prefixsum);
        setActiveNodeArray<<<grid, block>>>(testNumNodes, activeNodeList, label, activeNodeLabelingPrefixD);
        setLabelDefault<<<grid, block>>>(activeNodesNum, activeNodeList, label);

        sssp_kernel<<<grid, block>>>(activeNodesNum, activeNodeList, nodePointersI, degree, edgeList, value, label);
        hipDeviceSynchronize();
        gpuErrorcheck(hipPeekAtLastError());
        //visitRecordByIteration.push_back(countDataByIteration(testNumEdge, testNumNodes, nodePointersI, degree, activeNodeLabelingD));
        setLabeling<<<grid, block>>>(testNumNodes, label, activeNodeLabelingD);
        activeNodesNum = thrust::reduce(ptr_labeling, ptr_labeling + testNumNodes);
        nodeSum += activeNodesNum;
        cout << "iter: " << iter << " activeNodes: " << activeNodesNum << endl;
    }
    hipDeviceSynchronize();
    //writeTrunkVistInIteration(visitRecordByIteration, "./CountByIterationSSSP.txt");

    cout << "nodeSum: " << nodeSum << endl;

    auto endRead = std::chrono::steady_clock::now();
    long durationRead = std::chrono::duration_cast<std::chrono::milliseconds>(endRead - start).count();
    cout << "iter sum is " << iter << " finish time : " << durationRead << " ms" << endl;
    return durationRead;
}

//std::thread
void ssspDynamic(int tId,
                 int numThreads,
                 unsigned int overloadNodeBegin,
                 unsigned int numActiveNodes,
                 unsigned int *outDegree,
                 unsigned int *activeNodesPointer,
                 unsigned int *nodePointer,
                 unsigned int *activeNodes,
                 EdgeWithWeight *edgeListOverload,
                 EdgeWithWeight *edgeList) {

    unsigned int chunkSize = ceil((numActiveNodes - overloadNodeBegin) / numThreads) + 1;
    unsigned int left, right;
    left = tId * chunkSize + overloadNodeBegin;
    right = min(left + chunkSize, numActiveNodes);
    unsigned int thisNode;
    unsigned int thisDegree;
    unsigned int fromHere;
    unsigned int fromThere;

    for (unsigned int i = left; i < right; i++) {
        thisNode = activeNodes[i];
        thisDegree = outDegree[thisNode];
        fromHere = activeNodesPointer[i];
        fromThere = nodePointer[thisNode];
        for (unsigned int j = 0; j < thisDegree; j++) {
            uint temToNode, temWeight;
            temToNode = edgeList[fromThere + j].toNode;
            temWeight = edgeList[fromThere + j].weight;
            edgeListOverload[fromHere + j].toNode = temToNode;
            edgeListOverload[fromHere + j].weight = temWeight;
        }
    }

}

void ssspKernelThread(uint staticNodeNum, uint *activeNodeListD,
                      uint *staticNodePointerD, uint *degreeD,
                      EdgeWithWeight *staticEdgeListD, uint *valueD,
                      uint *isActiveD1,
                      uint *isActiveD2,
                      bool *isFinishedManaged, dim3 grid, dim3 block, hipStream_t steamStatic) {
    uint itr = 0;
    bool isFinishedHost = true;
    do {
        itr++;
        isFinishedHost = true;
        hipMemcpy(isFinishedManaged, &isFinishedHost, sizeof(bool), hipMemcpyHostToDevice);
        sssp_kernelStaticSwapOpt2Label<<<grid, block, 0, steamStatic>>>(staticNodeNum, activeNodeListD,
                                                                        staticNodePointerD, degreeD,
                                                                        staticEdgeListD, valueD,
                                                                        itr % 2 == 1 ? isActiveD1 : isActiveD2,
                                                                        itr % 2 == 1 ? isActiveD2 : isActiveD1,
                                                                        isFinishedManaged);
        hipDeviceSynchronize();
        gpuErrorcheck(hipPeekAtLastError())
        hipMemcpy(&isFinishedHost, isFinishedManaged, sizeof(bool), hipMemcpyDeviceToHost);
        isFinishedHost = true;
    } while (!isFinishedHost);
}

long
ssspCaculateCommonMemoryInnerAsync(uint testNumNodes, uint testNumEdge, uint *nodePointersI, EdgeWithWeight *edgeList,
                                   uint sourceNode, float adviseK) {
    cout << "=========ssspCaculateCommonMemoryInnerAsync========" << endl;
    ulong edgeIterationMax = 0;
    auto start = std::chrono::steady_clock::now();
    auto startPreCaculate = std::chrono::steady_clock::now();
    ulong transferSum = 0;
    //CPU
    long durationRead;
    uint fragmentNum = testNumEdge / fragment_size;
    unsigned long max_partition_size;
    unsigned long total_gpu_size;
    uint staticFragmentNum;
    uint maxStaticNode = 0;
    uint *degree;
    uint *value;
    uint *label;
    uint *staticFragmentToNormalMap;
    bool *isInStatic;
    uint *overloadNodeList;
    uint *staticNodePointer;
    uint *staticFragmentData;
    uint *overloadFragmentData;
    uint *activeNodeList;
    uint *activeOverloadNodePointers;
    vector<PartEdgeListInfo> partEdgeListInfoArr;
    EdgeWithWeight *overloadEdgeList;
    FragmentData *fragmentData;
    bool isFromTail = true;
    //GPU
    EdgeWithWeight *staticEdgeListD;
    EdgeWithWeight *overloadEdgeListD;
    bool *isInStaticD;
    uint *overloadNodeListD;
    uint *staticNodePointerD;
    uint *nodePointerD;
    uint *staticFragmentVisitRecordsD;
    uint *staticFragmentDataD;
    uint *canSwapStaticFragmentDataD;
    uint *canSwapFragmentPrefixSumD;
    uint *degreeD;
    // async need two labels
    uint *isActiveD1;
    uint *isActiveD2;
    uint *isStaticActive;
    uint *isOverloadActive;
    uint *valueD;
    uint *activeNodeListD;
    uint *activeNodeLabelingPrefixD;
    uint *activeOverloadNodePointersD;
    uint *activeOverloadDegreeD;
    bool *isFinishedDevice;

    degree = new uint[testNumNodes];
    value = new uint[testNumNodes];
    label = new uint[testNumNodes];
    isInStatic = new bool[testNumNodes];
    overloadNodeList = new uint[testNumNodes];
    staticNodePointer = new uint[testNumNodes];
    activeNodeList = new uint[testNumNodes];
    activeOverloadNodePointers = new uint[testNumNodes];
    fragmentData = new FragmentData[fragmentNum];

    getMaxPartitionSize(max_partition_size, total_gpu_size, testNumNodes, adviseK, sizeof(EdgeWithWeight), testNumEdge,
                        15);

    staticFragmentNum = max_partition_size / fragment_size;
    staticFragmentToNormalMap = new uint[staticFragmentNum];
    staticFragmentData = new uint[staticFragmentNum];
    overloadFragmentData = new uint[fragmentNum];
    //caculate degree
    uint meanDegree = testNumEdge / testNumNodes;
    cout << " meanDegree " << meanDegree << endl;
    uint degree0Sum = 0;
    for (uint i = 0; i < testNumNodes - 1; i++) {
        if (nodePointersI[i] > testNumEdge) {
            cout << i << "   " << nodePointersI[i] << endl;
            break;
        }
        degree[i] = nodePointersI[i + 1] - nodePointersI[i];
    }
    degree[testNumNodes - 1] = testNumEdge - nodePointersI[testNumNodes - 1];
    memcpy(staticNodePointer, nodePointersI, testNumNodes * sizeof(uint));

    //caculate static staticEdgeListD
    gpuErrorcheck(hipMalloc(&isFinishedDevice, 1 * sizeof(bool)));
    gpuErrorcheck(hipMalloc(&staticEdgeListD, max_partition_size * sizeof(EdgeWithWeight)));
    auto startmove = std::chrono::steady_clock::now();
    gpuErrorcheck(
            hipMemcpy(staticEdgeListD, edgeList, max_partition_size * sizeof(EdgeWithWeight), hipMemcpyHostToDevice));
    auto endMove = std::chrono::steady_clock::now();
    long testDuration = std::chrono::duration_cast<std::chrono::milliseconds>(
            endMove - startmove).count();
    cout << "move duration " << testDuration << endl;

    gpuErrorcheck(hipMalloc(&isInStaticD, testNumNodes * sizeof(bool)))
    gpuErrorcheck(hipMalloc(&overloadNodeListD, testNumNodes * sizeof(uint)));
    gpuErrorcheck(hipMalloc(&staticNodePointerD, testNumNodes * sizeof(uint)))
    gpuErrorcheck(hipMemcpy(staticNodePointerD, nodePointersI, testNumNodes * sizeof(uint), hipMemcpyHostToDevice));
    gpuErrorcheck(hipMalloc(&nodePointerD, testNumNodes * sizeof(uint)));
    gpuErrorcheck(hipMemcpy(nodePointerD, nodePointersI, testNumNodes * sizeof(uint), hipMemcpyHostToDevice));

    for (uint i = 0; i < testNumNodes; i++) {
        label[i] = 0;
        value[i] = UINT_MAX;

        uint pointStartFragmentIndex = nodePointersI[i] / fragment_size;
        uint pointEndFragmentIndex =
                degree[i] == 0 ? pointStartFragmentIndex : (nodePointersI[i] + degree[i] - 1) / fragment_size;
        if (pointStartFragmentIndex == pointEndFragmentIndex && pointStartFragmentIndex >= 0 &&
            pointStartFragmentIndex < fragmentNum) {
            if (fragmentData[pointStartFragmentIndex].vertexNum == 0) {
                fragmentData[pointStartFragmentIndex].startVertex = i;
            } else if (fragmentData[pointStartFragmentIndex].startVertex > i) {
                fragmentData[pointStartFragmentIndex].startVertex = i;
            }
            fragmentData[pointStartFragmentIndex].vertexNum++;
        }

        if (nodePointersI[i] < max_partition_size && (nodePointersI[i] + degree[i] - 1) < max_partition_size) {
            isInStatic[i] = true;
            if (i > maxStaticNode) maxStaticNode = i;
        } else {
            isInStatic[i] = false;
        }
    }
    label[sourceNode] = 1;
    value[sourceNode] = 1;
    hipMemcpy(isInStaticD, isInStatic, testNumNodes * sizeof(bool), hipMemcpyHostToDevice);
    cout << "max_partition_size: " << max_partition_size << "  maxStaticNode: " << maxStaticNode << endl;
    cout << "fragmentNum " << fragmentNum << " staticFragmentNum " << staticFragmentNum << endl;
    for (int i = 0; i < staticFragmentNum; i++) {
        fragmentData[i].isIn = true;
    }
    for (uint i = 0; i < staticFragmentNum; i++) {
        staticFragmentToNormalMap[i] = i;
    }
    uint partOverloadSize = total_gpu_size - max_partition_size;
    uint overloadSize = testNumEdge - nodePointersI[maxStaticNode + 1];
    cout << " partOverloadSize " << partOverloadSize << " overloadSize " << overloadSize << endl;
    //overloadEdgeList = (EdgeWithWeight *) malloc(overloadSize * sizeof(EdgeWithWeight));
    overloadEdgeList = new EdgeWithWeight[overloadSize];
    gpuErrorcheck(hipMalloc(&overloadEdgeListD, partOverloadSize * sizeof(EdgeWithWeight)));
    //gpuErrorcheck(hipMallocManaged(&edgeListOverloadManage, overloadSize * sizeof(uint)));
    gpuErrorcheck(hipMalloc(&staticFragmentDataD, staticFragmentNum * sizeof(uint)));
    gpuErrorcheck(hipMalloc(&staticFragmentVisitRecordsD, staticFragmentNum * sizeof(uint)));
    gpuErrorcheck(hipMalloc(&canSwapStaticFragmentDataD, staticFragmentNum * sizeof(uint)));
    gpuErrorcheck(hipMalloc(&canSwapFragmentPrefixSumD, staticFragmentNum * sizeof(uint)));
    thrust::device_ptr<unsigned int> ptr_canSwapFragment(canSwapStaticFragmentDataD);
    thrust::device_ptr<unsigned int> ptr_canSwapFragmentPrefixSum(canSwapFragmentPrefixSumD);
    gpuErrorcheck(hipMalloc(&degreeD, testNumNodes * sizeof(uint)));
    gpuErrorcheck(hipMalloc(&isActiveD1, testNumNodes * sizeof(uint)));
    gpuErrorcheck(hipMalloc(&isActiveD2, testNumNodes * sizeof(uint)));
    gpuErrorcheck(hipMalloc(&isStaticActive, testNumNodes * sizeof(uint)));
    gpuErrorcheck(hipMalloc(&isOverloadActive, testNumNodes * sizeof(uint)));
    gpuErrorcheck(hipMalloc(&valueD, testNumNodes * sizeof(uint)));
    gpuErrorcheck(hipMalloc(&activeNodeLabelingPrefixD, testNumNodes * sizeof(unsigned int)));
    gpuErrorcheck(hipMalloc(&activeNodeListD, testNumNodes * sizeof(uint)));
    gpuErrorcheck(hipMalloc(&activeOverloadNodePointersD, testNumNodes * sizeof(uint)));
    gpuErrorcheck(hipMalloc(&activeOverloadDegreeD, testNumNodes * sizeof(uint)));
    gpuErrorcheck(hipMemcpy(degreeD, degree, testNumNodes * sizeof(uint), hipMemcpyHostToDevice));
    gpuErrorcheck(hipMemcpy(valueD, value, testNumNodes * sizeof(uint), hipMemcpyHostToDevice));
    gpuErrorcheck(hipMemcpy(isActiveD1, label, testNumNodes * sizeof(uint), hipMemcpyHostToDevice));
    gpuErrorcheck(hipMemset(isActiveD2, 0, testNumNodes * sizeof(uint)));
    gpuErrorcheck(hipMemset(isStaticActive, 0, testNumNodes * sizeof(uint)));
    gpuErrorcheck(hipMemset(isOverloadActive, 0, testNumNodes * sizeof(uint)));

    //cacaulate the active node And make active node array
    dim3 grid = dim3(56, 1, 1);
    dim3 block = dim3(1024, 1, 1);

    //setLabeling<<<grid, block>>>(testNumNodes, labelD, activeNodeLabelingD);
    thrust::device_ptr<unsigned int> ptr_labeling(isActiveD1);
    thrust::device_ptr<unsigned int> ptr_labeling_static(isStaticActive);
    thrust::device_ptr<unsigned int> ptr_labeling_overload(isOverloadActive);
    thrust::device_ptr<unsigned int> ptr_labeling_prefixsum(activeNodeLabelingPrefixD);
    thrust::device_ptr<unsigned int> ptrOverloadDegree(activeOverloadDegreeD);
    thrust::device_ptr<unsigned int> ptrOverloadPrefixsum(activeOverloadNodePointersD);

    uint activeNodesNum = thrust::reduce(ptr_labeling, ptr_labeling + testNumNodes);
    int iter = 0;
    uint nodeSum = activeNodesNum;
    ulong overloadEdgeSum = 0;
    auto startCpu = std::chrono::steady_clock::now();
    auto endReadCpu = std::chrono::steady_clock::now();
    long durationReadCpu = 0;

    auto startSwap = std::chrono::steady_clock::now();
    auto endSwap = std::chrono::steady_clock::now();
    long durationSwap = 0;

    auto startGpuProcessing = std::chrono::steady_clock::now();
    auto endGpuProcessing = std::chrono::steady_clock::now();
    long durationGpuProcessing = 0;

    auto startOverloadGpuProcessing = std::chrono::steady_clock::now();
    auto endOverloadGpuProcessing = std::chrono::steady_clock::now();
    long durationOverloadGpuProcessing = 0;

    auto startPreGpuProcessing = std::chrono::steady_clock::now();
    auto endPreGpuProcessing = std::chrono::steady_clock::now();
    long durationPreGpuProcessing = 0;
    auto endPreCaculate = std::chrono::steady_clock::now();
    long durationPreCaculate = std::chrono::duration_cast<std::chrono::milliseconds>(
            endPreCaculate - startPreCaculate).count();
    cout << "durationPreCaculate time : " << durationPreCaculate << " ms" << endl;
    hipStream_t steamStatic, streamDynamic;
    hipStreamCreate(&steamStatic);
    hipStreamCreate(&streamDynamic);
    auto startMemoryTraverse = std::chrono::steady_clock::now();
    auto endMemoryTraverse = std::chrono::steady_clock::now();
    long durationMemoryTraverse = 0;
    uint cursorStartSwap = isFromTail ? fragmentNum - 1 : staticFragmentNum + 1;
    //uint cursorStartSwap = staticFragmentNum + 1;
    uint swapValidNodeSum = 0;
    uint swapValidEdgeSum = 0;
    uint swapNotValidNodeSum = 0;
    uint swapNotValidEdgeSum = 0;
    uint visitEdgeSum = 0;
    uint swapInEdgeSum = 0;
    auto startProcessing = std::chrono::steady_clock::now();
    auto startTest = std::chrono::steady_clock::now();
    auto endTest = std::chrono::steady_clock::now();
    long durationTest = 0;
    while (activeNodesNum > 0) {
        iter++;
        cout << "iter " << iter << " activeNodesNum " << activeNodesNum << endl;
        startPreGpuProcessing = std::chrono::steady_clock::now();
        cleanStaticAndOverloadLabel<<<grid, block>>>(testNumNodes, isStaticActive, isOverloadActive);
        setStaticAndOverloadLabel<<<grid, block>>>(testNumNodes, isActiveD1, isStaticActive, isOverloadActive,
                                                   isInStaticD);
        uint staticNodeNum = thrust::reduce(ptr_labeling_static, ptr_labeling_static + testNumNodes);
        cout << "iter " << iter << " staticNodeNum " << staticNodeNum << endl;
        thrust::exclusive_scan(ptr_labeling_static, ptr_labeling_static + testNumNodes, ptr_labeling_prefixsum);
        setStaticActiveNodeArray<<<grid, block>>>(testNumNodes, activeNodeListD, isStaticActive,
                                                  activeNodeLabelingPrefixD);

        uint overloadNodeNum = thrust::reduce(ptr_labeling_overload, ptr_labeling_overload + testNumNodes);
        cout << "iter " << iter << " overloadNodeNum " << overloadNodeNum << endl;

        thrust::exclusive_scan(ptr_labeling_overload, ptr_labeling_overload + testNumNodes, ptr_labeling_prefixsum);
        setOverloadNodePointerSwap<<<grid, block>>>(testNumNodes, overloadNodeListD, activeOverloadDegreeD,
                                                    isOverloadActive,
                                                    activeNodeLabelingPrefixD, degreeD);

        thrust::exclusive_scan(ptrOverloadDegree, ptrOverloadDegree + overloadNodeNum, activeOverloadNodePointersD);
        uint overloadEdgeNum = thrust::reduce(thrust::device, ptrOverloadDegree,
                                              ptrOverloadDegree + overloadNodeNum, 0);
        cout << "iter " << iter << " overloadEdgeNum " << overloadEdgeNum << endl;
        overloadEdgeSum += overloadEdgeNum;
        if (overloadEdgeNum > edgeIterationMax) {
            edgeIterationMax = overloadEdgeNum;
        }

        endPreGpuProcessing = std::chrono::steady_clock::now();
        durationPreGpuProcessing += std::chrono::duration_cast<std::chrono::milliseconds>(
                endPreGpuProcessing - startPreGpuProcessing).count();

        startGpuProcessing = std::chrono::steady_clock::now();
        mixDynamicPartLabel<<<grid, block, 0, steamStatic>>>(staticNodeNum, 0, activeNodeListD, isActiveD1,
                                                             isActiveD2);

        /*sssp_kernelStaticSwapOpt2Label<<<grid, block, 0, steamStatic>>>(staticNodeNum, activeNodeListD,
                                                                        staticNodePointerD, degreeD,
                                                                        staticEdgeListD, valueD, isActiveD1, isActiveD2,
                                                                        isFinishedManaged);*/
        thread staticSSSPKernel = thread(ssspKernelThread, staticNodeNum, activeNodeListD, staticNodePointerD, degreeD,
                                         staticEdgeListD, valueD, isActiveD1, isActiveD2, isFinishedDevice, grid,
                                         block,
                                         steamStatic);

        if (overloadNodeNum > 0) {
            startCpu = std::chrono::steady_clock::now();
            /*hipMemcpyAsync(staticActiveNodeList, activeNodeListD, activeNodesNum * sizeof(uint), hipMemcpyDeviceToHost,
                            streamDynamic);*/
            hipMemcpyAsync(overloadNodeList, overloadNodeListD, overloadNodeNum * sizeof(uint), hipMemcpyDeviceToHost,
                            streamDynamic);
            hipMemcpyAsync(activeOverloadNodePointers, activeOverloadNodePointersD, overloadNodeNum * sizeof(uint),
                            hipMemcpyDeviceToHost, streamDynamic);

            int threadNum = 20;
            if (overloadNodeNum < 50) {
                threadNum = 1;
            }
            thread runThreads[threadNum];

            for (int i = 0; i < threadNum; i++) {
                runThreads[i] = thread(ssspDynamic,
                                       i,
                                       threadNum,
                                       0,
                                       overloadNodeNum,
                                       degree,
                                       activeOverloadNodePointers,
                                       nodePointersI,
                                       overloadNodeList,
                                       overloadEdgeList,
                                       edgeList);
            }

            for (unsigned int t = 0; t < threadNum; t++) {
                runThreads[t].join();
            }
            caculatePartInfoForEdgeList(activeOverloadNodePointers, overloadNodeList, degree, partEdgeListInfoArr,
                                        overloadNodeNum, partOverloadSize, overloadEdgeNum);

            endReadCpu = std::chrono::steady_clock::now();
            durationReadCpu += std::chrono::duration_cast<std::chrono::milliseconds>(endReadCpu - startCpu).count();
            if (staticSSSPKernel.joinable()) {
                staticSSSPKernel.join();
            }
            hipDeviceSynchronize();
            endGpuProcessing = std::chrono::steady_clock::now();
            durationGpuProcessing += std::chrono::duration_cast<std::chrono::milliseconds>(
                    endGpuProcessing - startGpuProcessing).count();

            for (auto &i : partEdgeListInfoArr) {
                startTest = std::chrono::steady_clock::now();
                startMemoryTraverse = std::chrono::steady_clock::now();
                gpuErrorcheck(hipMemcpy(overloadEdgeListD, overloadEdgeList +
                                                            activeOverloadNodePointers[i.partStartIndex],
                                         i.partEdgeNums * sizeof(EdgeWithWeight),
                                         hipMemcpyHostToDevice))
                transferSum += i.partEdgeNums;
                endMemoryTraverse = std::chrono::steady_clock::now();
                durationMemoryTraverse += std::chrono::duration_cast<std::chrono::milliseconds>(
                        endMemoryTraverse - startMemoryTraverse).count();

                startOverloadGpuProcessing = std::chrono::steady_clock::now();
                mixDynamicPartLabel<<<grid, block, 0, streamDynamic>>>(i.partActiveNodeNums,
                                                                       i.partStartIndex,
                                                                       overloadNodeListD, isActiveD1,
                                                                       isActiveD2);
                uint itr = 0;
                bool isFinishedHost = true;
                do {
                    itr++;
                    isFinishedHost = true;
                    hipMemcpy(isFinishedDevice, &isFinishedHost, sizeof(bool), hipMemcpyHostToDevice);

                    sssp_kernelDynamicSwap2Label<<<grid, block, 0, streamDynamic>>>(
                            i.partStartIndex,
                            i.partActiveNodeNums,
                            overloadNodeListD, degreeD,
                            valueD, itr % 2 == 1 ? isActiveD1
                                                 : isActiveD2,
                            itr % 2 == 1 ? isActiveD2
                                         : isActiveD1,
                            overloadEdgeListD,
                            activeOverloadNodePointersD,
                            isFinishedDevice);
                    hipDeviceSynchronize();
                    hipMemcpy(&isFinishedHost, isFinishedDevice, sizeof(bool), hipMemcpyDeviceToHost);
                    //cout << "dynamic itr " << itr << " *isFinishedManaged " << *isFinishedManaged << endl;
                    //*isFinishedManaged = true;
                    isFinishedHost = true;
                } while (!isFinishedHost);
                endOverloadGpuProcessing = std::chrono::steady_clock::now();
                durationOverloadGpuProcessing += std::chrono::duration_cast<std::chrono::milliseconds>(
                        endOverloadGpuProcessing - startOverloadGpuProcessing).count();

                endTest = std::chrono::steady_clock::now();
                durationTest += std::chrono::duration_cast<std::chrono::milliseconds>(
                        endTest - startTest).count();
            }
            //gpuErrorcheck(hipPeekAtLastError())

        } else {
            if (staticSSSPKernel.joinable()) {
                staticSSSPKernel.join();
            }
            endGpuProcessing = std::chrono::steady_clock::now();
            durationGpuProcessing += std::chrono::duration_cast<std::chrono::milliseconds>(
                    endGpuProcessing - startGpuProcessing).count();
        }


        startPreGpuProcessing = std::chrono::steady_clock::now();
        mixCommonLabel<<<grid, block, 0, streamDynamic>>>(testNumNodes, isActiveD1, isActiveD2);
        activeNodesNum = thrust::reduce(ptr_labeling, ptr_labeling + testNumNodes);
        nodeSum += activeNodesNum;
        endPreGpuProcessing = std::chrono::steady_clock::now();
        durationPreGpuProcessing += std::chrono::duration_cast<std::chrono::milliseconds>(
                endPreGpuProcessing - startPreGpuProcessing).count();
    }
    auto endRead = std::chrono::steady_clock::now();
    durationRead = std::chrono::duration_cast<std::chrono::milliseconds>(endRead - startProcessing).count();
    hipDeviceSynchronize();

    cout << "nodeSum: " << nodeSum << endl;
    transferSum += max_partition_size;
    cout << "iterationSum " << iter << endl;
    double edgeIterationAvg = (double) overloadEdgeSum / (double) testNumEdge / iter;
    double edgeIterationMaxAvg = (double) edgeIterationMax / (double) testNumEdge;
    cout << "edgeIterationAvg " << edgeIterationAvg << " edgeIterationMaxAvg " << edgeIterationMaxAvg << endl;
    cout << "transferSum : " << transferSum * sizeof(EdgeWithWeight) << " bytes" << endl;
    cout << "finish time : " << durationRead << " ms" << endl;
    cout << "total time : " << testDuration + durationRead << " ms" << endl;
    cout << "cpu time : " << durationReadCpu << " ms" << endl;
    cout << "pre fact processing time : " << durationGpuProcessing << " ms" << endl;
    cout << "dynamic fact processing time : " << durationOverloadGpuProcessing << " ms" << endl;
    cout << "dynamic move time : " << durationMemoryTraverse << " ms" << endl;

    cout << "gpu pre processing time : " << durationPreGpuProcessing << " ms" << endl;
    cout << "swap processing time : " << durationSwap << " ms" << endl;
    cout << "overloadEdgeSum : " << overloadEdgeSum << " " << endl;
    cout << "durationTest : " << durationTest << " " << endl;

    cout << "swapValidNodeSum " << swapValidNodeSum << " swapValidEdgeSum " << swapValidEdgeSum << endl;
    cout << "swapNotValidNodeSum " << swapNotValidNodeSum << " swapNotValidEdgeSum " << swapNotValidEdgeSum
         << " visitSum " << visitEdgeSum << " swapInEdgeSum " << swapInEdgeSum << endl;


    hipFree(nodePointerD);
    hipFree(staticEdgeListD);
    //hipFree(edgeListOverloadManage);
    hipFree(degreeD);
    hipFree(isActiveD1);
    hipFree(isActiveD2);
    hipFree(valueD);
    hipFree(activeNodeListD);
    hipFree(activeNodeLabelingPrefixD);
    hipFree(activeOverloadNodePointersD);
    hipFree(activeOverloadDegreeD);
    hipFree(isInStaticD);
    hipFree(staticNodePointerD);
    hipFree(overloadNodeListD);
    hipFree(staticFragmentVisitRecordsD);
    hipFree(staticFragmentDataD);
    hipFree(canSwapStaticFragmentDataD);
    hipFree(canSwapFragmentPrefixSumD);
    hipFree(overloadEdgeListD);
    hipFree(isStaticActive);
    hipFree(isOverloadActive);
    hipFree(isFinishedDevice);
    delete[]            label;
    delete[]            degree;
    delete[]            value;
    delete[]            activeNodeList;
    delete[]            activeOverloadNodePointers;
    delete[] staticFragmentData;
    delete[] isInStatic;
    delete[] overloadNodeList;
    delete[] staticNodePointer;
    delete[] staticFragmentToNormalMap;
    delete[] fragmentData;
    delete[] overloadFragmentData;
    delete[] overloadEdgeList;
    partEdgeListInfoArr.clear();
    return durationRead;
}

long
ssspCaculateUVM(uint testNumNodes, uint testNumEdge, uint *nodePointersI, EdgeWithWeight *edgeList,
                uint sourceNode) {
    cout << "=========ssspCaculateCommonMemoryInnerAsync========" << endl;
    auto start = std::chrono::steady_clock::now();
    auto startPreCaculate = std::chrono::steady_clock::now();
    ulong transferSum = 0;
    //CPU
    long durationRead;
    uint fragmentNum = testNumEdge / fragment_size;
    unsigned long max_partition_size;
    unsigned long total_gpu_size;
    uint staticFragmentNum;
    uint maxStaticNode = 0;
    uint *degree;
    uint *value;
    uint *label;
    uint *staticFragmentToNormalMap;
    bool *isInStatic;
    uint *overloadNodeList;
    uint *staticNodePointer;
    uint *staticFragmentData;
    uint *overloadFragmentData;
    uint *activeNodeList;
    uint *activeOverloadNodePointers;
    //vector<PartEdgeListInfo> partEdgeListInfoArr;
    EdgeWithWeight *overloadEdgeListManage;
    FragmentData *fragmentData;
    bool isFromTail = true;
    //GPU
    EdgeWithWeight *staticEdgeListD;
    //EdgeWithWeight *overloadEdgeListD;
    bool *isInStaticD;
    uint *overloadNodeListD;
    uint *staticNodePointerD;
    uint *nodePointerD;
    uint *staticFragmentVisitRecordsD;
    uint *staticFragmentDataD;
    uint *canSwapStaticFragmentDataD;
    uint *canSwapFragmentPrefixSumD;
    uint *degreeD;
    // async need two labels
    uint *isActiveD1;
    uint *isActiveD2;
    uint *isStaticActive;
    uint *isOverloadActive;
    uint *valueD;
    uint *activeNodeListD;
    uint *activeNodeLabelingPrefixD;
    uint *activeOverloadNodePointersD;
    uint *activeOverloadDegreeD;
    bool *isFinishedDevice;

    degree = new uint[testNumNodes];
    value = new uint[testNumNodes];
    label = new uint[testNumNodes];
    isInStatic = new bool[testNumNodes];
    overloadNodeList = new uint[testNumNodes];
    staticNodePointer = new uint[testNumNodes];
    activeNodeList = new uint[testNumNodes];
    activeOverloadNodePointers = new uint[testNumNodes];
    fragmentData = new FragmentData[fragmentNum];

    getMaxPartitionSize(max_partition_size, total_gpu_size, testNumNodes, 0.88, sizeof(EdgeWithWeight), testNumEdge,
                        15);

    staticFragmentNum = max_partition_size / fragment_size;
    staticFragmentToNormalMap = new uint[staticFragmentNum];
    staticFragmentData = new uint[staticFragmentNum];
    overloadFragmentData = new uint[fragmentNum];
    //caculate degree
    uint meanDegree = testNumEdge / testNumNodes;
    cout << " meanDegree " << meanDegree << endl;
    uint degree0Sum = 0;
    for (uint i = 0; i < testNumNodes - 1; i++) {
        if (nodePointersI[i] > testNumEdge) {
            cout << i << "   " << nodePointersI[i] << endl;
            break;
        }
        degree[i] = nodePointersI[i + 1] - nodePointersI[i];
    }
    degree[testNumNodes - 1] = testNumEdge - nodePointersI[testNumNodes - 1];
    memcpy(staticNodePointer, nodePointersI, testNumNodes * sizeof(uint));

    //caculate static staticEdgeListD
    gpuErrorcheck(hipMalloc(&isFinishedDevice, 1 * sizeof(bool)));
    gpuErrorcheck(hipMalloc(&staticEdgeListD, max_partition_size * sizeof(EdgeWithWeight)));
    gpuErrorcheck(
            hipMemcpy(staticEdgeListD, edgeList, max_partition_size * sizeof(EdgeWithWeight), hipMemcpyHostToDevice));
    gpuErrorcheck(hipMalloc(&isInStaticD, testNumNodes * sizeof(bool)))
    gpuErrorcheck(hipMalloc(&overloadNodeListD, testNumNodes * sizeof(uint)));
    gpuErrorcheck(hipMalloc(&staticNodePointerD, testNumNodes * sizeof(uint)))
    gpuErrorcheck(hipMemcpy(staticNodePointerD, nodePointersI, testNumNodes * sizeof(uint), hipMemcpyHostToDevice));
    gpuErrorcheck(hipMalloc(&nodePointerD, testNumNodes * sizeof(uint)));
    gpuErrorcheck(hipMemcpy(nodePointerD, nodePointersI, testNumNodes * sizeof(uint), hipMemcpyHostToDevice));

    for (uint i = 0; i < testNumNodes; i++) {
        label[i] = 0;
        value[i] = UINT_MAX;

        uint pointStartFragmentIndex = nodePointersI[i] / fragment_size;
        uint pointEndFragmentIndex =
                degree[i] == 0 ? pointStartFragmentIndex : (nodePointersI[i] + degree[i] - 1) / fragment_size;
        if (pointStartFragmentIndex == pointEndFragmentIndex && pointStartFragmentIndex >= 0 &&
            pointStartFragmentIndex < fragmentNum) {
            if (fragmentData[pointStartFragmentIndex].vertexNum == 0) {
                fragmentData[pointStartFragmentIndex].startVertex = i;
            } else if (fragmentData[pointStartFragmentIndex].startVertex > i) {
                fragmentData[pointStartFragmentIndex].startVertex = i;
            }
            fragmentData[pointStartFragmentIndex].vertexNum++;
        }

        if (nodePointersI[i] < max_partition_size && (nodePointersI[i] + degree[i] - 1) < max_partition_size) {
            isInStatic[i] = true;
            if (i > maxStaticNode) maxStaticNode = i;
        } else {
            isInStatic[i] = false;
        }
    }
    label[sourceNode] = 1;
    value[sourceNode] = 1;
    hipMemcpy(isInStaticD, isInStatic, testNumNodes * sizeof(bool), hipMemcpyHostToDevice);
    cout << "max_partition_size: " << max_partition_size << "  maxStaticNode: " << maxStaticNode << endl;
    cout << "fragmentNum " << fragmentNum << " staticFragmentNum " << staticFragmentNum << endl;
    for (int i = 0; i < staticFragmentNum; i++) {
        fragmentData[i].isIn = true;
    }
    for (uint i = 0; i < staticFragmentNum; i++) {
        staticFragmentToNormalMap[i] = i;
    }
    uint partOverloadSize = total_gpu_size - max_partition_size;
    uint overloadSize = testNumEdge - nodePointersI[maxStaticNode + 1];
    cout << " partOverloadSize " << partOverloadSize << " overloadSize " << overloadSize << endl;
    //overloadEdgeList = (EdgeWithWeight *) malloc(overloadSize * sizeof(EdgeWithWeight));
    gpuErrorcheck(hipMallocManaged(&overloadEdgeListManage, overloadSize * sizeof(EdgeWithWeight)));
    //overloadEdgeList = new EdgeWithWeight[overloadSize];
    //gpuErrorcheck(hipMalloc(&overloadEdgeListD, partOverloadSize * sizeof(EdgeWithWeight)));
    //gpuErrorcheck(hipMallocManaged(&edgeListOverloadManage, overloadSize * sizeof(uint)));
    gpuErrorcheck(hipMalloc(&staticFragmentDataD, staticFragmentNum * sizeof(uint)));
    gpuErrorcheck(hipMalloc(&staticFragmentVisitRecordsD, staticFragmentNum * sizeof(uint)));
    gpuErrorcheck(hipMalloc(&canSwapStaticFragmentDataD, staticFragmentNum * sizeof(uint)));
    gpuErrorcheck(hipMalloc(&canSwapFragmentPrefixSumD, staticFragmentNum * sizeof(uint)));
    thrust::device_ptr<unsigned int> ptr_canSwapFragment(canSwapStaticFragmentDataD);
    thrust::device_ptr<unsigned int> ptr_canSwapFragmentPrefixSum(canSwapFragmentPrefixSumD);
    gpuErrorcheck(hipMalloc(&degreeD, testNumNodes * sizeof(uint)));
    gpuErrorcheck(hipMalloc(&isActiveD1, testNumNodes * sizeof(uint)));
    gpuErrorcheck(hipMalloc(&isActiveD2, testNumNodes * sizeof(uint)));
    gpuErrorcheck(hipMalloc(&isStaticActive, testNumNodes * sizeof(uint)));
    gpuErrorcheck(hipMalloc(&isOverloadActive, testNumNodes * sizeof(uint)));
    gpuErrorcheck(hipMalloc(&valueD, testNumNodes * sizeof(uint)));
    gpuErrorcheck(hipMalloc(&activeNodeLabelingPrefixD, testNumNodes * sizeof(unsigned int)));
    gpuErrorcheck(hipMalloc(&activeNodeListD, testNumNodes * sizeof(uint)));
    gpuErrorcheck(hipMalloc(&activeOverloadNodePointersD, testNumNodes * sizeof(uint)));
    gpuErrorcheck(hipMalloc(&activeOverloadDegreeD, testNumNodes * sizeof(uint)));
    gpuErrorcheck(hipMemcpy(degreeD, degree, testNumNodes * sizeof(uint), hipMemcpyHostToDevice));
    gpuErrorcheck(hipMemcpy(valueD, value, testNumNodes * sizeof(uint), hipMemcpyHostToDevice));
    gpuErrorcheck(hipMemcpy(isActiveD1, label, testNumNodes * sizeof(uint), hipMemcpyHostToDevice));
    gpuErrorcheck(hipMemset(isActiveD2, 0, testNumNodes * sizeof(uint)));
    gpuErrorcheck(hipMemset(isStaticActive, 0, testNumNodes * sizeof(uint)));
    gpuErrorcheck(hipMemset(isOverloadActive, 0, testNumNodes * sizeof(uint)));

    //cacaulate the active node And make active node array
    dim3 grid = dim3(56, 1, 1);
    dim3 block = dim3(1024, 1, 1);

    //setLabeling<<<grid, block>>>(testNumNodes, labelD, activeNodeLabelingD);
    thrust::device_ptr<unsigned int> ptr_labeling(isActiveD1);
    thrust::device_ptr<unsigned int> ptr_labeling_static(isStaticActive);
    thrust::device_ptr<unsigned int> ptr_labeling_overload(isOverloadActive);
    thrust::device_ptr<unsigned int> ptr_labeling_prefixsum(activeNodeLabelingPrefixD);
    thrust::device_ptr<unsigned int> ptrOverloadDegree(activeOverloadDegreeD);
    thrust::device_ptr<unsigned int> ptrOverloadPrefixsum(activeOverloadNodePointersD);

    uint activeNodesNum = thrust::reduce(ptr_labeling, ptr_labeling + testNumNodes);
    int iter = 0;
    uint nodeSum = activeNodesNum;
    uint overloadEdgeSum = 0;
    auto startCpu = std::chrono::steady_clock::now();
    auto endReadCpu = std::chrono::steady_clock::now();
    long durationReadCpu = 0;

    auto startSwap = std::chrono::steady_clock::now();
    auto endSwap = std::chrono::steady_clock::now();
    long durationSwap = 0;

    auto startGpuProcessing = std::chrono::steady_clock::now();
    auto endGpuProcessing = std::chrono::steady_clock::now();
    long durationGpuProcessing = 0;

    auto startOverloadGpuProcessing = std::chrono::steady_clock::now();
    auto endOverloadGpuProcessing = std::chrono::steady_clock::now();
    long durationOverloadGpuProcessing = 0;

    auto startPreGpuProcessing = std::chrono::steady_clock::now();
    auto endPreGpuProcessing = std::chrono::steady_clock::now();
    long durationPreGpuProcessing = 0;
    auto endPreCaculate = std::chrono::steady_clock::now();
    long durationPreCaculate = std::chrono::duration_cast<std::chrono::milliseconds>(
            endPreCaculate - startPreCaculate).count();
    cout << "durationPreCaculate time : " << durationPreCaculate << " ms" << endl;
    hipStream_t steamStatic, streamDynamic;
    hipStreamCreate(&steamStatic);
    hipStreamCreate(&streamDynamic);
    auto startMemoryTraverse = std::chrono::steady_clock::now();
    auto endMemoryTraverse = std::chrono::steady_clock::now();
    long durationMemoryTraverse = 0;
    auto startProcessing = std::chrono::steady_clock::now();
    uint cursorStartSwap = isFromTail ? fragmentNum - 1 : staticFragmentNum + 1;
    //uint cursorStartSwap = staticFragmentNum + 1;
    uint swapValidNodeSum = 0;
    uint swapValidEdgeSum = 0;
    uint swapNotValidNodeSum = 0;
    uint swapNotValidEdgeSum = 0;
    uint visitEdgeSum = 0;
    uint swapInEdgeSum = 0;

    while (activeNodesNum > 0) {
        iter++;
        cout << "iter " << iter << " activeNodesNum " << activeNodesNum << endl;
        startPreGpuProcessing = std::chrono::steady_clock::now();
        cleanStaticAndOverloadLabel<<<grid, block>>>(testNumNodes, isStaticActive, isOverloadActive);
        setStaticAndOverloadLabel<<<grid, block>>>(testNumNodes, isActiveD1, isStaticActive, isOverloadActive,
                                                   isInStaticD);
        uint staticNodeNum = thrust::reduce(ptr_labeling_static, ptr_labeling_static + testNumNodes);
        cout << "iter " << iter << " staticNodeNum " << staticNodeNum << endl;
        thrust::exclusive_scan(ptr_labeling_static, ptr_labeling_static + testNumNodes, ptr_labeling_prefixsum);
        setStaticActiveNodeArray<<<grid, block>>>(testNumNodes, activeNodeListD, isStaticActive,
                                                  activeNodeLabelingPrefixD);

        uint overloadNodeNum = thrust::reduce(ptr_labeling_overload, ptr_labeling_overload + testNumNodes);
        cout << "iter " << iter << " overloadNodeNum " << overloadNodeNum << endl;

        thrust::exclusive_scan(ptr_labeling_overload, ptr_labeling_overload + testNumNodes, ptr_labeling_prefixsum);
        setOverloadNodePointerSwap<<<grid, block>>>(testNumNodes, overloadNodeListD, activeOverloadDegreeD,
                                                    isOverloadActive,
                                                    activeNodeLabelingPrefixD, degreeD);

        thrust::exclusive_scan(ptrOverloadDegree, ptrOverloadDegree + overloadNodeNum, activeOverloadNodePointersD);
        uint overloadEdgeNum = thrust::reduce(thrust::device, ptrOverloadDegree,
                                              ptrOverloadDegree + overloadNodeNum, 0);
        cout << "iter " << iter << " overloadEdgeNum " << overloadEdgeNum << endl;
        overloadEdgeSum += overloadEdgeNum;

        endPreGpuProcessing = std::chrono::steady_clock::now();
        durationPreGpuProcessing += std::chrono::duration_cast<std::chrono::milliseconds>(
                endPreGpuProcessing - startPreGpuProcessing).count();
        startGpuProcessing = std::chrono::steady_clock::now();
        mixDynamicPartLabel<<<grid, block, 0, steamStatic>>>(staticNodeNum, 0, activeNodeListD, isActiveD1,
                                                             isActiveD2);

        /*sssp_kernelStaticSwapOpt2Label<<<grid, block, 0, steamStatic>>>(staticNodeNum, activeNodeListD,
                                                                        staticNodePointerD, degreeD,
                                                                        staticEdgeListD, valueD, isActiveD1, isActiveD2,
                                                                        isFinishedManaged);*/
        thread staticSSSPKernel = thread(ssspKernelThread, staticNodeNum, activeNodeListD, staticNodePointerD, degreeD,
                                         staticEdgeListD, valueD, isActiveD1, isActiveD2, isFinishedDevice, grid,
                                         block,
                                         steamStatic);
        if (overloadNodeNum > 0) {
            startCpu = std::chrono::steady_clock::now();
            /*hipMemcpyAsync(staticActiveNodeList, activeNodeListD, activeNodesNum * sizeof(uint), hipMemcpyDeviceToHost,
                            streamDynamic);*/
            hipMemcpyAsync(overloadNodeList, overloadNodeListD, overloadNodeNum * sizeof(uint), hipMemcpyDeviceToHost,
                            streamDynamic);
            hipMemcpyAsync(activeOverloadNodePointers, activeOverloadNodePointersD, overloadNodeNum * sizeof(uint),
                            hipMemcpyDeviceToHost, streamDynamic);

            int threadNum = 20;
            if (overloadNodeNum < 50) {
                threadNum = 1;
            }
            thread runThreads[threadNum];
            hipMemAdvise(overloadEdgeListManage, overloadEdgeNum * sizeof(EdgeWithWeight), hipMemAdviseSetAccessedBy,
                          hipCpuDeviceId);
            for (int i = 0; i < threadNum; i++) {
                runThreads[i] = thread(ssspDynamic,
                                       i,
                                       threadNum,
                                       0,
                                       overloadNodeNum,
                                       degree,
                                       activeOverloadNodePointers,
                                       nodePointersI,
                                       overloadNodeList,
                                       overloadEdgeListManage,
                                       edgeList);
            }

            for (unsigned int t = 0; t < threadNum; t++) {
                runThreads[t].join();
            }
            hipMemAdvise(overloadEdgeListManage, overloadEdgeNum * sizeof(EdgeWithWeight),
                          hipMemAdviseUnsetAccessedBy,
                          hipCpuDeviceId);
            /*caculatePartInfoForEdgeList(activeOverloadNodePointers, overloadNodeList, degree, partEdgeListInfoArr,
                                        overloadNodeNum, partOverloadSize, overloadEdgeNum);*/

            endReadCpu = std::chrono::steady_clock::now();
            durationReadCpu += std::chrono::duration_cast<std::chrono::milliseconds>(endReadCpu - startCpu).count();
            if (staticSSSPKernel.joinable()) {
                staticSSSPKernel.join();
            }
            hipDeviceSynchronize();
            endGpuProcessing = std::chrono::steady_clock::now();
            durationGpuProcessing += std::chrono::duration_cast<std::chrono::milliseconds>(
                    endGpuProcessing - startGpuProcessing).count();

            //hipMemAdvise(overloadEdgeListManage, overloadEdgeNum * sizeof(EdgeWithWeight), hipMemAdviseSetReadMostly, 0);
            mixDynamicPartLabel<<<grid, block, 0, streamDynamic>>>(overloadNodeNum,
                                                                   0,
                                                                   overloadNodeListD, isActiveD1,
                                                                   isActiveD2);
            hipDeviceSynchronize();
            gpuErrorcheck(hipPeekAtLastError())
            cout << "sssp_kernelDynamicSwap2Label" << endl;


            sssp_kernelDynamicUvm<<<grid, block, 0, streamDynamic>>>(
                    overloadNodeNum, overloadNodeListD, activeOverloadNodePointersD, degreeD, overloadEdgeListManage,
                    valueD,
                    isActiveD1, isActiveD2);

            hipDeviceSynchronize();
            //hipMemAdvise(overloadEdgeListManage, overloadEdgeNum * sizeof(EdgeWithWeight), hipMemAdviseUnsetReadMostly, 0);
            gpuErrorcheck(hipPeekAtLastError())

        } else {
            if (staticSSSPKernel.joinable()) {
                staticSSSPKernel.join();
            }
            endGpuProcessing = std::chrono::steady_clock::now();
            durationGpuProcessing += std::chrono::duration_cast<std::chrono::milliseconds>(
                    endGpuProcessing - startGpuProcessing).count();
        }
        mixCommonLabel<<<grid, block, 0, streamDynamic>>>(testNumNodes, isActiveD1, isActiveD2);
        startPreGpuProcessing = std::chrono::steady_clock::now();
        activeNodesNum = thrust::reduce(ptr_labeling, ptr_labeling + testNumNodes);
        nodeSum += activeNodesNum;
        endPreGpuProcessing = std::chrono::steady_clock::now();
        durationPreGpuProcessing += std::chrono::duration_cast<std::chrono::milliseconds>(
                endPreGpuProcessing - startPreGpuProcessing).count();
        /*if (iter == 3) {
            break;
        } */
    }
    hipDeviceSynchronize();
    /*hipMemcpy(value, valueD, testNumNodes * sizeof(uint), hipMemcpyDeviceToHost);
    uint sum1 = 0;
    for (uint i = 0; i < testNumNodes; i++) {
        if (value[i] == 1) {
            sum1++;
        }
    }
    cout << "sum1: " << sum1 << endl;*/

    cout << "nodeSum: " << nodeSum << endl;
    auto endRead = std::chrono::steady_clock::now();
    durationRead = std::chrono::duration_cast<std::chrono::milliseconds>(endRead - startProcessing).count();
    transferSum += max_partition_size;
    cout << "transferSum : " << transferSum * sizeof(EdgeWithWeight) << " bytes" << endl;
    cout << "finish time : " << durationRead << " ms" << endl;
    cout << "cpu time : " << durationReadCpu << " ms" << endl;
    cout << "pre fact processing time : " << durationGpuProcessing << " ms" << endl;
    cout << "overload fact processing time : " << durationOverloadGpuProcessing << " ms" << endl;

    cout << "gpu pre processing time : " << durationPreGpuProcessing << " ms" << endl;
    cout << "swap processing time : " << durationSwap << " ms" << endl;
    cout << "overloadEdgeSum : " << overloadEdgeSum << " " << endl;

    cout << "swapValidNodeSum " << swapValidNodeSum << " swapValidEdgeSum " << swapValidEdgeSum << endl;
    cout << "swapNotValidNodeSum " << swapNotValidNodeSum << " swapNotValidEdgeSum " << swapNotValidEdgeSum
         << " visitSum " << visitEdgeSum << " swapInEdgeSum " << swapInEdgeSum << endl;


    hipFree(nodePointerD);
    hipFree(staticEdgeListD);
    //hipFree(edgeListOverloadManage);
    hipFree(degreeD);
    hipFree(isActiveD1);
    hipFree(isActiveD2);
    hipFree(valueD);
    hipFree(activeNodeListD);
    hipFree(activeNodeLabelingPrefixD);
    hipFree(activeOverloadNodePointersD);
    hipFree(activeOverloadDegreeD);
    hipFree(isInStaticD);
    hipFree(staticNodePointerD);
    hipFree(overloadNodeListD);
    hipFree(staticFragmentVisitRecordsD);
    hipFree(staticFragmentDataD);
    hipFree(canSwapStaticFragmentDataD);
    hipFree(canSwapFragmentPrefixSumD);
    //hipFree(overloadEdgeListD);
    hipFree(isStaticActive);
    hipFree(isOverloadActive);
    hipFree(isFinishedDevice);
    delete[]            label;
    delete[]            degree;
    delete[]            value;
    delete[]            activeNodeList;
    delete[]            activeOverloadNodePointers;
    delete[] staticFragmentData;
    delete[] isInStatic;
    delete[] overloadNodeList;
    delete[] staticNodePointer;
    delete[] staticFragmentToNormalMap;
    delete[] fragmentData;
    delete[] overloadFragmentData;
    //delete[] overloadEdgeList;
    //partEdgeListInfoArr.clear();
    return durationRead;
}

long
ssspCaculateCommonMemoryInnerAsyncVisitRecord(uint testNumNodes, uint testNumEdge, uint *nodePointersI,
                                              EdgeWithWeight *edgeList,
                                              uint sourceNode, float adviseK) {
    cout << "=========ssspCaculateCommonMemoryInnerAsync========" << endl;
    ulong edgeIterationMax = 0;
    auto start = std::chrono::steady_clock::now();
    auto startPreCaculate = std::chrono::steady_clock::now();
    ulong transferSum = 0;
    //CPU
    long durationRead;
    uint fragmentNum = testNumEdge / fragment_size;
    unsigned long max_partition_size;
    unsigned long total_gpu_size;
    uint staticFragmentNum;
    uint maxStaticNode = 0;
    uint *degree;
    uint *value;
    uint *label;
    uint *staticFragmentToNormalMap;
    bool *isInStatic;
    uint *overloadNodeList;
    uint *staticNodePointer;
    uint *staticFragmentData;
    uint *overloadFragmentData;
    uint *activeNodeList;
    uint *activeOverloadNodePointers;
    vector<PartEdgeListInfo> partEdgeListInfoArr;
    EdgeWithWeight *overloadEdgeList;
    FragmentData *fragmentData;
    bool isFromTail = true;
    //GPU
    EdgeWithWeight *staticEdgeListD;
    EdgeWithWeight *overloadEdgeListD;
    bool *isInStaticD;
    uint *overloadNodeListD;
    uint *staticNodePointerD;
    uint *nodePointerD;
    uint *staticFragmentVisitRecordsD;
    uint *staticFragmentDataD;
    uint *canSwapStaticFragmentDataD;
    uint *canSwapFragmentPrefixSumD;
    uint *degreeD;
    // async need two labels
    uint *isActiveD1;
    uint *isActiveD2;
    uint *isStaticActive;
    uint *isOverloadActive;
    uint *valueD;
    uint *activeNodeListD;
    uint *activeNodeLabelingPrefixD;
    uint *activeOverloadNodePointersD;
    uint *activeOverloadDegreeD;
    bool *isFinishedDevice;
    uint *vertexVisitRecord;
    uint *vertexVisitRecordD;
    vertexVisitRecord = new uint[testNumNodes];
    hipMalloc(&vertexVisitRecordD, testNumNodes * sizeof(uint));
    hipMemset(vertexVisitRecordD, 0, testNumNodes * sizeof(uint));

    degree = new uint[testNumNodes];
    value = new uint[testNumNodes];
    label = new uint[testNumNodes];
    isInStatic = new bool[testNumNodes];
    overloadNodeList = new uint[testNumNodes];
    staticNodePointer = new uint[testNumNodes];
    activeNodeList = new uint[testNumNodes];
    activeOverloadNodePointers = new uint[testNumNodes];
    fragmentData = new FragmentData[fragmentNum];

    getMaxPartitionSize(max_partition_size, total_gpu_size, testNumNodes, adviseK, sizeof(EdgeWithWeight), testNumEdge,
                        16);

    staticFragmentNum = max_partition_size / fragment_size;
    staticFragmentToNormalMap = new uint[staticFragmentNum];
    staticFragmentData = new uint[staticFragmentNum];
    overloadFragmentData = new uint[fragmentNum];
    //caculate degree
    uint meanDegree = testNumEdge / testNumNodes;
    cout << " meanDegree " << meanDegree << endl;
    uint degree0Sum = 0;
    for (uint i = 0; i < testNumNodes - 1; i++) {
        if (nodePointersI[i] > testNumEdge) {
            cout << i << "   " << nodePointersI[i] << endl;
            break;
        }
        degree[i] = nodePointersI[i + 1] - nodePointersI[i];
    }
    degree[testNumNodes - 1] = testNumEdge - nodePointersI[testNumNodes - 1];
    memcpy(staticNodePointer, nodePointersI, testNumNodes * sizeof(uint));

    //caculate static staticEdgeListD
    gpuErrorcheck(hipMalloc(&isFinishedDevice, 1 * sizeof(bool)));
    gpuErrorcheck(hipMalloc(&staticEdgeListD, max_partition_size * sizeof(EdgeWithWeight)));
    auto startmove = std::chrono::steady_clock::now();
    gpuErrorcheck(
            hipMemcpy(staticEdgeListD, edgeList, max_partition_size * sizeof(EdgeWithWeight), hipMemcpyHostToDevice));
    auto endMove = std::chrono::steady_clock::now();
    long testDuration = std::chrono::duration_cast<std::chrono::milliseconds>(
            endMove - startmove).count();
    cout << "move duration " << testDuration << endl;

    gpuErrorcheck(hipMalloc(&isInStaticD, testNumNodes * sizeof(bool)))
    gpuErrorcheck(hipMalloc(&overloadNodeListD, testNumNodes * sizeof(uint)));
    gpuErrorcheck(hipMalloc(&staticNodePointerD, testNumNodes * sizeof(uint)))
    gpuErrorcheck(hipMemcpy(staticNodePointerD, nodePointersI, testNumNodes * sizeof(uint), hipMemcpyHostToDevice));
    gpuErrorcheck(hipMalloc(&nodePointerD, testNumNodes * sizeof(uint)));
    gpuErrorcheck(hipMemcpy(nodePointerD, nodePointersI, testNumNodes * sizeof(uint), hipMemcpyHostToDevice));

    for (uint i = 0; i < testNumNodes; i++) {
        label[i] = 0;
        value[i] = UINT_MAX;

        uint pointStartFragmentIndex = nodePointersI[i] / fragment_size;
        uint pointEndFragmentIndex =
                degree[i] == 0 ? pointStartFragmentIndex : (nodePointersI[i] + degree[i] - 1) / fragment_size;
        if (pointStartFragmentIndex == pointEndFragmentIndex && pointStartFragmentIndex >= 0 &&
            pointStartFragmentIndex < fragmentNum) {
            if (fragmentData[pointStartFragmentIndex].vertexNum == 0) {
                fragmentData[pointStartFragmentIndex].startVertex = i;
            } else if (fragmentData[pointStartFragmentIndex].startVertex > i) {
                fragmentData[pointStartFragmentIndex].startVertex = i;
            }
            fragmentData[pointStartFragmentIndex].vertexNum++;
        }

        if (nodePointersI[i] < max_partition_size && (nodePointersI[i] + degree[i] - 1) < max_partition_size) {
            isInStatic[i] = true;
            if (i > maxStaticNode) maxStaticNode = i;
        } else {
            isInStatic[i] = false;
        }
    }
    label[sourceNode] = 1;
    value[sourceNode] = 1;
    hipMemcpy(isInStaticD, isInStatic, testNumNodes * sizeof(bool), hipMemcpyHostToDevice);
    cout << "max_partition_size: " << max_partition_size << "  maxStaticNode: " << maxStaticNode << endl;
    cout << "fragmentNum " << fragmentNum << " staticFragmentNum " << staticFragmentNum << endl;
    for (int i = 0; i < staticFragmentNum; i++) {
        fragmentData[i].isIn = true;
    }
    for (uint i = 0; i < staticFragmentNum; i++) {
        staticFragmentToNormalMap[i] = i;
    }
    uint partOverloadSize = total_gpu_size - max_partition_size;
    uint overloadSize = testNumEdge - nodePointersI[maxStaticNode + 1];
    cout << " partOverloadSize " << partOverloadSize << " overloadSize " << overloadSize << endl;
    //overloadEdgeList = (EdgeWithWeight *) malloc(overloadSize * sizeof(EdgeWithWeight));
    overloadEdgeList = new EdgeWithWeight[overloadSize];
    gpuErrorcheck(hipMalloc(&overloadEdgeListD, partOverloadSize * sizeof(EdgeWithWeight)));
    //gpuErrorcheck(hipMallocManaged(&edgeListOverloadManage, overloadSize * sizeof(uint)));
    gpuErrorcheck(hipMalloc(&staticFragmentDataD, staticFragmentNum * sizeof(uint)));
    gpuErrorcheck(hipMalloc(&staticFragmentVisitRecordsD, staticFragmentNum * sizeof(uint)));
    gpuErrorcheck(hipMalloc(&canSwapStaticFragmentDataD, staticFragmentNum * sizeof(uint)));
    gpuErrorcheck(hipMalloc(&canSwapFragmentPrefixSumD, staticFragmentNum * sizeof(uint)));
    thrust::device_ptr<unsigned int> ptr_canSwapFragment(canSwapStaticFragmentDataD);
    thrust::device_ptr<unsigned int> ptr_canSwapFragmentPrefixSum(canSwapFragmentPrefixSumD);
    gpuErrorcheck(hipMalloc(&degreeD, testNumNodes * sizeof(uint)));
    gpuErrorcheck(hipMalloc(&isActiveD1, testNumNodes * sizeof(uint)));
    gpuErrorcheck(hipMalloc(&isActiveD2, testNumNodes * sizeof(uint)));
    gpuErrorcheck(hipMalloc(&isStaticActive, testNumNodes * sizeof(uint)));
    gpuErrorcheck(hipMalloc(&isOverloadActive, testNumNodes * sizeof(uint)));
    gpuErrorcheck(hipMalloc(&valueD, testNumNodes * sizeof(uint)));
    gpuErrorcheck(hipMalloc(&activeNodeLabelingPrefixD, testNumNodes * sizeof(unsigned int)));
    gpuErrorcheck(hipMalloc(&activeNodeListD, testNumNodes * sizeof(uint)));
    gpuErrorcheck(hipMalloc(&activeOverloadNodePointersD, testNumNodes * sizeof(uint)));
    gpuErrorcheck(hipMalloc(&activeOverloadDegreeD, testNumNodes * sizeof(uint)));
    gpuErrorcheck(hipMemcpy(degreeD, degree, testNumNodes * sizeof(uint), hipMemcpyHostToDevice));
    gpuErrorcheck(hipMemcpy(valueD, value, testNumNodes * sizeof(uint), hipMemcpyHostToDevice));
    gpuErrorcheck(hipMemcpy(isActiveD1, label, testNumNodes * sizeof(uint), hipMemcpyHostToDevice));
    gpuErrorcheck(hipMemset(isActiveD2, 0, testNumNodes * sizeof(uint)));
    gpuErrorcheck(hipMemset(isStaticActive, 0, testNumNodes * sizeof(uint)));
    gpuErrorcheck(hipMemset(isOverloadActive, 0, testNumNodes * sizeof(uint)));

    //cacaulate the active node And make active node array
    dim3 grid = dim3(56, 1, 1);
    dim3 block = dim3(1024, 1, 1);

    //setLabeling<<<grid, block>>>(testNumNodes, labelD, activeNodeLabelingD);
    thrust::device_ptr<unsigned int> ptr_labeling(isActiveD1);
    thrust::device_ptr<unsigned int> ptr_labeling_static(isStaticActive);
    thrust::device_ptr<unsigned int> ptr_labeling_overload(isOverloadActive);
    thrust::device_ptr<unsigned int> ptr_labeling_prefixsum(activeNodeLabelingPrefixD);
    thrust::device_ptr<unsigned int> ptrOverloadDegree(activeOverloadDegreeD);
    thrust::device_ptr<unsigned int> ptrOverloadPrefixsum(activeOverloadNodePointersD);

    uint activeNodesNum = thrust::reduce(ptr_labeling, ptr_labeling + testNumNodes);
    int iter = 0;
    uint nodeSum = activeNodesNum;
    ulong overloadEdgeSum = 0;
    auto startCpu = std::chrono::steady_clock::now();
    auto endReadCpu = std::chrono::steady_clock::now();
    long durationReadCpu = 0;

    auto startSwap = std::chrono::steady_clock::now();
    auto endSwap = std::chrono::steady_clock::now();
    long durationSwap = 0;

    auto startGpuProcessing = std::chrono::steady_clock::now();
    auto endGpuProcessing = std::chrono::steady_clock::now();
    long durationGpuProcessing = 0;

    auto startOverloadGpuProcessing = std::chrono::steady_clock::now();
    auto endOverloadGpuProcessing = std::chrono::steady_clock::now();
    long durationOverloadGpuProcessing = 0;

    auto startPreGpuProcessing = std::chrono::steady_clock::now();
    auto endPreGpuProcessing = std::chrono::steady_clock::now();
    long durationPreGpuProcessing = 0;
    auto endPreCaculate = std::chrono::steady_clock::now();
    long durationPreCaculate = std::chrono::duration_cast<std::chrono::milliseconds>(
            endPreCaculate - startPreCaculate).count();
    cout << "durationPreCaculate time : " << durationPreCaculate << " ms" << endl;
    hipStream_t steamStatic, streamDynamic;
    hipStreamCreate(&steamStatic);
    hipStreamCreate(&streamDynamic);
    auto startMemoryTraverse = std::chrono::steady_clock::now();
    auto endMemoryTraverse = std::chrono::steady_clock::now();
    long durationMemoryTraverse = 0;
    auto startProcessing = std::chrono::steady_clock::now();
    uint cursorStartSwap = isFromTail ? fragmentNum - 1 : staticFragmentNum + 1;
    //uint cursorStartSwap = staticFragmentNum + 1;
    uint swapValidNodeSum = 0;
    uint swapValidEdgeSum = 0;
    uint swapNotValidNodeSum = 0;
    uint swapNotValidEdgeSum = 0;
    uint visitEdgeSum = 0;
    uint swapInEdgeSum = 0;

    while (activeNodesNum > 0) {
        iter++;
        //cout << "iter " << iter << " activeNodesNum " << activeNodesNum << endl;
        startPreGpuProcessing = std::chrono::steady_clock::now();
        cleanStaticAndOverloadLabel<<<grid, block>>>(testNumNodes, isStaticActive, isOverloadActive);
        setStaticAndOverloadLabelAndRecord<<<grid, block>>>(testNumNodes, isActiveD1, isStaticActive, isOverloadActive,
                                                            isInStaticD, vertexVisitRecordD);
        uint staticNodeNum = thrust::reduce(ptr_labeling_static, ptr_labeling_static + testNumNodes);
        //cout << "iter " << iter << " staticNodeNum " << staticNodeNum << endl;
        thrust::exclusive_scan(ptr_labeling_static, ptr_labeling_static + testNumNodes, ptr_labeling_prefixsum);
        setStaticActiveNodeArray<<<grid, block>>>(testNumNodes, activeNodeListD, isStaticActive,
                                                  activeNodeLabelingPrefixD);

        uint overloadNodeNum = thrust::reduce(ptr_labeling_overload, ptr_labeling_overload + testNumNodes);
        //cout << "iter " << iter << " overloadNodeNum " << overloadNodeNum << endl;

        thrust::exclusive_scan(ptr_labeling_overload, ptr_labeling_overload + testNumNodes, ptr_labeling_prefixsum);
        setOverloadNodePointerSwap<<<grid, block>>>(testNumNodes, overloadNodeListD, activeOverloadDegreeD,
                                                    isOverloadActive,
                                                    activeNodeLabelingPrefixD, degreeD);

        thrust::exclusive_scan(ptrOverloadDegree, ptrOverloadDegree + overloadNodeNum, activeOverloadNodePointersD);
        uint overloadEdgeNum = thrust::reduce(thrust::device, ptrOverloadDegree,
                                              ptrOverloadDegree + overloadNodeNum, 0);
        //cout << "iter " << iter << " overloadEdgeNum " << overloadEdgeNum << endl;
        overloadEdgeSum += overloadEdgeNum;
        if (overloadEdgeNum > edgeIterationMax) {
            edgeIterationMax = overloadEdgeNum;
        }

        endPreGpuProcessing = std::chrono::steady_clock::now();
        durationPreGpuProcessing += std::chrono::duration_cast<std::chrono::milliseconds>(
                endPreGpuProcessing - startPreGpuProcessing).count();
        startGpuProcessing = std::chrono::steady_clock::now();
        mixDynamicPartLabel<<<grid, block, 0, steamStatic>>>(staticNodeNum, 0, activeNodeListD, isActiveD1,
                                                             isActiveD2);

        /*sssp_kernelStaticSwapOpt2Label<<<grid, block, 0, steamStatic>>>(staticNodeNum, activeNodeListD,
                                                                        staticNodePointerD, degreeD,
                                                                        staticEdgeListD, valueD, isActiveD1, isActiveD2,
                                                                        isFinishedManaged);*/
        thread staticSSSPKernel = thread(ssspKernelThread, staticNodeNum, activeNodeListD, staticNodePointerD, degreeD,
                                         staticEdgeListD, valueD, isActiveD1, isActiveD2, isFinishedDevice, grid,
                                         block,
                                         steamStatic);

        if (staticSSSPKernel.joinable()) {
            staticSSSPKernel.join();
        }

        if (overloadNodeNum > 0) {
            startCpu = std::chrono::steady_clock::now();
            /*hipMemcpyAsync(staticActiveNodeList, activeNodeListD, activeNodesNum * sizeof(uint), hipMemcpyDeviceToHost,
                            streamDynamic);*/
            hipMemcpyAsync(overloadNodeList, overloadNodeListD, overloadNodeNum * sizeof(uint), hipMemcpyDeviceToHost,
                            streamDynamic);
            hipMemcpyAsync(activeOverloadNodePointers, activeOverloadNodePointersD, overloadNodeNum * sizeof(uint),
                            hipMemcpyDeviceToHost, streamDynamic);

            int threadNum = 20;
            if (overloadNodeNum < 50) {
                threadNum = 1;
            }
            thread runThreads[threadNum];

            for (int i = 0; i < threadNum; i++) {
                runThreads[i] = thread(ssspDynamic,
                                       i,
                                       threadNum,
                                       0,
                                       overloadNodeNum,
                                       degree,
                                       activeOverloadNodePointers,
                                       nodePointersI,
                                       overloadNodeList,
                                       overloadEdgeList,
                                       edgeList);
            }

            for (unsigned int t = 0; t < threadNum; t++) {
                runThreads[t].join();
            }
            caculatePartInfoForEdgeList(activeOverloadNodePointers, overloadNodeList, degree, partEdgeListInfoArr,
                                        overloadNodeNum, partOverloadSize, overloadEdgeNum);

            endReadCpu = std::chrono::steady_clock::now();
            durationReadCpu += std::chrono::duration_cast<std::chrono::milliseconds>(endReadCpu - startCpu).count();
            if (staticSSSPKernel.joinable()) {
                staticSSSPKernel.join();
            }
            hipDeviceSynchronize();
            endGpuProcessing = std::chrono::steady_clock::now();
            durationGpuProcessing += std::chrono::duration_cast<std::chrono::milliseconds>(
                    endGpuProcessing - startGpuProcessing).count();

            for (int i = 0; i < partEdgeListInfoArr.size(); i++) {
                startMemoryTraverse = std::chrono::steady_clock::now();
                gpuErrorcheck(hipMemcpy(overloadEdgeListD, overloadEdgeList +
                                                            activeOverloadNodePointers[partEdgeListInfoArr[i].partStartIndex],
                                         partEdgeListInfoArr[i].partEdgeNums * sizeof(EdgeWithWeight),
                                         hipMemcpyHostToDevice))
                transferSum += partEdgeListInfoArr[i].partEdgeNums;
                endMemoryTraverse = std::chrono::steady_clock::now();
                durationMemoryTraverse = std::chrono::duration_cast<std::chrono::milliseconds>(
                        endMemoryTraverse - startMemoryTraverse).count();
                /*cout << "iter " << iter << " part " << i << " durationMemoryTraverse "
                     << durationMemoryTraverse << endl;*/
                startOverloadGpuProcessing = std::chrono::steady_clock::now();
                mixDynamicPartLabel<<<grid, block, 0, streamDynamic>>>(partEdgeListInfoArr[i].partActiveNodeNums,
                                                                       partEdgeListInfoArr[i].partStartIndex,
                                                                       overloadNodeListD, isActiveD1,
                                                                       isActiveD2);
                uint itr = 0;
                bool isFinishedHost = true;
                do {
                    itr++;
                    isFinishedHost = true;
                    hipMemcpy(isFinishedDevice, &isFinishedHost, sizeof(bool), hipMemcpyHostToDevice);

                    sssp_kernelDynamicSwap2Label<<<grid, block, 0, streamDynamic>>>(
                            partEdgeListInfoArr[i].partStartIndex,
                            partEdgeListInfoArr[i].partActiveNodeNums,
                            overloadNodeListD, degreeD,
                            valueD, itr % 2 == 1 ? isActiveD1
                                                 : isActiveD2,
                            itr % 2 == 1 ? isActiveD2
                                         : isActiveD1,
                            overloadEdgeListD,
                            activeOverloadNodePointersD,
                            isFinishedDevice);
                    hipDeviceSynchronize();
                    hipMemcpy(&isFinishedHost, isFinishedDevice, sizeof(bool), hipMemcpyDeviceToHost);
                    //cout << "dynamic itr " << itr << " *isFinishedManaged " << *isFinishedManaged << endl;
                    //*isFinishedManaged = true;
                    isFinishedHost = true;
                } while (!isFinishedHost);
                endOverloadGpuProcessing = std::chrono::steady_clock::now();
                durationOverloadGpuProcessing = std::chrono::duration_cast<std::chrono::milliseconds>(
                        endOverloadGpuProcessing - startOverloadGpuProcessing).count();
                /*cout << "iter " << iter << " part " << i << " durationOverloadGpuProcessing "
                     << durationOverloadGpuProcessing << endl;*/
            }
            gpuErrorcheck(hipPeekAtLastError())

        } else {
            if (staticSSSPKernel.joinable()) {
                staticSSSPKernel.join();
            }
            endGpuProcessing = std::chrono::steady_clock::now();
            durationGpuProcessing += std::chrono::duration_cast<std::chrono::milliseconds>(
                    endGpuProcessing - startGpuProcessing).count();
        }
        mixCommonLabel<<<grid, block, 0, streamDynamic>>>(testNumNodes, isActiveD1, isActiveD2);
        startPreGpuProcessing = std::chrono::steady_clock::now();
        activeNodesNum = thrust::reduce(ptr_labeling, ptr_labeling + testNumNodes);
        nodeSum += activeNodesNum;
        endPreGpuProcessing = std::chrono::steady_clock::now();
        durationPreGpuProcessing += std::chrono::duration_cast<std::chrono::milliseconds>(
                endPreGpuProcessing - startPreGpuProcessing).count();
    }
    hipDeviceSynchronize();

    cout << "nodeSum: " << nodeSum << endl;
    auto endRead = std::chrono::steady_clock::now();
    durationRead = std::chrono::duration_cast<std::chrono::milliseconds>(endRead - startProcessing).count();
    hipMemcpy(vertexVisitRecord, vertexVisitRecordD, testNumNodes * sizeof(uint), hipMemcpyDeviceToHost);
    uint partNum = 50;
    uint partSize = testNumEdge / partNum;
    vector<uint> partVistRecordList(partNum + 1);
    uint partSizeCursor = 0;
    for (uint i = 0; i < testNumNodes; i++) {
        uint edgeStartIndex = nodePointersI[i];
        uint edgeEndIndex = nodePointersI[i] + degree[i];
        uint maxPartIndex = partSizeCursor * partSize + partSize;

        if (edgeStartIndex < maxPartIndex && edgeEndIndex < maxPartIndex) {
            partVistRecordList[partSizeCursor] += vertexVisitRecord[i] * degree[i];
        } else if (edgeStartIndex < maxPartIndex && edgeEndIndex >= maxPartIndex) {
            partVistRecordList[partSizeCursor] += vertexVisitRecord[i] * (maxPartIndex - edgeStartIndex);
            partSizeCursor += 1;
            partVistRecordList[partSizeCursor] += vertexVisitRecord[i] * (edgeEndIndex - maxPartIndex);
        } else {
            partSizeCursor += 1;
            partVistRecordList[partSizeCursor] += vertexVisitRecord[i] * degree[i];
        }
    }
    for (uint i = 0; i < partNum + 1; i++) {
        cout << "part " << i << " is " << partVistRecordList[i] << endl;
    }
    for (uint i = 0; i < partNum + 1; i++) {
        cout << partVistRecordList[i] << "\t";
    }

    transferSum += max_partition_size;
    cout << "iterationSum " << iter << endl;
    double edgeIterationAvg = (double) overloadEdgeSum / (double) testNumEdge / iter;
    double edgeIterationMaxAvg = (double) edgeIterationMax / (double) testNumEdge;
    cout << "edgeIterationAvg " << edgeIterationAvg << " edgeIterationMaxAvg " << edgeIterationMaxAvg << endl;
    cout << "transferSum : " << transferSum * sizeof(EdgeWithWeight) << " bytes" << endl;
    cout << "finish time : " << durationRead << " ms" << endl;
    cout << "total time : " << testDuration + durationRead << " ms" << endl;
    cout << "cpu time : " << durationReadCpu << " ms" << endl;
    cout << "pre fact processing time : " << durationGpuProcessing << " ms" << endl;
    cout << "overload fact processing time : " << durationOverloadGpuProcessing << " ms" << endl;

    cout << "gpu pre processing time : " << durationPreGpuProcessing << " ms" << endl;
    cout << "swap processing time : " << durationSwap << " ms" << endl;
    cout << "overloadEdgeSum : " << overloadEdgeSum << " " << endl;

    cout << "swapValidNodeSum " << swapValidNodeSum << " swapValidEdgeSum " << swapValidEdgeSum << endl;
    cout << "swapNotValidNodeSum " << swapNotValidNodeSum << " swapNotValidEdgeSum " << swapNotValidEdgeSum
         << " visitSum " << visitEdgeSum << " swapInEdgeSum " << swapInEdgeSum << endl;


    hipFree(nodePointerD);
    hipFree(staticEdgeListD);
    //hipFree(edgeListOverloadManage);
    hipFree(degreeD);
    hipFree(isActiveD1);
    hipFree(isActiveD2);
    hipFree(valueD);
    hipFree(activeNodeListD);
    hipFree(activeNodeLabelingPrefixD);
    hipFree(activeOverloadNodePointersD);
    hipFree(activeOverloadDegreeD);
    hipFree(isInStaticD);
    hipFree(staticNodePointerD);
    hipFree(overloadNodeListD);
    hipFree(staticFragmentVisitRecordsD);
    hipFree(staticFragmentDataD);
    hipFree(canSwapStaticFragmentDataD);
    hipFree(canSwapFragmentPrefixSumD);
    hipFree(overloadEdgeListD);
    hipFree(isStaticActive);
    hipFree(isOverloadActive);
    hipFree(isFinishedDevice);
    delete[]            label;
    delete[]            degree;
    delete[]            value;
    delete[]            activeNodeList;
    delete[]            activeOverloadNodePointers;
    delete[] staticFragmentData;
    delete[] isInStatic;
    delete[] overloadNodeList;
    delete[] staticNodePointer;
    delete[] staticFragmentToNormalMap;
    delete[] fragmentData;
    delete[] overloadFragmentData;
    delete[] overloadEdgeList;
    partEdgeListInfoArr.clear();
    return durationRead;
}


void ssspShareTrace(uint sourceNodeSample, string ssspPath) {
    uint testNumNodes = 0;
    ulong testNumEdge = 0;
    uint *nodePointersI;
    EdgeWithWeight *edgeList;

    auto startReadGraph = std::chrono::steady_clock::now();
    ifstream infile(ssspPath, ios::in | ios::binary);
    infile.read((char *) &testNumNodes, sizeof(uint));
    uint numEdge = 0;
    infile.read((char *) &numEdge, sizeof(uint));
    testNumEdge = numEdge;
    cout << "vertex num: " << testNumNodes << " edge num: " << testNumEdge << endl;
    nodePointersI = new uint[testNumNodes];
    infile.read((char *) nodePointersI, sizeof(uint) * testNumNodes);
    gpuErrorcheck(hipMallocManaged(&edgeList, (numEdge) * sizeof(EdgeWithWeight)));
    hipMemAdvise(nodePointersI, (testNumNodes + 1) * sizeof(uint), hipMemAdviseSetReadMostly, 0);
    hipMemAdvise(edgeList, (numEdge) * sizeof(EdgeWithWeight), hipMemAdviseSetReadMostly, 0);
    infile.read((char *) edgeList, sizeof(EdgeWithWeight) * testNumEdge);
    infile.close();
    //preprocessData(nodePointersI, edgeList, testNumNodes, testNumEdge);
    auto endReadGraph = std::chrono::steady_clock::now();
    long durationReadGraph = std::chrono::duration_cast<std::chrono::milliseconds>(
            endReadGraph - startReadGraph).count();
    cout << "read graph time : " << durationReadGraph << "ms" << endl;
    int testTimes = 1;
    long timeSum = 0;
    for (int i = 0; i < testTimes; i++) {
        uint sourceNode = sourceNodeSample;
        cout << i << " sourceNode " << sourceNode << endl;
        ssspCaculateInShareTrace(testNumNodes, testNumEdge, nodePointersI, edgeList, sourceNode);
        break;
    }
}

long ssspCaculateInShareTrace(uint testNumNodes, uint testNumEdge, uint *nodePointersI, EdgeWithWeight *edgeList,
                              uint sourceNode) {
    cout << "==================ssspshare==============" << endl;
    auto start = std::chrono::steady_clock::now();
    uint *degree = new uint[testNumNodes];
    uint *value = new uint[testNumNodes];
    uint sourceCode = 0;

    auto startPreCaculate = std::chrono::steady_clock::now();
    for (uint i = 0; i < testNumNodes - 1; i++) {
        degree[i] = nodePointersI[i + 1] - nodePointersI[i];
    }

    degree[testNumNodes - 1] = testNumEdge - nodePointersI[testNumNodes - 1];
    sourceCode = sourceNode;
    bool *label = new bool[testNumNodes];
    for (uint i = 0; i < testNumNodes; i++) {
        label[i] = false;
        value[i] = UINT_MAX;
    }

    label[sourceCode] = true;
    value[sourceCode] = 1;
    uint *activeNodeListD;
    uint *degreeD;
    uint *valueD;
    bool *labelD;
    uint *nodePointersD;
    hipMalloc(&activeNodeListD, testNumNodes * sizeof(uint));
    hipMalloc(&nodePointersD, testNumNodes * sizeof(uint));
    hipMalloc(&degreeD, testNumNodes * sizeof(uint));
    hipMalloc(&valueD, testNumNodes * sizeof(uint));
    hipMalloc(&labelD, testNumNodes * sizeof(bool));
    hipMemcpy(degreeD, degree, testNumNodes * sizeof(uint), hipMemcpyHostToDevice);
    hipMemcpy(valueD, value, testNumNodes * sizeof(uint), hipMemcpyHostToDevice);
    hipMemcpy(labelD, label, testNumNodes * sizeof(bool), hipMemcpyHostToDevice);
    hipMemcpy(nodePointersD, nodePointersI, testNumNodes * sizeof(uint), hipMemcpyHostToDevice);
    //cacaulate the active node And make active node array
    uint *activeNodeLabelingD;
    gpuErrorcheck(hipMalloc(&activeNodeLabelingD, testNumNodes * sizeof(unsigned int)));
    uint *activeNodeLabelingPrefixD;
    gpuErrorcheck(hipMalloc(&activeNodeLabelingPrefixD, testNumNodes * sizeof(unsigned int)));
    dim3 grid = dim3(56, 1, 1);
    dim3 block = dim3(1024, 1, 1);

    auto endPreCaculate = std::chrono::steady_clock::now();
    long durationPreCaculate = std::chrono::duration_cast<std::chrono::milliseconds>(
            endPreCaculate - startPreCaculate).count();
    cout << "durationPreCaculate time : " << durationPreCaculate << " ms" << endl;

    setLabeling<<<grid, block>>>(testNumNodes, labelD, activeNodeLabelingD);
    thrust::device_ptr<unsigned int> ptr_labeling(activeNodeLabelingD);
    thrust::device_ptr<unsigned int> ptr_labeling_prefixsum(activeNodeLabelingPrefixD);
    uint activeNodesNum = thrust::reduce(ptr_labeling, ptr_labeling + testNumNodes);
    int iter = 0;
    uint nodeSum = activeNodesNum;
    while (activeNodesNum > 0) {
        iter++;
        thrust::exclusive_scan(ptr_labeling, ptr_labeling + testNumNodes, ptr_labeling_prefixsum);
        setActiveNodeArray<<<grid, block>>>(testNumNodes, activeNodeListD, labelD, activeNodeLabelingPrefixD);
        setLabelDefault<<<grid, block>>>(activeNodesNum, activeNodeListD, labelD);

        sssp_kernel<<<grid, block>>>(activeNodesNum, activeNodeListD, nodePointersD, degreeD, edgeList, valueD, labelD);
        hipDeviceSynchronize();
        gpuErrorcheck(hipPeekAtLastError());
        long temp = 0;
        for (uint j = 0; j < testNumEdge; j++) {
            temp += edgeList[j].toNode % 10;
        }
        cout << "iter " << iter << " " << temp;

        hipMemAdvise(edgeList, testNumEdge * sizeof(uint), hipMemAdviseSetAccessedBy,
                      hipCpuDeviceId);
        hipMemAdvise(edgeList, testNumEdge * sizeof(uint), hipMemAdviseUnsetAccessedBy,
                      hipCpuDeviceId);

        setLabeling<<<grid, block>>>(testNumNodes, labelD, activeNodeLabelingD);
        activeNodesNum = thrust::reduce(ptr_labeling, ptr_labeling + testNumNodes);
        nodeSum += activeNodesNum;
        cout << "iter: " << iter << " activeNodes: " << activeNodesNum << endl;
    }
    hipDeviceSynchronize();

    cout << "nodeSum: " << nodeSum << endl;

    auto endRead = std::chrono::steady_clock::now();
    long durationRead = std::chrono::duration_cast<std::chrono::milliseconds>(endRead - start).count();
    cout << "iter sum is " << iter << " finish time : " << durationRead << " ms" << endl;
    return durationRead;
}

long
ssspCaculateCommonMemoryInnerAsyncRandom(uint testNumNodes, uint testNumEdge, uint *nodePointersI,
                                         EdgeWithWeight *edgeList,
                                         uint sourceNode, float adviseK) {
    cout << "=========ssspCaculateCommonMemoryInnerAsyncRandom========" << endl;
    ulong edgeIterationMax = 0;
    auto start = std::chrono::steady_clock::now();
    auto startPreCaculate = std::chrono::steady_clock::now();
    ulong transferSum = 0;
    //CPU
    long durationRead;
    uint fragmentNum = testNumEdge / fragment_size;
    unsigned long max_partition_size;
    unsigned long total_gpu_size;
    uint staticFragmentNum;
    uint maxStaticNode = 0;
    uint *degree;
    uint *value;
    uint *label;
    uint *staticFragmentToNormalMap;
    bool *isInStatic;
    uint *overloadNodeList;
    uint *staticNodePointer;
    uint *staticFragmentData;
    uint *overloadFragmentData;
    uint *activeNodeList;
    uint *activeOverloadNodePointers;
    vector<PartEdgeListInfo> partEdgeListInfoArr;
    EdgeWithWeight *overloadEdgeList;
    FragmentData *fragmentData;
    bool isFromTail = true;
    //GPU
    EdgeWithWeight *staticEdgeListD;
    EdgeWithWeight *overloadEdgeListD;
    bool *isInStaticD;
    uint *overloadNodeListD;
    uint *staticNodePointerD;
    uint *nodePointerD;
    uint *staticFragmentVisitRecordsD;
    uint *staticFragmentDataD;
    uint *canSwapStaticFragmentDataD;
    uint *canSwapFragmentPrefixSumD;
    uint *degreeD;
    // async need two labels
    uint *isActiveD1;
    uint *isActiveD2;
    uint *isStaticActive;
    uint *isOverloadActive;
    uint *valueD;
    uint *activeNodeListD;
    uint *activeNodeLabelingPrefixD;
    uint *activeOverloadNodePointersD;
    uint *activeOverloadDegreeD;
    bool *isFinishedDevice;
    gpuErrorcheck(hipMalloc(&isFinishedDevice, 1 * sizeof(bool)));
    degree = new uint[testNumNodes];
    value = new uint[testNumNodes];
    label = new uint[testNumNodes];
    isInStatic = new bool[testNumNodes];
    overloadNodeList = new uint[testNumNodes];
    staticNodePointer = new uint[testNumNodes];
    activeNodeList = new uint[testNumNodes];
    activeOverloadNodePointers = new uint[testNumNodes];
    fragmentData = new FragmentData[fragmentNum];

    getMaxPartitionSize(max_partition_size, total_gpu_size, testNumNodes, adviseK, sizeof(EdgeWithWeight), testNumEdge,
                        15);

    staticFragmentNum = max_partition_size / fragment_size;
    staticFragmentToNormalMap = new uint[staticFragmentNum];
    staticFragmentData = new uint[staticFragmentNum];
    overloadFragmentData = new uint[fragmentNum];
    //caculate degree
    uint meanDegree = testNumEdge / testNumNodes;
    cout << " meanDegree " << meanDegree << endl;
    calculateDegree(testNumNodes, nodePointersI, testNumEdge, degree);
    //memcpy(staticNodePointer, nodePointersI, testNumNodes * sizeof(uint));
    uint edgesInStatic = 0;
    float startRate = (1 - (float) max_partition_size / (float) testNumEdge) / 2;
    uint startIndex = (float) testNumNodes * startRate;
    /*uint tempStaticSum = 0;
    for (uint i = testNumNodes - 1; i >= 0; i--) {
        tempStaticSum += degree[i];
        if (tempStaticSum > max_partition_size) {
            startIndex = i;
            break;
        }
    }*/
    //startIndex = 0;
    if (nodePointersI[startIndex] + max_partition_size > testNumEdge) {
        startIndex = (float) testNumNodes * 0.1f;
    }
    startIndex = (float) testNumNodes * 0.1f;
    for (uint i = 0; i < testNumNodes; i++) {
        label[i] = 0;
        value[i] = UINT_MAX;
        if (i >= startIndex && nodePointersI[i] < nodePointersI[startIndex] + max_partition_size - degree[i]) {
            isInStatic[i] = true;
            staticNodePointer[i] = nodePointersI[i] - nodePointersI[startIndex];
            if (i > maxStaticNode) {
                maxStaticNode = i;
            }
            edgesInStatic += degree[i];
        } else {
            isInStatic[i] = false;
        }
    }

    gpuErrorcheck(hipMalloc(&staticEdgeListD, max_partition_size * sizeof(EdgeWithWeight)));
    auto startmove = std::chrono::steady_clock::now();
    gpuErrorcheck(
            hipMemcpy(staticEdgeListD, edgeList + nodePointersI[startIndex],
                       max_partition_size * sizeof(EdgeWithWeight),
                       hipMemcpyHostToDevice));
    auto endMove = std::chrono::steady_clock::now();
    long testDuration = std::chrono::duration_cast<std::chrono::milliseconds>(
            endMove - startmove).count();
    cout << "move duration " << testDuration << endl;

    gpuErrorcheck(hipMalloc(&isInStaticD, testNumNodes * sizeof(bool)))
    gpuErrorcheck(hipMalloc(&overloadNodeListD, testNumNodes * sizeof(uint)));
    gpuErrorcheck(hipMalloc(&staticNodePointerD, testNumNodes * sizeof(uint)))
    gpuErrorcheck(
            hipMemcpy(staticNodePointerD, staticNodePointer, testNumNodes * sizeof(uint), hipMemcpyHostToDevice));
    hipMemcpy(isInStaticD, isInStatic, testNumNodes * sizeof(bool), hipMemcpyHostToDevice);
    label[sourceNode] = 1;
    value[sourceNode] = 1;
    uint partOverloadSize = total_gpu_size - max_partition_size;
    uint overloadSize = testNumEdge - edgesInStatic;
    cout << " partOverloadSize " << partOverloadSize << " overloadSize " << overloadSize << endl;
    //overloadEdgeList = (EdgeWithWeight *) malloc(overloadSize * sizeof(EdgeWithWeight));
    overloadEdgeList = new EdgeWithWeight[overloadSize];
    gpuErrorcheck(hipMalloc(&overloadEdgeListD, partOverloadSize * sizeof(EdgeWithWeight)));
    //gpuErrorcheck(hipMallocManaged(&edgeListOverloadManage, overloadSize * sizeof(uint)));
    gpuErrorcheck(hipMalloc(&staticFragmentDataD, staticFragmentNum * sizeof(uint)));
    gpuErrorcheck(hipMalloc(&staticFragmentVisitRecordsD, staticFragmentNum * sizeof(uint)));
    gpuErrorcheck(hipMalloc(&canSwapStaticFragmentDataD, staticFragmentNum * sizeof(uint)));
    gpuErrorcheck(hipMalloc(&canSwapFragmentPrefixSumD, staticFragmentNum * sizeof(uint)));
    thrust::device_ptr<unsigned int> ptr_canSwapFragment(canSwapStaticFragmentDataD);
    thrust::device_ptr<unsigned int> ptr_canSwapFragmentPrefixSum(canSwapFragmentPrefixSumD);
    gpuErrorcheck(hipMalloc(&degreeD, testNumNodes * sizeof(uint)));
    gpuErrorcheck(hipMalloc(&isActiveD1, testNumNodes * sizeof(uint)));
    gpuErrorcheck(hipMalloc(&isActiveD2, testNumNodes * sizeof(uint)));
    gpuErrorcheck(hipMalloc(&isStaticActive, testNumNodes * sizeof(uint)));
    gpuErrorcheck(hipMalloc(&isOverloadActive, testNumNodes * sizeof(uint)));
    gpuErrorcheck(hipMalloc(&valueD, testNumNodes * sizeof(uint)));
    gpuErrorcheck(hipMalloc(&activeNodeLabelingPrefixD, testNumNodes * sizeof(unsigned int)));
    gpuErrorcheck(hipMalloc(&activeNodeListD, testNumNodes * sizeof(uint)));
    gpuErrorcheck(hipMalloc(&activeOverloadNodePointersD, testNumNodes * sizeof(uint)));
    gpuErrorcheck(hipMalloc(&activeOverloadDegreeD, testNumNodes * sizeof(uint)));
    gpuErrorcheck(hipMemcpy(degreeD, degree, testNumNodes * sizeof(uint), hipMemcpyHostToDevice));
    gpuErrorcheck(hipMemcpy(valueD, value, testNumNodes * sizeof(uint), hipMemcpyHostToDevice));
    gpuErrorcheck(hipMemcpy(isActiveD1, label, testNumNodes * sizeof(uint), hipMemcpyHostToDevice));
    gpuErrorcheck(hipMemset(isActiveD2, 0, testNumNodes * sizeof(uint)));
    gpuErrorcheck(hipMemset(isStaticActive, 0, testNumNodes * sizeof(uint)));
    gpuErrorcheck(hipMemset(isOverloadActive, 0, testNumNodes * sizeof(uint)));

    //cacaulate the active node And make active node array
    dim3 grid = dim3(56, 1, 1);
    dim3 block = dim3(1024, 1, 1);

    //setLabeling<<<grid, block>>>(testNumNodes, labelD, activeNodeLabelingD);
    thrust::device_ptr<unsigned int> ptr_labeling(isActiveD1);
    thrust::device_ptr<unsigned int> ptr_labeling_static(isStaticActive);
    thrust::device_ptr<unsigned int> ptr_labeling_overload(isOverloadActive);
    thrust::device_ptr<unsigned int> ptr_labeling_prefixsum(activeNodeLabelingPrefixD);
    thrust::device_ptr<unsigned int> ptrOverloadDegree(activeOverloadDegreeD);
    thrust::device_ptr<unsigned int> ptrOverloadPrefixsum(activeOverloadNodePointersD);

    uint activeNodesNum = thrust::reduce(ptr_labeling, ptr_labeling + testNumNodes);
    int iter = 0;
    uint nodeSum = activeNodesNum;
    ulong overloadEdgeSum = 0;
    auto startCpu = std::chrono::steady_clock::now();
    auto endReadCpu = std::chrono::steady_clock::now();
    long durationReadCpu = 0;

    auto startSwap = std::chrono::steady_clock::now();
    auto endSwap = std::chrono::steady_clock::now();
    long durationSwap = 0;

    auto startGpuProcessing = std::chrono::steady_clock::now();
    auto endGpuProcessing = std::chrono::steady_clock::now();
    long durationGpuProcessing = 0;

    auto startOverloadGpuProcessing = std::chrono::steady_clock::now();
    auto endOverloadGpuProcessing = std::chrono::steady_clock::now();
    long durationOverloadGpuProcessing = 0;

    auto startPreGpuProcessing = std::chrono::steady_clock::now();
    auto endPreGpuProcessing = std::chrono::steady_clock::now();
    long durationPreGpuProcessing = 0;
    auto endPreCaculate = std::chrono::steady_clock::now();
    long durationPreCaculate = std::chrono::duration_cast<std::chrono::milliseconds>(
            endPreCaculate - startPreCaculate).count();
    cout << "durationPreCaculate time : " << durationPreCaculate << " ms" << endl;
    hipStream_t steamStatic, streamDynamic;
    hipStreamCreate(&steamStatic);
    hipStreamCreate(&streamDynamic);
    auto startMemoryTraverse = std::chrono::steady_clock::now();
    auto endMemoryTraverse = std::chrono::steady_clock::now();
    long durationMemoryTraverse = 0;
    uint cursorStartSwap = isFromTail ? fragmentNum - 1 : staticFragmentNum + 1;
    //uint cursorStartSwap = staticFragmentNum + 1;
    uint swapValidNodeSum = 0;
    uint swapValidEdgeSum = 0;
    uint swapNotValidNodeSum = 0;
    uint swapNotValidEdgeSum = 0;
    uint visitEdgeSum = 0;
    uint swapInEdgeSum = 0;
    long TIME = 0;
    int testTimes = 1;
    for (int testIndex = 0; testIndex < testTimes; testIndex++) {

        for (uint i = 0; i < testNumNodes; i++) {
            label[i] = 0;
            value[i] = UINT_MAX;
        }
        label[sourceNode] = 1;
        value[sourceNode] = 1;
        hipMemcpy(isInStaticD, isInStatic, testNumNodes * sizeof(bool), hipMemcpyHostToDevice);
        gpuErrorcheck(hipMemcpy(valueD, value, testNumNodes * sizeof(uint), hipMemcpyHostToDevice));
        gpuErrorcheck(hipMemcpy(isActiveD1, label, testNumNodes * sizeof(uint), hipMemcpyHostToDevice));
        gpuErrorcheck(hipMemset(isActiveD2, 0, testNumNodes * sizeof(uint)));
        gpuErrorcheck(hipMemset(isStaticActive, 0, testNumNodes * sizeof(uint)));
        gpuErrorcheck(hipMemset(isOverloadActive, 0, testNumNodes * sizeof(uint)));
        activeNodesNum = thrust::reduce(ptr_labeling, ptr_labeling + testNumNodes);
        iter = 0;

        auto startProcessing = std::chrono::steady_clock::now();
        auto startTest = std::chrono::steady_clock::now();
        auto endTest = std::chrono::steady_clock::now();
        long durationTest = 0;
        while (activeNodesNum > 0) {
            iter++;
            //cout << "iter " << iter << " activeNodesNum " << activeNodesNum << endl;
            startPreGpuProcessing = std::chrono::steady_clock::now();
            cleanStaticAndOverloadLabel<<<grid, block>>>(testNumNodes, isStaticActive, isOverloadActive);
            setStaticAndOverloadLabel<<<grid, block>>>(testNumNodes, isActiveD1, isStaticActive, isOverloadActive,
                                                       isInStaticD);
            uint staticNodeNum = thrust::reduce(ptr_labeling_static, ptr_labeling_static + testNumNodes);
            //cout << "iter " << iter << " staticNodeNum " << staticNodeNum << endl;
            thrust::exclusive_scan(ptr_labeling_static, ptr_labeling_static + testNumNodes, ptr_labeling_prefixsum);
            setStaticActiveNodeArray<<<grid, block>>>(testNumNodes, activeNodeListD, isStaticActive,
                                                      activeNodeLabelingPrefixD);

            uint overloadNodeNum = thrust::reduce(ptr_labeling_overload, ptr_labeling_overload + testNumNodes);
            //cout << "iter " << iter << " overloadNodeNum " << overloadNodeNum << endl;

            thrust::exclusive_scan(ptr_labeling_overload, ptr_labeling_overload + testNumNodes, ptr_labeling_prefixsum);
            setOverloadNodePointerSwap<<<grid, block>>>(testNumNodes, overloadNodeListD, activeOverloadDegreeD,
                                                        isOverloadActive,
                                                        activeNodeLabelingPrefixD, degreeD);

            thrust::exclusive_scan(ptrOverloadDegree, ptrOverloadDegree + overloadNodeNum, activeOverloadNodePointersD);
            uint overloadEdgeNum = thrust::reduce(thrust::device, ptrOverloadDegree,
                                                  ptrOverloadDegree + overloadNodeNum, 0);
            //cout << "iter " << iter << " overloadEdgeNum " << overloadEdgeNum << endl;
            overloadEdgeSum += overloadEdgeNum;
            if (overloadEdgeNum > edgeIterationMax) {
                edgeIterationMax = overloadEdgeNum;
            }

            endPreGpuProcessing = std::chrono::steady_clock::now();
            durationPreGpuProcessing += std::chrono::duration_cast<std::chrono::milliseconds>(
                    endPreGpuProcessing - startPreGpuProcessing).count();

            startGpuProcessing = std::chrono::steady_clock::now();
            mixDynamicPartLabel<<<grid, block, 0, steamStatic>>>(staticNodeNum, 0, activeNodeListD, isActiveD1,
                                                                 isActiveD2);

            /*sssp_kernelStaticSwapOpt2Label<<<grid, block, 0, steamStatic>>>(staticNodeNum, activeNodeListD,
                                                                            staticNodePointerD, degreeD,
                                                                            staticEdgeListD, valueD, isActiveD1, isActiveD2,
                                                                            isFinishedManaged);*/
            thread staticSSSPKernel = thread(ssspKernelThread, staticNodeNum, activeNodeListD, staticNodePointerD,
                                             degreeD,
                                             staticEdgeListD, valueD, isActiveD1, isActiveD2, isFinishedDevice, grid,
                                             block,
                                             steamStatic);

            if (overloadNodeNum > 0) {
                startCpu = std::chrono::steady_clock::now();
                /*hipMemcpyAsync(staticActiveNodeList, activeNodeListD, activeNodesNum * sizeof(uint), hipMemcpyDeviceToHost,
                                streamDynamic);*/
                hipMemcpyAsync(overloadNodeList, overloadNodeListD, overloadNodeNum * sizeof(uint),
                                hipMemcpyDeviceToHost,
                                streamDynamic);
                hipMemcpyAsync(activeOverloadNodePointers, activeOverloadNodePointersD, overloadNodeNum * sizeof(uint),
                                hipMemcpyDeviceToHost, streamDynamic);

                int threadNum = 20;
                if (overloadNodeNum < 50) {
                    threadNum = 1;
                }
                thread runThreads[threadNum];
                for (int i = 0; i < threadNum; i++) {
                    runThreads[i] = thread(ssspDynamic,
                                           i,
                                           threadNum,
                                           0,
                                           overloadNodeNum,
                                           degree,
                                           activeOverloadNodePointers,
                                           nodePointersI,
                                           overloadNodeList,
                                           overloadEdgeList,
                                           edgeList);
                }

                for (unsigned int t = 0; t < threadNum; t++) {
                    runThreads[t].join();
                }
                caculatePartInfoForEdgeList(activeOverloadNodePointers, overloadNodeList, degree, partEdgeListInfoArr,
                                            overloadNodeNum, partOverloadSize, overloadEdgeNum);

                endReadCpu = std::chrono::steady_clock::now();
                durationReadCpu += std::chrono::duration_cast<std::chrono::milliseconds>(endReadCpu - startCpu).count();
                if (staticSSSPKernel.joinable()) {
                    staticSSSPKernel.join();
                }
                hipDeviceSynchronize();
                endGpuProcessing = std::chrono::steady_clock::now();
                durationGpuProcessing += std::chrono::duration_cast<std::chrono::milliseconds>(
                        endGpuProcessing - startGpuProcessing).count();

                for (auto &i : partEdgeListInfoArr) {
                    startTest = std::chrono::steady_clock::now();
                    startMemoryTraverse = std::chrono::steady_clock::now();
                    gpuErrorcheck(hipMemcpy(overloadEdgeListD, overloadEdgeList +
                                                                activeOverloadNodePointers[i.partStartIndex],
                                             i.partEdgeNums * sizeof(EdgeWithWeight),
                                             hipMemcpyHostToDevice))
                    transferSum += i.partEdgeNums;
                    endMemoryTraverse = std::chrono::steady_clock::now();
                    durationMemoryTraverse += std::chrono::duration_cast<std::chrono::milliseconds>(
                            endMemoryTraverse - startMemoryTraverse).count();

                    startOverloadGpuProcessing = std::chrono::steady_clock::now();
                    mixDynamicPartLabel<<<grid, block, 0, streamDynamic>>>(i.partActiveNodeNums,
                                                                           i.partStartIndex,
                                                                           overloadNodeListD, isActiveD1,
                                                                           isActiveD2);
                    uint itr = 0;
                    bool isFinishedHost = true;
                    do {
                        itr++;
                        isFinishedHost = true;
                        hipMemcpy(isFinishedDevice, &isFinishedHost, sizeof(bool), hipMemcpyHostToDevice);

                        sssp_kernelDynamicSwap2Label<<<grid, block, 0, streamDynamic>>>(
                                i.partStartIndex,
                                i.partActiveNodeNums,
                                overloadNodeListD, degreeD,
                                valueD, itr % 2 == 1 ? isActiveD1
                                                     : isActiveD2,
                                itr % 2 == 1 ? isActiveD2
                                             : isActiveD1,
                                overloadEdgeListD,
                                activeOverloadNodePointersD,
                                isFinishedDevice);
                        hipDeviceSynchronize();
                        hipMemcpy(&isFinishedHost, isFinishedDevice, sizeof(bool), hipMemcpyDeviceToHost);
                        //cout << "dynamic itr " << itr << " *isFinishedManaged " << *isFinishedManaged << endl;
                        //*isFinishedManaged = true;
                        isFinishedHost = true;
                    } while (!isFinishedHost);
                    endOverloadGpuProcessing = std::chrono::steady_clock::now();
                    durationOverloadGpuProcessing += std::chrono::duration_cast<std::chrono::milliseconds>(
                            endOverloadGpuProcessing - startOverloadGpuProcessing).count();

                    endTest = std::chrono::steady_clock::now();
                    durationTest += std::chrono::duration_cast<std::chrono::milliseconds>(
                            endTest - startTest).count();
                }
                //gpuErrorcheck(hipPeekAtLastError())

            } else {
                if (staticSSSPKernel.joinable()) {
                    staticSSSPKernel.join();
                }
                endGpuProcessing = std::chrono::steady_clock::now();
                durationGpuProcessing += std::chrono::duration_cast<std::chrono::milliseconds>(
                        endGpuProcessing - startGpuProcessing).count();
            }


            startPreGpuProcessing = std::chrono::steady_clock::now();
            mixCommonLabel<<<grid, block, 0, streamDynamic>>>(testNumNodes, isActiveD1, isActiveD2);
            activeNodesNum = thrust::reduce(ptr_labeling, ptr_labeling + testNumNodes);
            nodeSum += activeNodesNum;
            endPreGpuProcessing = std::chrono::steady_clock::now();
            durationPreGpuProcessing += std::chrono::duration_cast<std::chrono::milliseconds>(
                    endPreGpuProcessing - startPreGpuProcessing).count();
        }
        auto endRead = std::chrono::steady_clock::now();
        durationRead = std::chrono::duration_cast<std::chrono::milliseconds>(endRead - startProcessing).count();
        hipDeviceSynchronize();

        cout << "nodeSum: " << nodeSum << endl;
        transferSum += max_partition_size;
        cout << "iterationSum " << iter << endl;
        double edgeIterationAvg = (double) overloadEdgeSum / (double) testNumEdge / iter;
        double edgeIterationMaxAvg = (double) edgeIterationMax / (double) testNumEdge;
        cout << "edgeIterationAvg " << edgeIterationAvg << " edgeIterationMaxAvg " << edgeIterationMaxAvg << endl;
        cout << "transferSum : " << transferSum * sizeof(EdgeWithWeight) << " bytes" << endl;
        cout << "finish time : " << durationRead << " ms" << endl;
        cout << "total time : " << testDuration + durationRead << " ms" << endl;
        cout << "cpu time : " << durationReadCpu << " ms" << endl;
        cout << "pre fact processing time : " << durationGpuProcessing << " ms" << endl;
        cout << "dynamic fact processing time : " << durationOverloadGpuProcessing << " ms" << endl;
        cout << "dynamic move time : " << durationMemoryTraverse << " ms" << endl;

        cout << "gpu pre processing time : " << durationPreGpuProcessing << " ms" << endl;
        cout << "swap processing time : " << durationSwap << " ms" << endl;
        cout << "overloadEdgeSum : " << overloadEdgeSum << " " << endl;
        cout << "durationTest : " << durationTest << " " << endl;

        cout << "swapValidNodeSum " << swapValidNodeSum << " swapValidEdgeSum " << swapValidEdgeSum << endl;
        cout << "swapNotValidNodeSum " << swapNotValidNodeSum << " swapNotValidEdgeSum " << swapNotValidEdgeSum
             << " visitSum " << visitEdgeSum << " swapInEdgeSum " << swapInEdgeSum << endl;
        TIME += durationRead;

    }
    cout << "TIME " << (float) TIME / (float) testTimes << endl;
    hipFree(nodePointerD);
    hipFree(staticEdgeListD);
    //hipFree(edgeListOverloadManage);
    hipFree(degreeD);
    hipFree(isActiveD1);
    hipFree(isActiveD2);
    hipFree(valueD);
    hipFree(activeNodeListD);
    hipFree(activeNodeLabelingPrefixD);
    hipFree(activeOverloadNodePointersD);
    hipFree(activeOverloadDegreeD);
    hipFree(isInStaticD);
    hipFree(staticNodePointerD);
    hipFree(overloadNodeListD);
    hipFree(staticFragmentVisitRecordsD);
    hipFree(staticFragmentDataD);
    hipFree(canSwapStaticFragmentDataD);
    hipFree(canSwapFragmentPrefixSumD);
    hipFree(overloadEdgeListD);
    hipFree(isStaticActive);
    hipFree(isOverloadActive);
    hipFree(isFinishedDevice);
    delete[]            label;
    delete[]            degree;
    delete[]            value;
    delete[]            activeNodeList;
    delete[]            activeOverloadNodePointers;
    delete[] staticFragmentData;
    delete[] isInStatic;
    delete[] overloadNodeList;
    delete[] staticNodePointer;
    delete[] staticFragmentToNormalMap;
    delete[] fragmentData;
    delete[] overloadFragmentData;
    delete[] overloadEdgeList;
    partEdgeListInfoArr.clear();
    return durationRead;
}